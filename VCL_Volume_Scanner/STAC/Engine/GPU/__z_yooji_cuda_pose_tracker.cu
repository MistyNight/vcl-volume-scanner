#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////
// z_yooji_cuda_volume_integrator.cpp
/////////////////////////////////////////////////////////////////////////////////////////////
//#include "_yooji_2017_cuda_object_scanner.cuh"
//#define __CUDASCAN__
#include "../../_yooji_2017_cuda_object_scanner.cuh"

__constant__ float K_dev[4];
__constant__ float T_gc_dev_c[16];
__constant__ float T_cg_dev_c[16];
__constant__ int dim_map_dev[2];

__constant__ float th_icp_dev[1];

// ////////////////////////////////////////////////////////////////////////////////
// Local functions .///////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////
// ===============================================================================
// Host functions.
// ===============================================================================
__host__ bool z_uit_Update_Incremental_Tracking(const float *sol_x, float *T)
//********************************************************************************************
{

 	static float tm[16];
 
 	const float *p_x = sol_x;
 	float *p_mat = T;
 	
 	/// CAUTION
 	// in_x is 6-d vector.	[ r1, r2, r3, t1, t2, t3 ]	
 	// + this transformation Tg,k transforms the k-th camera coordinates to global model coordinates.
 	//        |	1	r3	-r2	t1	|
 	// Tg,k = |	-r3	1	r1	t2	|
 	//        |	r2	-r1	1	t3	|
 	for(int i=0; i<16; i++)	tm[i] = p_mat[i];
 	
 	for(int i = 0; i<4; i++)	p_mat[i] = +1.0f	 *tm[i]	+p_x[2]	*tm[i+4]	-p_x[1]	*tm[i+8]	+p_x[3]*tm[i+12];
 	for(int i = 0; i<4; i++)	p_mat[i+4] = -p_x[2] *tm[i]	+1.0f	*tm[i+4]	+p_x[0]	*tm[i+8]	+p_x[4]*tm[i+12];
 	for(int i = 0; i<4; i++)	p_mat[i+8] = +p_x[1] *tm[i]	-p_x[0]	*tm[i+4]	+1.0f	*tm[i+8]	+p_x[5]*tm[i+12];

	//p_mat[12] = p_mat[13] =  p_mat[14] = 0.0f; p_mat[15] = 1.0f;
	
	return true;
}

// ===============================================================================
// Device functions.
// ===============================================================================
__device__ bool d_css_Compute_Single_Summand(
	int x,int y,
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_vertex_t0,
	const float *in_map_normal_t0,
	const float *in_T_cg_est,
	int ww,int hh,
	float in_th_icp,
	float *ATA,
	float *ATb,
	float *out_b)
{
	// for rigid transformation.
	Vector2f tpix,p2d_pred;	Vector3f tp3d,p3d_proj,p3d_pred,norm_pred_g;

	// for linear system.
	Vector3f A1,A2;
	float A[6] ={0.0f}; float b = 0.0f;  // A = [A1 A2]

	float td0,td1;
	int tidx,i,j;

	tpix.x = x; tpix.y = y;
	tidx = y*ww + x;

	// check depth validity of map t1.
	if((td1 = in_map_depth_t1[tidx]) < 1e-8f) return false;

	// compute 3d point back-projected from current estimated pose in the global coordinates.
	// + [p3d_proj]
	d_bp_Back_Project(tpix,K_dev,td1,tp3d);
	d_t_Transform(tp3d,in_T_cg_est,p3d_proj);
	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
	// + [p3d_pred]
	d_t_Transform(p3d_proj,T_gc_dev_c,tp3d);
	d_p_Project(tp3d,K_dev,p2d_pred);
	// + check depth validity.
	//	if(!d_gid_Get_Interpolated_Depth(p2d_pred,ww,hh,in_map_depth_t0,td0)) return false;
	// 	d_bp_Back_Project(p2d_pred,K_dev,td0,tp3d);
	if(!d_giv_Get_Interpolated_Vertex(p2d_pred,ww,hh,in_map_vertex_t0,tp3d)) return false;
	d_t_Transform(tp3d,T_cg_dev_c,p3d_pred);

	// /////////////////////////////////////////////////////////////////////////////
	// check distance between [p3d_proj] and [p3d_pred].
	td0 = SQUARE(p3d_pred.x - p3d_proj.x) + SQUARE(p3d_pred.y - p3d_proj.y) + SQUARE(p3d_pred.z - p3d_proj.z);
	//if(sqrtf(td0) > in_th_icp)	return false;
	if(td0 > in_th_icp)	return false;

	// check correlation between normals of [p3d_proj] and [p3d_pred].
	// + compute normal of [p3d_proj].

	// + compute normal of [p3d_pred].
	if(!d_gin_Get_Interpolated_Normal(p2d_pred,ww,hh,in_map_normal_t0,norm_pred_g)) return false;

	// generate 6x6 linear system by computing the derivative of the objective function (22).
	// refer to equation (24) of "KinectFusion: Real-Time Dense Surface Mapping and Tracking", ISMAR 2011.
	// + compute A.	

	A[0]= __fadd_rn(__fmul_rn(p3d_proj.z,norm_pred_g.y),__fmul_rn(-p3d_proj.y,norm_pred_g.z));
	A[1]= __fadd_rn(__fmul_rn(-p3d_proj.z,norm_pred_g.x),__fmul_rn(p3d_proj.x,norm_pred_g.z));
	A[2]= __fadd_rn(__fmul_rn(p3d_proj.y,norm_pred_g.x),__fmul_rn(-p3d_proj.x,norm_pred_g.y));
	A[3]= norm_pred_g.x;	A[4]= norm_pred_g.y;	A[5]= norm_pred_g.z;

	// + compute b.
	// b = Ng,t-1^T*(Vg,t-1 - Vg,t)
	float tsum,b1,b2,b3;

	//b = dot(norm_pred, (p3d_pred - p3d_proj));

	tp3d = p3d_pred - p3d_proj; b = 0.0f;
	b = __fmaf_rn(norm_pred_g.x,tp3d.x,b);
	b = __fmaf_rn(norm_pred_g.y,tp3d.y,b);
	b = __fmaf_rn(norm_pred_g.z,tp3d.z,b);

	// update linear system.
	// for ATA
	for(i = 0; i<6; i++) for(j = 0; j<6; j++){
		{
			ATA[i*6 + j] = __fmul_rn(A[i],A[j]);
		}
	}
	// for ATb
	for(i = 0; i<6; i++) ATb[i] = __fmul_rn(A[i],b);
	//////////////////////////////////////
	// for b
	out_b[0] = td0; // squared point-to-point distance.
	//////////////////////////////////////

	return true;
}

// __device__ bool d_css_Compute_Single_Summand(
// 	int x,int y,
// 	const float *in_map_depth_t1,
// 	const float *in_map_depth_t0,
// 	const float *in_map_vertex_t0,
// 	const float *in_map_normal_t0,
// 	const float *in_T_cg_est,
// 	int ww,int hh,
// 	float in_th_icp,
// 	float *ATA,
// 	float *ATb,
// 	float *out_b,
// 	bool *flag_inlier)
// {
// 	// for rigid transformation.
// 	Vector2f tpix,p2d_pred;	Vector3f tp3d,p3d_proj,p3d_pred,norm_pred_g;
// 
// 	// for linear system.
// 	Vector3f A1,A2;
// 	float A[6] ={0.0f}; float b = 0.0f;  // A = [A1 A2]
// 
// 	float td0,td1;
// 	int tidx,i,j;
// 
// 	tpix.x = x; tpix.y = y;
// 	tidx = y*ww + x;
// 
// 	// check depth validity of map t1.
// 	if((td1 = in_map_depth_t1[tidx]) < 1e-8f) return false;
// 
// 	// compute 3d point back-projected from current estimated pose in the global coordinates.
// 	// + [p3d_proj]
// 	d_bp_Back_Project(tpix,K_dev,td1,tp3d);
// 	d_t_Transform(tp3d,in_T_cg_est,p3d_proj);
// 	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
// 	// + [p3d_pred]
// 	d_t_Transform(p3d_proj,T_gc_dev_c,tp3d);
// 	d_p_Project(tp3d,K_dev,p2d_pred);
// 	// + check depth validity.
// //	if(!d_gid_Get_Interpolated_Depth(p2d_pred,ww,hh,in_map_depth_t0,td0)) return false;
// // 	d_bp_Back_Project(p2d_pred,K_dev,td0,tp3d);
// 	if(!d_giv_Get_Interpolated_Vertex(p2d_pred,ww,hh,in_map_vertex_t0,tp3d)) return false;
// 	d_t_Transform(tp3d,T_cg_dev_c,p3d_pred);
// 
// 	// /////////////////////////////////////////////////////////////////////////////
// 	// check distance between [p3d_proj] and [p3d_pred].
// 	td0 = SQUARE(p3d_pred.x - p3d_proj.x) + SQUARE(p3d_pred.y - p3d_proj.y) + SQUARE(p3d_pred.z - p3d_proj.z);
// 	//if(sqrtf(td0) > in_th_icp)	return false;
// 	if(td0 > in_th_icp)	return false;
// 
// 	// check correlation between normals of [p3d_proj] and [p3d_pred].
// 	// + compute normal of [p3d_proj].
// 
// 
// 
// 	// + compute normal of [p3d_pred].
// 	if(!d_gin_Get_Interpolated_Normal(p2d_pred,ww,hh,in_map_normal_t0,norm_pred_g)) return false;
// 
// 	// generate 6x6 linear system by computing the derivative of the objective function (22).
// 	// refer to equation (24) of "KinectFusion: Real-Time Dense Surface Mapping and Tracking", ISMAR 2011.
// 	// + compute A.	
// 
// 	A[0]= __fadd_rn(__fmul_rn(p3d_proj.z,norm_pred_g.y),__fmul_rn(-p3d_proj.y,norm_pred_g.z));
// 	A[1]= __fadd_rn(__fmul_rn(-p3d_proj.z,norm_pred_g.x),__fmul_rn(p3d_proj.x,norm_pred_g.z));
// 	A[2]= __fadd_rn(__fmul_rn(p3d_proj.y,norm_pred_g.x),__fmul_rn(-p3d_proj.x,norm_pred_g.y));
// 	A[3]= norm_pred_g.x;	A[4]= norm_pred_g.y;	A[5]= norm_pred_g.z;
// 
// 	// + compute b.
// 	// b = Ng,t-1^T*(Vg,t-1 - Vg,t)
// 	float tsum,b1,b2,b3;
// 
// 	//b = dot(norm_pred, (p3d_pred - p3d_proj));
// 
// 	tp3d = p3d_pred - p3d_proj; b = 0.0f;
// 	b = __fmaf_rn(norm_pred_g.x,tp3d.x,b);
// 	b = __fmaf_rn(norm_pred_g.y,tp3d.y,b);
// 	b = __fmaf_rn(norm_pred_g.z,tp3d.z,b);
// 
// 	// update linear system.
// 	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
// 	// This part is important.
// 	// Shared memory �Ἥ block ������ ��� �������� ��������
// 	// reduction �Ἥ block ���� sum ���ľ� ��.
// 	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
// 	// copy ATA and ATb values of current thread to shared memory.
// 	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
// 	// ������ ��� total thread number (=grid_dim * block dim) �� vector ��ü
// 	// size ���� �۱� ������ �� �κ��� �ʿ��ϴ�.
// 	// ����ó�� �̷��� while ������ control �ϴ� ���� �ʿ��Ѱ�?
// 	// �ϳ��� thread ���� �ϳ��� cache �� ������ �Ǵ°� �ƴ� �׳�?	
// 	//while(tx<ww && ty<hh){
// 
// 	// for ATA
// 	for(i = 0; i<6; i++) for(j = 0; j<6; j++){
// 		{
// 			ATA[i*6 + j] = __fmul_rn(A[i],A[j]);
// 		}
// 	}
// 	// for ATb
// 	for(i = 0; i<6; i++) ATb[i] = __fmul_rn(A[i],b);
// 	//////////////////////////////////////
// 	// for b
// 	out_b[0] = td0; // squared point-to-point distance.
// 	// for inlier.
// 	if(td0 < /*0.09**/in_th_icp) flag_inlier[0] = true;
// 	else flag_inlier[0] = false;
// 	//////////////////////////////////////
// 
// 	return true;
// }
__device__ bool d_css_Compute_Single_Summand(
	int x, int y, 
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_normal_t0,
	const float *in_T_cg_est,
	int ww, int hh,
	float in_th_icp,
	float *ATA,
	float *ATb,
	float *out_b,
	bool *flag_inlier)
{
	// for rigid transformation.
	Vector2f tpix, p2d_pred;	Vector3f tp3d, p3d_curr_g, p3d_prev_g, norm_pred_g;

	// for linear system.
	Vector3f A1,A2; 
	float A[6] ={0.0f}; float b = 0.0f;  // A = [A1 A2]

	float td0, td1;
	int tidx, i, j;

	tpix.x = x; tpix.y = y;
	tidx = y*ww + x;

	// check depth validity of map t1.
	if((td1 = in_map_depth_t1[tidx]) < 1e-8f) return false;

	// compute 3d point back-projected from current estimated pose in the global coordinates.
	// + [p3d_proj]
	d_bp_Back_Project(tpix, K_dev, td1, tp3d);
	d_t_Transform(tp3d, in_T_cg_est, p3d_curr_g);

	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
	// + [p3d_pred]
	d_t_Transform(p3d_curr_g,T_gc_dev_c,tp3d);
	d_p_Project(tp3d,K_dev,p2d_pred);

	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
	// + [p3d_pred]
	// with interpolation.
	if(!d_gid_Get_Interpolated_Depth(p2d_pred, ww, hh, in_map_depth_t0, td0)) return false;	
	d_bp_Back_Project(p2d_pred, K_dev, td0, tp3d);
	// without interpolation.
// 	i = __float2int_rn(p2d_pred.x);	j = __float2int_rn(p2d_pred.y);
// 	td0 = in_map_depth_t0[j*ww + i];
// 	p2d_pred.x = i; p2d_pred.y = j;
// 	d_bp_Back_Project(p2d_pred,K_dev,td0,tp3d);

	d_t_Transform(tp3d, T_cg_dev_c, p3d_prev_g);

	// /////////////////////////////////////////////////////////////////////////////
	// check distance between [p3d_proj] and [p3d_pred].
	td0 = SQUARE(p3d_prev_g.x - p3d_curr_g.x) + SQUARE(p3d_prev_g.y - p3d_curr_g.y) + SQUARE(p3d_prev_g.z - p3d_curr_g.z);
	//if(sqrtf(td0) > in_th_icp)	return false;
	if(td0 > in_th_icp)	return false;


	// /////////////////////////////////////////////////////////////////////////////

	// /////////////////////////////////////////////////////////////////////////////
	// /////////////////////////////////////////////////////////////////////////////


	// check correlation between normals of [p3d_proj] and [p3d_pred].
	// + compute normal of [p3d_proj].
	// �߰� !!!!!
	// �߰� !!!!!
	// �߰� !!!!!
	// �߰� !!!!!

	// + compute normal of [p3d_pred].
	if(!d_gin_Get_Interpolated_Normal(p2d_pred, ww, hh, in_map_normal_t0, norm_pred_g)) return false;

	// generate 6x6 linear system by computing the derivative of the objective function (22).
	// refer to equation (24) of "KinectFusion: Real-Time Dense Surface Mapping and Tracking", ISMAR 2011.
	// + compute A.	
	// A = [(Vg,t)x|I]^T*Ng,t-1
	// A1 = -(Vg,t)x*Ng | A2 = Ng,t-1.	
// 	A1 = -cross(p3d_proj, norm_pred); 	A2 = norm_pred;
// 	A[0] = A1.x; A[1] = A1.y; A[2] = A1.z;
// 	A[3] = A2.x; A[4] = A2.y; A[5] = A2.z;

	A[0]= __fadd_rn(__fmul_rn(p3d_curr_g.z,norm_pred_g.y),__fmul_rn(-p3d_curr_g.y,norm_pred_g.z));
	A[1]= __fadd_rn(__fmul_rn(-p3d_curr_g.z,norm_pred_g.x),__fmul_rn(p3d_curr_g.x,norm_pred_g.z));
	A[2]= __fadd_rn(__fmul_rn(p3d_curr_g.y,norm_pred_g.x),__fmul_rn(-p3d_curr_g.x,norm_pred_g.y));
	A[3]= norm_pred_g.x;	A[4]= norm_pred_g.y;	A[5]= norm_pred_g.z;

// 	A[0]= 0.0f; A[0] = __fmaf_rn(p3d_proj.z,norm_pred_g.y,A[0]); A[0] = __fmaf_rn(-p3d_proj.y,norm_pred_g.z,A[0]);
// 	A[1]= 0.0f; A[1] = __fmaf_rn(-p3d_proj.z,norm_pred_g.x,A[1]); A[1] = __fmaf_rn(p3d_proj.x,norm_pred_g.z,A[1]);
// 	A[2]= 0.0f; A[2] = __fmaf_rn(p3d_proj.y,norm_pred_g.x,A[2]); A[2] = __fmaf_rn(-p3d_proj.x,norm_pred_g.y,A[2]);
// 	A[3] = norm_pred_g.x;	A[4] = norm_pred_g.y;	A[5] = norm_pred_g.z;

// 	 	A[0] = +p3d_proj.z*norm_pred.y -p3d_proj.y*norm_pred.z;
// 	 	A[1] = -p3d_proj.z*norm_pred.x +p3d_proj.x*norm_pred.z;
// 	 	A[2] = +p3d_proj.y*norm_pred.x -p3d_proj.x*norm_pred.y;
// 	 	A[3] = norm_pred.x;	A[4] = norm_pred.y;	A[5] = norm_pred.z;

	// + compute b.
	// b = Ng,t-1^T*(Vg,t-1 - Vg,t)
	float tsum,b1,b2,b3;

	//b = dot(norm_pred, (p3d_pred - p3d_proj));

 	tp3d = p3d_prev_g - p3d_curr_g; b = 0.0f; 
	b = __fmaf_rn(norm_pred_g.x,tp3d.x,b); 
	b = __fmaf_rn(norm_pred_g.y,tp3d.y,b); 
	b = __fmaf_rn(norm_pred_g.z,tp3d.z,b);

// 	b1 = __fmul_rn(norm_pred_g.x,__fadd_rn(p3d_pred.x,-p3d_proj.x));
// 	b2 = __fmul_rn(norm_pred_g.y,__fadd_rn(p3d_pred.y,-p3d_proj.y));
// 	b3 = __fmul_rn(norm_pred_g.z,__fadd_rn(p3d_pred.z,-p3d_proj.z));
// 	tsum = __fadd_rn(b1,b2); b = __fadd_rn(tsum,b3);

// 		b = norm_pred.x*(p3d_pred.x-p3d_proj.x)
// 		+norm_pred.y*(p3d_pred.y-p3d_proj.y)
// 		+norm_pred.z*(p3d_pred.z-p3d_proj.z);

	// update linear system.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// This part is important.
	// Shared memory �Ἥ block ������ ��� �������� ��������
	// reduction �Ἥ block ���� sum ���ľ� ��.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// copy ATA and ATb values of current thread to shared memory.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// ������ ��� total thread number (=grid_dim * block dim) �� vector ��ü
	// size ���� �۱� ������ �� �κ��� �ʿ��ϴ�.
	// ����ó�� �̷��� while ������ control �ϴ� ���� �ʿ��Ѱ�?
	// �ϳ��� thread ���� �ϳ��� cache �� ������ �Ǵ°� �ƴ� �׳�?	
	//while(tx<ww && ty<hh){

	// for ATA
	for(i = 0; i<6; i++) for(j = 0; j<6; j++){
		{
			ATA[i*6 + j] = __fmul_rn(A[i],A[j]);
		}
	}
	// for ATb
	for(i = 0; i<6; i++) ATb[i] = __fmul_rn(A[i],b);
	//////////////////////////////////////
	// for b
	out_b[0] = td0; // squared point-to-point distance.
	// for inlier.
	if(td0 < /*0.09**/in_th_icp) flag_inlier[0] = true;
	else flag_inlier[0] = false;
	//////////////////////////////////////

	return true;
}

__global__ void g_gls_Generate_Linear_System(
	float *out_ATA_partial,
	float *out_ATb_partial,
	//float *out_b_partial,
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_vertex_t0,
	const float *in_map_normal_t0,
	const float *in_T_cg_est)
{
	// for computing overall summation.
	// 	__shared__ float cache_ATA[6*6*CV_CUDA_MAX_BLOCK_SIZE], cache_ATb[6*CV_CUDA_MAX_BLOCK_SIZE];
	// 	__shared__ int cache_cnt[CV_CUDA_MAX_BLOCK_SIZE];

	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;

	const int cidx = threadIdx.x + threadIdx.y*blockDim.x;
	const int bidx = blockIdx.x + blockIdx.y*gridDim.x;

	__shared__ float cache[CV_CUDA_MAX_BLOCK_SIZE];

	// for etc. parameters.	
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// initialize cache values.
	cache[cidx] = 0.0f;
	__syncthreads();

	// for constant variables.
	const int ww = dim_map_dev[0];
	const int hh = dim_map_dev[1];
	const float th_icp = th_icp_dev[0];

	const int dim_ATA = 6*6,dim_ATb = 6;
	const int dim_block = blockDim.x*blockDim.y;

	// for linear system.
	float tATA[6*6],tATb[6],tb[1];
	bool flag_valid = false,tinlier[1]={false};

	// initialize linear system.
	for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
	for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;

	if(tx >= 0 && tx < ww && ty >= 0 && ty < hh){
		flag_valid = d_css_Compute_Single_Summand(
			tx,ty,
			in_map_depth_t1,
			in_map_depth_t0,
			in_map_vertex_t0,
			in_map_normal_t0,
			in_T_cg_est,
			ww,hh,
			th_icp,
			tATA,
			tATb,
			tb);

		if(!flag_valid){
			for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
			for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;
			tb[0] = 0.0f;
		}
	}

	// set cache values.
	// for ATA
	for(int pidx = 0; pidx<dim_ATA; pidx++){
		cache[cidx] = tATA[pidx];
		__syncthreads();

		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
		while(i!=0){
			if(cidx < i){
				// i is idx in cache memory.
				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
				//cache[cidx] += cache[cidx + i];
			}
			__syncthreads();
			i /= 2;
		}

		// Update local block sum from shared memory to global memory.
		// Size of global memory should be block number.
		if(cidx == 0)	out_ATA_partial[bidx*dim_ATA + pidx] = cache[0];
	}
	__syncthreads();
	// for ATb
	for(int pidx = 0; pidx<6; pidx++){

		cache[cidx] = tATb[pidx];
		__syncthreads();

		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
		while(i!=0){
			if(cidx < i){
				// i is idx in cache memory.
				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
				//cache[cidx] += cache[cidx + i];
			}
			__syncthreads();
			i /= 2;
		}

		// Update local block sum from shared memory to global memory.
		// Size of global memory should be block number.
		if(cidx == 0)	out_ATb_partial[bidx*6 + pidx] = cache[0];
	}
	__syncthreads();

}

//__global__ void g_gls_Generate_Linear_System(
//	float *out_ATA_partial,
//	float *out_ATb_partial,
//	float *out_b_partial,
//	int *out_num_val_partial,
//	int *out_num_inlier_partial,
//	const float *in_map_depth_t1,
//	const float *in_map_depth_t0,
//	const float *in_map_vertex_t0,
//	const float *in_map_normal_t0,
//	const float *in_T_cg_est)
//{
//	// for computing overall summation.
//// 	__shared__ float cache_ATA[6*6*CV_CUDA_MAX_BLOCK_SIZE], cache_ATb[6*CV_CUDA_MAX_BLOCK_SIZE];
//// 	__shared__ int cache_cnt[CV_CUDA_MAX_BLOCK_SIZE];
//	
//	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
//	const int ty = threadIdx.y + blockIdx.y*blockDim.y;
//
//	const int cidx = threadIdx.x + threadIdx.y*blockDim.x;
//	const int bidx = blockIdx.x + blockIdx.y*gridDim.x;
//
//	__shared__ float cache[CV_CUDA_MAX_BLOCK_SIZE];
//
//	// for etc. parameters.	
//	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
//	// initialize cache values.
//	cache[cidx] = 0.0f;
//	__syncthreads();
//
//	// for constant variables.
//	const int ww = dim_map_dev[0];
//	const int hh = dim_map_dev[1];
//	const float th_icp = th_icp_dev[0];
//
//	const int dim_ATA = 6*6,dim_ATb = 6;
//	const int dim_block = blockDim.x*blockDim.y;
//
//	// for linear system.
//	float tATA[6*6],tATb[6],tb[1];
//	bool flag_valid = false,tinlier[1]={false};
//
//	// initialize linear system.
//	for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
//	for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;
//
//	if(tx >= 0 && tx < ww && ty >= 0 && ty < hh){
//		flag_valid = d_css_Compute_Single_Summand(
//			tx,ty,
//			in_map_depth_t1,
//			in_map_depth_t0,
//			in_map_vertex_t0,
//			in_map_normal_t0,
//			in_T_cg_est,
//			ww,hh,
//			th_icp,
//			tATA,
//			tATb,
//			tb,
//			tinlier);
//
//		if(!flag_valid){
//			for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
//			for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;
//			tb[0] = 0.0f;
//			tinlier[0] = false;
//		}
//	}
//
//	// set cache values.
//	// for ATA
//	for(int pidx = 0; pidx<dim_ATA; pidx++){
//		cache[cidx] = tATA[pidx];
//		__syncthreads();
//
//		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
//		while(i!=0){
//			if(cidx < i){
//				// i is idx in cache memory.
//				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
//				//cache[cidx] += cache[cidx + i];
//			}
//			__syncthreads();
//			i /= 2;
//		}
//
//		// Update local block sum from shared memory to global memory.
//		// Size of global memory should be block number.
//		if(cidx == 0)	out_ATA_partial[bidx*dim_ATA + pidx] = cache[0];
//	}
//	__syncthreads();
//	// for ATb
//	for(int pidx = 0; pidx<6; pidx++){
//
//		cache[cidx] = tATb[pidx];
//		__syncthreads();
//
//		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
//		while(i!=0){
//			if(cidx < i){
//				// i is idx in cache memory.
//				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
//				//cache[cidx] += cache[cidx + i];
//			}
//			__syncthreads();
//			i /= 2;
//		}
//
//		// Update local block sum from shared memory to global memory.
//		// Size of global memory should be block number.
//		if(cidx == 0)	out_ATb_partial[bidx*6 + pidx] = cache[0];
//	}
//	__syncthreads();
//	// for b
//	{
//		cache[cidx] = tb[0];
//		__syncthreads();
//
//		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
//		while(i!=0){
//			if(cidx < i){
//				// i is idx in cache memory.
//				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
//				//cache[cidx] += cache[cidx + i];
//			}
//			__syncthreads();
//			i /= 2;
//		}
//
//		// Update local block sum from shared memory to global memory.
//		// Size of global memory should be block number.
//		if(cidx == 0)	out_b_partial[bidx] = cache[0];
//	}
//	__syncthreads();
//	// for counting number of valid threads.
//	{
//		cache[cidx] = flag_valid;
//		__syncthreads();
//
//		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
//		while(i!=0){
//			if(cidx < i){
//				// i is idx in cache memory.
//				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
//				//cache[cidx] += cache[cidx + i];
//			}
//			__syncthreads();
//			i /= 2;
//		}
//
//		// Update local block sum from shared memory to global memory.
//		// Size of global memory should be block number.
//		if(cidx == 0)	out_num_val_partial[bidx] = int(cache[0]);
//	}
//	__syncthreads();
//	// for counting number of inlier threads.
//	{
//		cache[cidx] = tinlier[0];
//		__syncthreads();
//
//		int i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
//		while(i!=0){
//			if(cidx < i){
//				// i is idx in cache memory.
//				cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
//				//cache[cidx] += cache[cidx + i];
//			}
//			__syncthreads();
//			i /= 2;
//		}
//
//		// Update local block sum from shared memory to global memory.
//		// Size of global memory should be block number.
//		if(cidx == 0)	out_num_inlier_partial[bidx] = int(cache[0]);
//	}
//
////
//// 	tidx = ty*ww + tx;
//// 
//// 	// check depth validity of map t1.
//// 	if((td1 = in_map_depth_t1[tidx]) == 0.0f) return ;
//// 	
////  	tpix.x = tx;	tpix.y = ty;
////  	// compute 3d point back-projected from current estimated pose in the global coordinates.
////  	// + [p3d_proj]
////  	d_bp_Back_Project(tpix, K_dev, td1, tp3d);
////  	d_t_Transform(tp3d, in_T_cg_est, p3d_proj);
////  	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
////  	// + [p3d_pred]
////  	d_t_Transform(p3d_proj, T_gc_dev_c, tp3d);
////  	d_p_Project(tp3d, K_dev, p2d_pred);
//// 	// + check depth validity.
////  	if(!d_gid_Get_Interpolated_Depth(p2d_pred, ww, hh, in_map_depth_t0, td0)) return ;
////  	d_bp_Back_Project(p2d_pred, K_dev, td0, tp3d);
////  	d_t_Transform(tp3d, T_cg_dev_c, p3d_pred);
//// 
//// 	// check distance between [p3d_proj] and [p3d_pred].
//// 	if(length(p3d_pred - p3d_proj) > th_icp)	return ;
//// 
//// 	// check correlation between normals of [p3d_proj] and [p3d_pred].
//// 	// + compute normal of [p3d_proj].
//// 	// + compute normal of [p3d_pred].
//// 	if(!d_gin_Get_Interpolated_Normal(p2d_pred, ww, hh, in_map_normal_t0, norm_pred)) return ;
////
//// 	// generate 6x6 linear system by computing the derivative of the objective function (22).
//// 	// refer to equation (24) of "KinectFusion: Real-Time Dense Surface Mapping and Tracking", ISMAR 2011.
//// 	// + compute A.	
//// 	// A = [(Vg,t)x|I]^T*Ng,t-1
//// 	// A1 = -(Vg,t)x*Ng | A2 = Ng,t-1.	
////   	A1 = -cross(p3d_proj, norm_pred); 	A2 = norm_pred;
////   	A[0] = A1.x; A[1] = A1.y; A[2] = A1.z;
////   	A[3] = A2.x; A[4] = A2.y; A[5] = A2.z;
////
////// 	A[0] = +p3d_proj.z*norm_pred.y -p3d_proj.y*norm_pred.z;
////// 	A[1] = -p3d_proj.z*norm_pred.x +p3d_proj.x*norm_pred.z;
////// 	A[2] = +p3d_proj.y*norm_pred.x -p3d_proj.x*norm_pred.y;
////// 	A[3] = norm_pred.x;	A[4] = norm_pred.y;	A[5] = norm_pred.z;
////  
////  	// + compute b.
////  	// b = Ng,t-1^T*(Vg,t-1 - Vg,t)
////  	b = dot(norm_pred, (p3d_pred - p3d_proj));
////// 	b = norm_pred.x*(p3d_pred.x-p3d_proj.x)
////// 	+norm_pred.y*(p3d_pred.y-p3d_proj.y)
////// 	+norm_pred.z*(p3d_pred.z-p3d_proj.z);
////
////	// update linear system.
////	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
////	// This part is important.
////	// Shared memory �Ἥ block ������ ��� �������� ��������
////	// reduction �Ἥ block ���� sum ���ľ� ��.
////	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
////	for(i=0; i<6*6; i++) tATA[i] = 0.0f;
////	for(i=0; i<6; i++) tATb[i] = 0.0f;
////
////	// copy ATA and ATb values of current thread to shared memory.
////	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
////	// ������ ��� total thread number (=grid_dim * block dim) �� vector ��ü
////	// size ���� �۱� ������ �� �κ��� �ʿ��ϴ�.
////	// ����ó�� �̷��� while ������ control �ϴ� ���� �ʿ��Ѱ�?
////	// �ϳ��� thread ���� �ϳ��� cache �� ������ �Ǵ°� �ƴ� �׳�?	
////	//while(tx<ww && ty<hh){
////	
////	// for ATA
//// 	for(i=0; i<6; i++) for(j=0; j<6; j++){{
////		tATA[i*6 + j] = A[i]*A[j];
//// 	}}
////	// for ATb
////	for(i=0; i<6; i++) tATb[i] = A[i]*b;
////
////		//tx += blockDim.x;
////		//ty += blockDim.y;
////	//}
//// 
//// 	// ===========================================================
//// 	// We should add multiple matrices in parallel.
//// 	// This is an extension of summation of all vector elements.
//// 	// -----------------------------------------------------------
//// 	// for reduction, threadsPerBlock must be a power of 2.
//// 	// in our case, block is 2-dimensional.
//// 	// hence, thread number per block is blockDim.x*blockDim.y.
//// 	// ===========================================================
//// 	i = blockDim.x*blockDim.y/2;	// half of total thread number per block.
//// 	while(i!=0){
//// 		if(cidx < i){			
//// 			// j is element idx in matrix.
//// 			// i is idx in cache memory.
//// 			for(j=0; j<6*6; j++) cache_ATA[cidx*6*6 + j] += cache_ATA[(cidx + i)*6*6 + j];
//// 			for(j=0; j<6; j++) cache_ATb[cidx*6 + j] += cache_ATb[(cidx + i)*6 + j];
//// 			cache_cnt[cidx] += cache_cnt[cidx + i];
//// 		}
//// 		__syncthreads();
//// 		i/=2;
//// 	}
//// 
//// 	// Update local block sum from shared memory to global memory.
//// 	// Size of global memory should be block number.
//// 	if(cidx == 0){
//// 		tidx = blockIdx.x + blockIdx.y*gridDim.x;
//// 		for(j = 0; j<6*6; j++) out_ATA_partial[tidx*6*6 + j] = cache_ATA[j];
//// 		for(j = 0; j<6; j++) out_ATb_partial[tidx*6 + j] = cache_ATb[j];
//// 		out_num_val_partial[tidx] = cache_cnt[0];
//// 	}
//
//		
//}

/////////////////////////////////////////////////////////////////////////////////////////////
// LGKvVolumeIntegrator
/////////////////////////////////////////////////////////////////////////////////////////////

// *******************************************************
__host__ LGKvPoseTracker::LGKvPoseTracker()
// *******************************************************
{
	z_ATA_partial_dev.create(6, 6, 1); z_ATb_partial_dev.create(6, 1, 1);
	z_b_partial_dev.create(1, 1, 1);

	z_T_cg_est.create(4, 4, 1);
	z_T_cg_prev.create(4, 4, 1);
	//z_T_cg_est1.create(4, 4, 1);

	z_ATA_partial_host = z_ATb_partial_host = z_b_partial_host = NULL;
	z_num_partial_host = z_num_inlier_partial_host = NULL;
}

// *******************************************************
__host__ LGKvPoseTracker::~LGKvPoseTracker()
// *******************************************************
{
	if(z_ATA_partial_host) delete[] z_ATA_partial_host;
	if(z_ATb_partial_host) delete[] z_ATb_partial_host;
	if(z_b_partial_host) delete[] z_b_partial_host;

	if(z_num_partial_host) delete[] z_num_partial_host;
	if(z_num_inlier_partial_host) delete[] z_num_inlier_partial_host;
}


// *******************************************************
__host__ void LGKvPoseTracker::ip_Initialize_Parameters(
	int ww, int hh,
	float fx, float fy,
	float px, float py,
	float th_icp)
// *******************************************************
{
	float intrins_host[4];
	float th_icp_host[1];
	int dim_map_host[2];

	dim_map_host[0] = ww;	dim_map_host[1] = hh;
	intrins_host[0] = fx;	intrins_host[1] = fy;	intrins_host[2] = px;	intrins_host[3] = py;
	th_icp_host[0] = th_icp;		

	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_dev), dim_map_host, 2 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(K_dev), intrins_host, 4 * sizeof(float));	
	hipMemcpyToSymbol(HIP_SYMBOL(th_icp_dev), th_icp_host, sizeof(float)); 
	
}

// *******************************************************
__host__ bool LGKvPoseTracker::tp_Track_Pose(
	GKvTrackingState *io_track_state,
	GKvMatrixFloat *in_map_depth_t1)
	//float *in_map_depth_t1)
// *******************************************************
{
	static int iter_num[4] ={10,7,7,7};
	static int th_num[4] ={500,300,100,100};
	static float T_cg_est_host[16],T_gc_est_host[16];

	const float *map_d_t1 = in_map_depth_t1->vp(); // in_map_depth_t1;//
	const float *map_d_t0 = io_track_state->vp_map_depth();
	const float *map_ver_t0 = io_track_state->vp_map_vertex();
	const float *map_norm_t0 = io_track_state->vp_map_normal();

	// current pose to estimate.
	float *p_T_cg_est_dev = z_T_cg_est.vp();
	float *T_cg_est_dev;

	//float *ATA_partial_dev,*ATb_partial_dev,*b_partial_dev;
	int *num_partial_dev,*num_inlier_partial_dev;

	Vector3f cent_prev,cent_est;
	Vector2i sz = io_track_state->sz_map;
	float *p_T_gc_dev = io_track_state->vp_T_gc();
	float *p_T_cg_dev = io_track_state->vp_T_cg();

	bool flag_valid;

	// Camera pose at t0.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_c),p_T_gc_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_c),p_T_cg_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
	// Camera pose going to be estimated in current step.
	//hipMalloc((void**)&T_cg_est_dev, 16*sizeof(float));
	hipMemcpy(p_T_cg_est_dev,p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToDevice);
	hipMemcpy(T_cg_est_host,p_T_cg_est_dev,16*sizeof(float),hipMemcpyDeviceToHost);
	// Previous camera center.
	hipMemcpy(z_T_cg_prev.vp(),p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToDevice);
	d_gcc_Get_Camera_Center(T_cg_est_host,cent_prev);

	// Cuda kernel.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(sz.x,threads.x),iDivUp(sz.y,threads.y));
	int dim_block = blocks.x*blocks.y;
	//printf("dim_block: %d\n", dim_block);

	if(z_ATA_partial_dev.nch() != dim_block)	z_ATA_partial_dev.create(6,6,dim_block);
	if(z_ATb_partial_dev.nch() != dim_block)	z_ATb_partial_dev.create(6,1,dim_block);
	if(z_b_partial_dev.nch() != dim_block)		z_b_partial_dev.create(1,1,dim_block);

// 	hipMalloc((void**)&ATA_partial_dev,6*6*dim_block*sizeof(float));
// 	hipMalloc((void**)&ATb_partial_dev,6*dim_block*sizeof(float));
// 	hipMalloc((void**)&b_partial_dev,dim_block*sizeof(float));

//	hipMalloc((void**)&num_partial_dev,dim_block*sizeof(int));
//	hipMalloc((void**)&num_inlier_partial_dev,dim_block*sizeof(int));

	if(!z_ATA_partial_host) z_ATA_partial_host = new float[6*6*dim_block];
	if(!z_ATb_partial_host) z_ATb_partial_host = new float[6*dim_block];
	if(!z_b_partial_host) z_b_partial_host = new float[dim_block];

	//if(!z_num_partial_host) z_num_partial_host = new int[dim_block];
	//if(!z_num_inlier_partial_host) z_num_inlier_partial_host = new int[dim_block];

	flag_valid = true;

	for(int i=0; i<iter_num[0]; i++){

		g_gls_Generate_Linear_System<<<blocks,threads>>>(
			z_ATA_partial_dev.vp(), z_ATb_partial_dev.vp(), 
			//z_ATb_partial_dev.vp(),
			//ATA_partial_dev,ATb_partial_dev,b_partial_dev,
			//num_partial_dev,num_inlier_partial_dev,
			map_d_t1,
			map_d_t0,map_ver_t0,map_norm_t0,
			p_T_cg_est_dev);

		hipMemcpy(z_ATA_partial_host,z_ATA_partial_dev.vp(),6*6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATA_partial_dev.vp(), 
		hipMemcpy(z_ATb_partial_host,z_ATb_partial_dev.vp(),6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
		//hipMemcpy(z_b_partial_host,z_b_partial_dev.vp(),dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),

		//hipMemcpy(z_ATA_partial_host,ATA_partial_dev,6*6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATA_partial_dev.vp(), 
		//hipMemcpy(z_ATb_partial_host,ATb_partial_dev,6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),		
		//hipMemcpy(z_b_partial_host,b_partial_dev,dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),

		//hipMemcpy(z_num_partial_host,num_partial_dev,dim_block*sizeof(int),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
		//hipMemcpy(z_num_inlier_partial_host,num_inlier_partial_dev,dim_block*sizeof(int),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),

		// Add partial sums.
		for(int k=0; k<6*6; k++) z_ATA[k] = 0.0f;
		for(int k=0; k<6; k++) z_ATb[k] = 0.0f;
		//z_b = 0.0f;
		//z_num_val = 0;
		//z_num_inlier = 0;

		for(int k = 0; k<dim_block; k++){
			for(int j = 0; j<6*6; j++) z_ATA[j] += z_ATA_partial_host[k*6*6 + j];
			for(int j = 0; j<6; j++) z_ATb[j] += z_ATb_partial_host[k*6 + j];
			//z_b += z_b_partial_host[k];     // squared point-to-point distance
			//z_num_val += z_num_partial_host[k];
			//z_num_inlier += z_num_inlier_partial_host[k];
		}

		d_pm_Printf_Matrix(z_ATA, 6, 6, "ATA");
//  		printf("(#%d) z_ATb: %f %f %f | %f %f %f\n",
//  		z_num_val,z_ATb[0],z_ATb[1],z_ATb[2]
//  		,z_ATb[3],z_ATb[4],z_ATb[5]);

		//printf("z_num_val: %d\n", z_num_val);

		// Check number of valid pixels.
		//if(z_num_val < th_num[0]){ flag_valid = false; break; }

		// solve linear system.
		float norm2_x;
		d_lld_LL_Decomposition(z_ATA,6,z_L);
		//if(!d_sls_Solve_Linear_System_using_LLD(z_L,z_ATb,z_y,6,z_sol_x)) { flag_valid = false; break; }
		//////////////////////////////////////////////////////////////////////////
		// �� �κ��� �̻���.
		// Diagonal element �� 0 �� �� ó���� �� ���־�� �� ��??
		// �� 0 �� ����� ���ϱ�?
		flag_valid = d_sls_Solve_Linear_System_using_LLD(z_L,z_ATb,z_y,6,z_sol_x);
		//////////////////////////////////////////////////////////////////////////
		if(!flag_valid) break;

		// Check 2-norm of solution vector x.
		d_n2v_Norm_2_Vector(z_sol_x,6,norm2_x);
// 		printf("x: %f %f %f | %f %f %f\n"
// 				,z_sol_x[0],z_sol_x[1],z_sol_x[2]
// 				,z_sol_x[3],z_sol_x[4],z_sol_x[5]);
// 		printf("norm2_x: %f (%d valids)\n",norm2_x,z_num_val);
		if(norm2_x < 6.0e-3) break;

		// Update incremental transformation. (T_cg_est_dev)
		z_uit_Update_Incremental_Tracking(z_sol_x,T_cg_est_host);
		hipMemcpy(p_T_cg_est_dev,T_cg_est_host,16*sizeof(float),hipMemcpyHostToDevice);


		// 		for(int j = 0; j<6; j++) printf("%f ", z_sol_x[j]);
		// 		printf("\n");
		//for(int k=0; k<6*6; k++)
	}

	if(flag_valid){


		// �� �Լ� ���� �̻�
		io_track_state->set_transform(&z_T_cg_est,true);

		///////////////////////////////////////////////////////////////////////////////////
		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
		// detect motion drift.
		// Estimated current camera center.
		float dist,rmse_surf,inlier_perc;
		//cent_est = io_track_state->center;
		//dist = length(cent_est - cent_prev);

		//rmse_surf = sqrt(float(z_b)/(float)z_num_val);
		//inlier_perc = 100.0f*float(z_num_inlier)/float(z_num_val);

		rmse_surf = 0.00f;
		inlier_perc = 100.0f;
		//printf(" >>> rmse_surf: %f mm (%% %5.2f)\n",rmse_surf,inlier_perc);

		//if(dist > 0.175f){  // 0.5m*20(deg)*PI/180.0f.
		if(rmse_surf > 0.05f || inlier_perc < 95.0f){  // 0.5m*20(deg)*PI/180.0f.
			io_track_state->set_transform(&z_T_cg_prev,true);
			printf("=================================== Drift! ==========================================");
			return false;
		}
		///////////////////////////////////////////////////////////////////////////////////

		// Transform from camera to global coordinates.
		//  		hipMemcpy(p_T_cg_dev, p_T_cg_est_dev, 16*sizeof(float), hipMemcpyDeviceToDevice);
		//  		// Transform from global to camera coordinates.
		//  		d_im_Inverse_Matrix_4x4(T_cg_est_host, T_gc_est_host);
		//  		hipMemcpy(p_T_gc_dev, T_gc_est_host, 16*sizeof(float), hipMemcpyHostToDevice);
		//  		// Update camera center.
		//  		d_gcc_Get_Camera_Center(T_gc_est_host, io_track_state->center);
	}


// 	hipFree(ATA_partial_dev);
// 	hipFree(ATb_partial_dev);

//	hipFree(num_partial_dev);
//	hipFree(num_inlier_partial_dev);

	//hipFree(T_cg_est_dev);

	return flag_valid;
}


//// *******************************************************
//__host__ bool LGKvPoseTracker::tp_Track_Pose(
//	float *out_T_cg_t1,
//	Vector2i in_sz_map,
//	const float *in_T_gc_t0, const float *in_T_cg_t0,
//	const float *in_map_depth_t0,
//	const float *in_map_normal_t0,
//	const float *in_map_depth_t1)
//// *******************************************************
//{
//	/// LLD test //////////
//	static int iter_num[4] ={10,7,7,7};
//	static int th_num[4] ={2000,500,100,100};
//	static float T_cg_est_host[16],T_gc_est_host[16];
//
//	const float *map_d_t1 = in_map_depth_t1;//in_map_depth_t1->vp();
//	const float *map_d_t0 = in_map_depth_t0;
//	const float *map_norm_t0 = in_map_normal_t0;
//
//	// current pose to estimate.
//	float *p_T_cg_est_dev = out_T_cg_t1;
//
//	float *ATA_partial_dev,*ATb_partial_dev,*b_partial_dev;
//	int *num_partial_dev,*num_inlier_partial_dev;
//
//	Vector3f cent_prev,cent_est;
//	Vector2i sz = in_sz_map;
//	const float *p_T_gc_dev = in_T_gc_t0;
//	const float *p_T_cg_dev = in_T_cg_t0;
//
//	bool flag_valid;
//
//	// Camera pose at t0.
//	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_c),p_T_gc_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
//	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_c),p_T_cg_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
//	// Camera pose going to be estimated in current step.
//	//hipMalloc((void**)&T_cg_est_dev, 16*sizeof(float));
//	hipMemcpy(p_T_cg_est_dev,p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToDevice);
//	hipMemcpy(T_cg_est_host,p_T_cg_est_dev,16*sizeof(float),hipMemcpyDeviceToHost);
//	// Previous camera center.
//	hipMemcpy(z_T_cg_prev.vp(),p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToDevice);
//	d_gcc_Get_Camera_Center(T_cg_est_host,cent_prev);
//
//	// Cuda kernel.
//	dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
//	dim3 blocks(iDivUp(sz.x,threads.x),iDivUp(sz.y,threads.y));
//	int dim_block = blocks.x*blocks.y;
//	//printf("dim_block: %d\n", dim_block);
//
//	if(z_ATA_partial_dev.nch() != dim_block)	z_ATA_partial_dev.create(6,6,dim_block);
//	if(z_ATb_partial_dev.nch() != dim_block)	z_ATA_partial_dev.create(1,1,dim_block);
//
//	hipMalloc((void**)&ATA_partial_dev,6*6*dim_block*sizeof(float));
//	hipMalloc((void**)&ATb_partial_dev,6*dim_block*sizeof(float));
//	hipMalloc((void**)&b_partial_dev,dim_block*sizeof(float));
//	hipMalloc((void**)&num_partial_dev,dim_block*sizeof(int));
//	hipMalloc((void**)&num_inlier_partial_dev,dim_block*sizeof(int));
//
//	if(!z_ATA_partial_host) z_ATA_partial_host = new float[6*6*dim_block];
//	if(!z_ATb_partial_host) z_ATb_partial_host = new float[6*dim_block];
//	if(!z_b_partial_host) z_b_partial_host = new float[dim_block];
//	
//	if(!z_num_partial_host) z_num_partial_host = new int[dim_block];
//	if(!z_num_inlier_partial_host) z_num_inlier_partial_host = new int[dim_block];
//
//	flag_valid = true;
//
//	for(int i=0; i<iter_num[0]; i++){
//
//// 		g_gls_Generate_Linear_System<<<blocks,threads>>>(
//// 			//z_ATA_partial_dev.vp(), z_ATb_partial_dev.vp(), 
//// 			ATA_partial_dev,ATb_partial_dev,b_partial_dev,
//// 			num_partial_dev,num_inlier_partial_dev,
//// 			map_d_t1,
//// 			map_d_t0,map_norm_t0,
//// 			p_T_cg_est_dev);
//
//		hipMemcpy(z_ATA_partial_host,ATA_partial_dev,6*6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATA_partial_dev.vp(), 
//		hipMemcpy(z_ATb_partial_host,ATb_partial_dev,6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
//		hipMemcpy(z_b_partial_host,b_partial_dev,dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
//		hipMemcpy(z_num_partial_host,num_partial_dev,dim_block*sizeof(int),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
//		hipMemcpy(z_num_inlier_partial_host,num_inlier_partial_dev,dim_block*sizeof(int),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
//
//		// Add partial sums.
//		for(int k=0; k<6*6; k++) z_ATA[k] = 0.0f;
//		for(int k=0; k<6; k++) z_ATb[k] = 0.0f;
//		z_b = 0.0f;
//		z_num_val = 0;
//		z_num_inlier = 0;
//
//		for(int k = 0; k<dim_block; k++){
//			for(int j = 0; j<6*6; j++) z_ATA[j] += z_ATA_partial_host[k*6*6 + j];
//			for(int j = 0; j<6; j++) z_ATb[j] += z_ATb_partial_host[k*6 + j];
//			z_b += z_b_partial_host[k];     // squared point-to-point distance
//			z_num_val += z_num_partial_host[k];
//			z_num_inlier += z_num_inlier_partial_host[k];
//		}
//
//// 		 		printf("(#%d) z_ATb: %f %f %f | %f %f %f\n",
//// 		 		z_num_val,z_ATb[0],z_ATb[1],z_ATb[2]
//// 		 		,z_ATb[3],z_ATb[4],z_ATb[5]);
//
//		// Check number of valid pixels.
//		if(z_num_val < th_num[0]){ flag_valid = false; break; }
//
//		// solve linear system.
//		float norm2_x;
//		d_lld_LL_Decomposition(z_ATA,6,z_L);
//		if(!d_sls_Solve_Linear_System_using_LLD(z_L,z_ATb,z_y,6,z_sol_x)) { flag_valid = false; break; }
//
//		// Check 2-norm of solution vector x.
//		d_n2v_Norm_2_Vector(z_sol_x,6,norm2_x);
//		 	//	printf("x: %f %f %f | %f %f %f\n"
//		 	//			, z_sol_x[0], z_sol_x[1], z_sol_x[2]
//		 	//			, z_sol_x[3], z_sol_x[4], z_sol_x[5]);
//				//printf("norm2_x: %f (%d valids)\n", norm2_x, z_num_val);
//		if(norm2_x < 6.0e-3) break;
//
//		// Update incremental transformation. (T_cg_est_dev)
//		z_uit_Update_Incremental_Tracking(z_sol_x,T_cg_est_host);
//		hipMemcpy(p_T_cg_est_dev,T_cg_est_host,16*sizeof(float),hipMemcpyHostToDevice);
//
//
//		// 		for(int j = 0; j<6; j++) printf("%f ", z_sol_x[j]);
//		// 		printf("\n");
//		//for(int k=0; k<6*6; k++)
//	}
//
//	if(flag_valid){
//
//
//		// �� �Լ� ���� �̻�
//		//io_track_state->set_transform(&z_T_cg_est,true);
//
//		///////////////////////////////////////////////////////////////////////////////////
//		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
//		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
//		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
//		// detect motion drift.
//		// Estimated current camera center.
//// 		float dist,rmse_surf,inlier_perc;
//// 		//cent_est = io_track_state->center;
//// 		//dist = length(cent_est - cent_prev);
//// 
//// 		rmse_surf = sqrt(float(z_b)/(float)z_num_val);
//// 		inlier_perc = 100.0f*float(z_num_inlier)/float(z_num_val);
//// 		printf(" >>> rmse_surf: %f mm (%% %5.2f)\n",rmse_surf,inlier_perc);
//
//		//if(dist > 0.175f){  // 0.5m*20(deg)*PI/180.0f.
//		// 		if(rmse_surf > 0.05f || inlier_perc < 90.0f){  // 0.5m*20(deg)*PI/180.0f.
//		//  			io_track_state->set_transform(&z_T_cg_prev, true);
//		//  			return false;
//		//  		}
//		///////////////////////////////////////////////////////////////////////////////////
//
//		// Transform from camera to global coordinates.
//		//  		hipMemcpy(p_T_cg_dev, p_T_cg_est_dev, 16*sizeof(float), hipMemcpyDeviceToDevice);
//		//  		// Transform from global to camera coordinates.
//		//  		d_im_Inverse_Matrix_4x4(T_cg_est_host, T_gc_est_host);
//		//  		hipMemcpy(p_T_gc_dev, T_gc_est_host, 16*sizeof(float), hipMemcpyHostToDevice);
//		//  		// Update camera center.
//		//  		d_gcc_Get_Camera_Center(T_gc_est_host, io_track_state->center);
//	}
//
//
//	hipFree(ATA_partial_dev);
//	hipFree(ATb_partial_dev);
//	hipFree(num_partial_dev);
//
//	//hipFree(T_cg_est_dev);
//
//	return flag_valid;
//}
//
