#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////
// z_yooji_cuda_volume_integrator.cpp
/////////////////////////////////////////////////////////////////////////////////////////////
//#include "_yooji_2017_cuda_object_scanner.cuh"
//#define __CUDASCAN__
#include "../../_yooji_2017_cuda_object_scanner.cuh"

__constant__ float K_dev[4];
__constant__ float K_rgb_dev[4];
__constant__ float T_drgb_dev[16];

__constant__ float T_gc_dev_const[16];
__constant__ float T_cg_dev_const[16];
__constant__ int dim_map_dev[2];
__constant__ int dim_map_rgb_dev[2];

__constant__ float origin_dev[3];
__constant__ int dim_cube_dev[3];
__constant__ int dim_sc_dev[1];

__constant__ float mu_dev[1];
__constant__ float r_cube_dev[1];
__constant__ float sz_vox_dev[1];
__constant__ float sz_vox_inv_dev[1];
__constant__ float max_w_dev[1];


//********************************************************************************************
__device__ bool d_gpv_Get_Position_in_Voxel2(Vector3f in_p3d, 
	const float *origin, const int *dim_cube, const float sz_vox_inv,
	Vector3f &out_vox)
//********************************************************************************************
{
	bool valid = true;

	// assume that position of origin point in the world is (-0.5, -0.5, -0.5) in voxel coordinates..
// 	out_vox.x = (in_p3d.x - origin[0])*sz_vox_inv - 0.5f;
// 	out_vox.y = (in_p3d.y - origin[1])*sz_vox_inv - 0.5f;
// 	out_vox.z = (in_p3d.z - origin[2])*sz_vox_inv - 0.5f;

// 	out_vox.x = __fadd_rn(__fmul_rn(__fadd_rn(in_p3d.x,-origin[0]),sz_vox_inv),-0.5f);
// 	out_vox.y = __fadd_rn(__fmul_rn(__fadd_rn(in_p3d.y,-origin[1]),sz_vox_inv),-0.5f);
// 	out_vox.z = __fadd_rn(__fmul_rn(__fadd_rn(in_p3d.z,-origin[2]),sz_vox_inv),-0.5f);

	out_vox.x = __fmaf_rn(__fadd_rn(in_p3d.x, -origin[0]),sz_vox_inv, -0.5f);
	out_vox.y = __fmaf_rn(__fadd_rn(in_p3d.y, -origin[1]),sz_vox_inv, -0.5f);
	out_vox.z = __fmaf_rn(__fadd_rn(in_p3d.z, -origin[2]),sz_vox_inv, -0.5f);

	if(out_vox.x < -0.5f || out_vox.x >= (float)dim_cube[0] - 0.5f ||
		out_vox.y < -0.5f || out_vox.y >= (float)dim_cube[1] - 0.5f ||
		out_vox.z < -0.5f || out_vox.z >= (float)dim_cube[2] - 0.5f)
		valid = false;

	return valid;
}
//********************************************************************************************
__device__ bool d_gpw_Get_Position_in_World2(Vector3f in_vox, 
	const float *origin, const int *dim_cube, const float sz_vox,
	Vector3f &out_p3d)
//********************************************************************************************
{
	bool valid = true;

	if(in_vox.x < -0.5f || in_vox.x >= (float)dim_cube[0] - 0.5f ||
		in_vox.y < -0.5f || in_vox.y >= (float)dim_cube[1] - 0.5f ||
		in_vox.z < -0.5f || in_vox.z >= (float)dim_cube[2] - 0.5f)
		valid = false;

// 	out_p3d.x = (in_vox.x + 0.5f)*sz_vox + origin[0];
// 	out_p3d.y = (in_vox.y + 0.5f)*sz_vox + origin[1];
// 	out_p3d.z = (in_vox.z + 0.5f)*sz_vox + origin[2];

// 	out_p3d.x = __fadd_rn(__fmul_rn(__fadd_rn(in_vox.x,+0.5f),sz_vox),origin[0]);
// 	out_p3d.y = __fadd_rn(__fmul_rn(__fadd_rn(in_vox.y,+0.5f),sz_vox),origin[1]);
// 	out_p3d.z = __fadd_rn(__fmul_rn(__fadd_rn(in_vox.z,+0.5f),sz_vox),origin[2]);

	out_p3d.x = __fmaf_rn(__fadd_rn(in_vox.x, +0.5f),sz_vox, origin[0]);
	out_p3d.y = __fmaf_rn(__fadd_rn(in_vox.y, +0.5f),sz_vox, origin[1]);
	out_p3d.z = __fmaf_rn(__fadd_rn(in_vox.z, +0.5f),sz_vox, origin[2]);

	return valid;
}

__global__ void g_fvsc_Find_Visible_Sub_Cubes(
	bool *sub_cube,
	const float *map_depth)
{
	Vector2f tpix; Vector3f p3d, p3d_block_e, tp3d, tvox, dvec;

	const int sz_sc = dim_sc_dev[0];
	const float mu = mu_dev[0];
	const float sz_vox_inv = sz_vox_inv_dev[0];

	const int ww = dim_map_dev[0];
	const int hh = dim_map_dev[1];

	const int ww_sc = (dim_cube_dev[0] + sz_sc - 1)/sz_sc;
	const int hh_sc = (dim_cube_dev[1] + sz_sc - 1)/sz_sc;
	const int dd_sc = (dim_cube_dev[2] + sz_sc - 1)/sz_sc;

	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;

	bool valid = false;
	int tidx, n;
	int x_sc, y_sc, z_sc;

	float td;	

	if(tx < 0 || tx >= ww || ty < 0 || ty >= hh)	return;

	tidx = ty*ww + tx;

	// back-project depth.
	td = map_depth[tidx];
	if(td > 0.0f){
		tpix.x = tx;	tpix.y = ty;
		// compute direction vector of the pixel ray.
		d_bp_Back_Project(tpix, K_dev, td, tp3d);		d_t_Transform(tp3d, T_cg_dev_const, p3d);
		//d_bp_Back_Project(tpix, K_dev, 2.0f, tp3d);	    d_t_Transform(tp3d, T_cg_dev_c, p3d_block_e);
		d_bp_Back_Project(tpix, K_dev, 100.0f, tp3d);	    d_t_Transform(tp3d, T_cg_dev_const, p3d_block_e);

		dvec = (p3d_block_e - p3d).normalised();

		// activates sub-cubes on the pixel ray between +-mu of depth point.
		for(n = -4; n<=1; n++){
			
			tp3d = p3d + (n*mu)*dvec;

			//if(d_gpv_Get_Position_in_Voxel2(tp3d, origin_dev, dim_cube_dev, sz_vox_inv, tvox)){
			if(d_gpv_Get_Position_in_Voxel(tp3d, origin_dev, dim_cube_dev, sz_vox_inv, tvox)){

				x_sc = ROUNDF(tvox.x)/sz_sc;
				y_sc = ROUNDF(tvox.y)/sz_sc;
				z_sc = ROUNDF(tvox.z)/sz_sc;

				if(x_sc<0 || x_sc>=ww_sc ||
				   y_sc<0 || y_sc>=hh_sc ||
				   z_sc<0 || z_sc>=dd_sc) break;

				tidx = z_sc*ww_sc*hh_sc +y_sc*ww_sc +x_sc;

				sub_cube[tidx] = true;

			}

		}

	}


}

__global__ void g_ctv_Compute_TSDF_of_Voxels(
	float *cube_tsdf,
	uchar *cube_weight,
	const bool *sub_cube,	
	const float *map_depth)
{
	Vector2f tpix; Vector3f p3d, p3d_block_e, tp3d, tvox, dvec;

	const int sz_sc = dim_sc_dev[0];
	const float mu = mu_dev[0];
	const float max_w = max_w_dev[0];

	const int ww = dim_map_dev[0];
	const int hh = dim_map_dev[1];
	
	const int ww_c = dim_cube_dev[0];
	const int hh_c = dim_cube_dev[1];
	const int dd_c = dim_cube_dev[2];

	const int ww_sc = (ww_c + sz_sc - 1)/sz_sc;
	const int hh_sc = (hh_c + sz_sc - 1)/sz_sc;
	const int dd_sc = (dd_c + sz_sc - 1)/sz_sc;

	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;
	const int tz = threadIdx.z + blockIdx.z*blockDim.z;

	bool valid = false;
	int tidx, n;
	int x_sc, y_sc, z_sc, x_c, y_c, z_c;
	int x, y;

	float td, td_map, td_vox;
	float tsd, oldW, newW;
	
	if(tx < 0 || tx >= ww_sc || 
	   ty < 0 || ty >= hh_sc ||
	   tz < 0 || tz >= dd_sc)	return;

	//////////////////////////////////////////////////////////////////////////
	// �� ������ ���� GPU �ڵ� ���� ����� �̻��ϰ� ������ ��� �߻�...
	// �� ������ ���� GPU �ڵ� ���� ����� �̻��ϰ� ������ ��� �߻�...
 	// check sub-cube validity.
 	if(!sub_cube[tz*ww_sc*hh_sc +ty*ww_sc +tx])	return ;
	//////////////////////////////////////////////////////////////////////////

 	// inside of each sub-cube.
 	for(z_sc = 0; z_sc<sz_sc; z_sc++){
		z_c = tz*sz_sc + z_sc;
		for(y_sc = 0; y_sc<sz_sc; y_sc++){
 			y_c = ty*sz_sc + y_sc;
 
 			// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
 			// set range of X.
 			// 						tX_min = X_sc*sz_sc;
 			// 						tX_max = X_sc*sz_sc +sz_sc-1;
 			// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
 			for(x_sc = 0; x_sc<sz_sc; x_sc++){

				x_c = tx*sz_sc + x_sc;

 				tvox.x = x_c; tvox.y = y_c; tvox.z = z_c;
				//d_gpw_Get_Position_in_World2(tvox, origin_dev, dim_cube_dev, sz_vox_dev[0], tp3d);
				d_gpw_Get_Position_in_World(tvox, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], tp3d);
 
 				// transform voxel from global coord. to depth camera coord.
				d_t_Transform(tp3d, T_gc_dev_const, p3d);
 				// project transformed voxel to depth camera.
				d_p_Project(p3d, K_dev, tpix);
 
 				// uses nearest neighbor lookup to prevent smearing of measurements at depth discontinuities.
 				// + for depth
 				x = ROUNDF(tpix.x);	y = ROUNDF(tpix.y);

 				if(x<0 || x>ww-1 || y<0 || y>hh-1)		continue;
 
 				// ================================================================
 				td_map = map_depth[y*ww + x];		// get depth.
// 				//if(!z_gid_Get_Interpolated_Depth(pp.x, pp.y, ww, hh, p_depth, td_map))	continue;
// 				// ================================================================							
 				td_vox = p3d.z;
 
 				if(td_map>0.0f){

					// calculate TSD value.
 					td = td_map - td_vox;	// difference(m) between depth of voxel and depth of pixel where the voxel was projected.
 
 					/// /////////////////////////////////////
 					if(td<-mu)	continue;		// no surface information is obtained from this range.
 					tsd = fminf(1.0f, td/mu);		// TSD value.
 					/// /////////////////////////////////////				
 
 					// calculate new weight.
 					// + get global voxel index.
 					tidx = z_c*ww_c*hh_c + y_c*ww_c + x_c;
 
 					oldW = (float)cube_weight[tidx];
 					newW = oldW + 1.0f;
 
 					// update voxel information.
 					// + for depth.
 					cube_tsdf[tidx] = (oldW*cube_tsdf[tidx] + tsd)/newW;
 					cube_weight[tidx] = (uchar)fminf(newW, (float)max_w);
	
				}
			}
		}
	}
}


__global__ void g_ctvc_Compute_TSDF_of_Voxels_with_Color(
	float *cube_tsdf,
	uchar *cube_weight,
	uchar *cube_color,
	const bool *sub_cube,
	const float *map_depth,
	const uchar *img_color,
	bool flag_on_rgb = false)
{
	Vector2f tp2d; Vector2i tpix; Vector3f p3d_c,p3d_g,p3d_block_e,tp3d,tvox,dvec; 
	uchar trgb[3], prgb[3];
	float trgbf[3];

	const int sz_sc = dim_sc_dev[0];
	const float mu = mu_dev[0];
	const float max_w = max_w_dev[0];

	const int ww = dim_map_dev[0];
	const int hh = dim_map_dev[1];

	const int ww_rgb = dim_map_rgb_dev[0];
	const int hh_rgb = dim_map_rgb_dev[1];

	const int ww_c = dim_cube_dev[0];
	const int hh_c = dim_cube_dev[1];
	const int dd_c = dim_cube_dev[2];

	const int ww_sc = (ww_c + sz_sc - 1)/sz_sc;
	const int hh_sc = (hh_c + sz_sc - 1)/sz_sc;
	const int dd_sc = (dd_c + sz_sc - 1)/sz_sc;

	const int step_2d = ww_rgb*hh_rgb;
	const int step_3d = ww_c*hh_c*dd_c;

	// sub-cube index.
	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;
	const int tz = threadIdx.z + blockIdx.z*blockDim.z;

	bool valid = false;
	int tidx,n,x_rgb,y_rgb;
	int x_sc,y_sc,z_sc,x_c,y_c,z_c;
	int x,y;

	float sdf,td_map;
	float tsdf,oldW,newW,inewW,lambda_inv,norm;

	if(tx < 0 || tx >= ww_sc ||
	   ty < 0 || ty >= hh_sc ||
	   tz < 0 || tz >= dd_sc)	return;

	//////////////////////////////////////////////////////////////////////////
	// check sub-cube validity.
	// �̰� �� �̻��� �� ����.
	// ����� sub_cube validity�� ���������� update �ϸ鼭 ���� �ϴµ�,
	// ��� volume update ������ ���� frame ������ sub_cube validity ���� �ʿ��ϴ�.
	// �̺κ� �����ؾ��Ѵ�!
	if(!sub_cube[tz*ww_sc*hh_sc +ty*ww_sc +tx])	return ;
	//////////////////////////////////////////////////////////////////////////

	// inside of each sub-cube.
	for(z_sc = 0; z_sc<sz_sc; z_sc++){
		z_c = tz*sz_sc + z_sc;
		for(y_sc = 0; y_sc<sz_sc; y_sc++){
			y_c = ty*sz_sc + y_sc;

			for(x_sc = 0; x_sc<sz_sc; x_sc++){

				x_c = tx*sz_sc + x_sc;

				tvox.x = x_c; tvox.y = y_c; tvox.z = z_c;
				//d_gpw_Get_Position_in_World2(tvox, origin_dev, dim_cube_dev, sz_vox_dev[0], tp3d);
				d_gpw_Get_Position_in_World(tvox,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d_g);

				// transform voxel from global coord. to depth camera coord.
				d_t_Transform(p3d_g,T_gc_dev_const,p3d_c);
				// project transformed voxel to depth camera.
				d_p_Project(p3d_c,K_dev,tp2d);
				
				// uses nearest neighbor lookup to prevent smearing of measurements at depth discontinuities.
				// + for depth
				//x = ROUNDF(tpix.x);	y = ROUNDF(tpix.y);
				x = __float2int_rn(tp2d.x);	y = __float2int_rn(tp2d.y);

				if(x<0 || x>ww-1 || y<0 || y>hh-1)		continue;

				// ================================================================
				td_map = map_depth[y*ww + x];		// get depth.
				// 				//if(!z_gid_Get_Interpolated_Depth(pp.x, pp.y, ww, hh, p_depth, td_map))	continue;
				// 				// ================================================================							
				//td_vox = p3d_c.z;

				if(td_map>0.0f){

					// compute lambda.
					tpix.x = x; tpix.y = y;
					d_bp_Back_Project(tpix,K_dev,1,tp3d);
					lambda_inv = 1.0f/sqrtf(tp3d.x*tp3d.x + tp3d.y*tp3d.y + 1.0f);

					// calculate SDF value.
					// if voxel is behind depth map, td value is (-), else (+).
					//////////////////////////////////////////////////////////////////////////
					tp3d.x = p3d_g.x - T_cg_dev_const[3];
					tp3d.y = p3d_g.y - T_cg_dev_const[7];
					tp3d.z = p3d_g.z - T_cg_dev_const[11];
					norm = sqrtf(tp3d.x*tp3d.x + tp3d.y*tp3d.y + tp3d.z*tp3d.z);

					sdf = td_map - lambda_inv*norm;	// difference(m) between depth of voxel and depth of pixel where the voxel was projected.
					//////////////////////////////////////////////////////////////////////////


					/// /////////////////////////////////////
					// truncate SDF value.
					if(sdf<-mu)	continue;		// no surface information is obtained from this range.
					//if(td<-mu || td>mu)	continue;		// no surface information is obtained from this range.
					tsdf = fminf(1.0f,sdf/mu);		// TSDF value.
					//tsdf = (sdf > 0.0f) ? fminf(1.0f, sdf/mu) : -fminf(1.0f, sdf/mu);		// TSDF value.
					/// /////////////////////////////////////				

					// calculate new weight.
					// + get global voxel index.
					tidx = z_c*ww_c*hh_c + y_c*ww_c + x_c;

					oldW = (float)cube_weight[tidx];

					// weight for SDF integration.
					newW = oldW + 1.0f;
					inewW = 1.0f/newW;

					/////////////////////////////////////////////////////////////////////////
					if(oldW >= float(max_w)) continue;
					//if(abs(tsdf) > 0.5f) continue; // encode color value close to surface only.
					//////////////////////////////////////////////////////////////////////////

					// update voxel information.
					// + for depth.
					cube_tsdf[tidx] = (oldW*cube_tsdf[tidx] + tsdf)*inewW;///newW;					
					cube_weight[tidx] = (uchar)newW;
										
					// + for color.
 					if(!flag_on_rgb){
 						// transform voxel from depth camera coord. to RGB camera coord.
 						d_t_Transform(p3d_c,T_drgb_dev,tp3d);
 						// project transformed voxel on RGB camera.
 						d_p_Project(tp3d,K_rgb_dev,tp2d);
 
 						x_rgb = __float2int_rn(tp2d.x);	y_rgb = __float2int_rn(tp2d.y);
 						if(x_rgb<0 || x_rgb>ww_rgb-1 || y_rgb<0 || y_rgb>hh_rgb-1)		continue;
 					} 
					else{
 						//// project voxel on RGB camera.
 						//d_p_Project(p3d_c,K_rgb_dev,tpix);
 						//tx_rgb = __float2int_rn(tpix.x);	ty_rgb = __float2int_rn(tpix.y);
 						//if(tx_rgb<0 || tx_rgb>ww_rgb-1 || ty_rgb<0 || ty_rgb>hh_rgb-1)		continue;
						//x_rgb = x; y_rgb = y;
 					}

					// RGB.
					//////////////////////////////////////////////////////////////////////////
					// no problem.
					for(int k=0; k<3; k++) d_gii_Get_Interpolated_Intensity(tp2d,ww,hh,&img_color[k*ww*hh],trgbf[k]);

					//for(int k=0; k<3; k++)	trgb[k] = img_color[y_rgb*ww_rgb+x_rgb + k*step_2d];	// get color.
					//////////////////////////////////////////////////////////////////////////
					//////////////////////////////////////////////////////////////////////////
					// cube_color �� R channel cube_color[0:step_3d] �� ��������� �ʱ�ȭ�ȴ�!!! 
					//////////////////////////////////////////////////////////////////////////
					for(int k=0; k<3; k++)	prgb[k] = cube_color[tidx + k*step_3d];

// 					// weight for SDF integration.
// 					newW = oldW + exp(-abs(tsdf));
// 					inewW = 1.0f/newW;

					for(int k=0; k<3; k++){
						cube_color[tidx + k*step_3d] = (uchar)(fmax(0.0f, fmin(255.0f,(oldW*float(prgb[k]) + (trgbf[k]))*inewW)));
					}

// 					for(int k=0; k<3; k++){
// 						cube_color[tidx + k*step_3d] = trgb[k];
// 					}

					//////////////////////////////////////////////////////////////////////////
					// �ӽ� �������� G channel ���� R channel��!!
					// for R channel.
 					//cube_color[tidx] = (uchar)(sqrtf((oldW*SQUARE(float(prgb[1])) + SQUARE(float(trgb[1])))*inewW));
					//////////////////////////////////////////////////////////////////////////
// 					cube_color[tidx + step_3d] = trgb[1];
// 					cube_color[tidx + 2*step_3d] = trgb[2];
					//for(int k=0; k<3; k++)	cube_color[tidx + 0*ww_c*hh_c*dd_c] = (uchar)255;
					//cube_color[tidx] = 255;

				}
			}
		}
	}
}

__global__ void g_ctvc_Compute_TSDF_of_Voxels_with_Color(
	float *cube_tsdf,	
	uchar *cube_color,
	uchar *cube_weight,
	uchar *cube_weight_color,
	const bool *sub_cube,
	const float *map_depth,
	const uchar *img_color,
	bool flag_on_rgb = false)
{
	Vector2f tp2d; Vector2i tpix; Vector3f p3d_c,p3d_g,p3d_block_e,tp3d,tvox,dvec;
	uchar trgb[3],prgb[3];
	float trgbf[3];

	const int sz_sc = dim_sc_dev[0];
	const float mu = mu_dev[0];
	const float max_w = max_w_dev[0];

	const int ww = dim_map_dev[0];
	const int hh = dim_map_dev[1];

	const int ww_rgb = dim_map_rgb_dev[0];
	const int hh_rgb = dim_map_rgb_dev[1];

	const int ww_c = dim_cube_dev[0];
	const int hh_c = dim_cube_dev[1];
	const int dd_c = dim_cube_dev[2];

	const int ww_sc = (ww_c + sz_sc - 1)/sz_sc;
	const int hh_sc = (hh_c + sz_sc - 1)/sz_sc;
	const int dd_sc = (dd_c + sz_sc - 1)/sz_sc;

	const int step_2d = ww_rgb*hh_rgb;
	const int step_3d = ww_c*hh_c*dd_c;

	// sub-cube index.
	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;
	const int tz = threadIdx.z + blockIdx.z*blockDim.z;

	bool valid = false;
	int tidx,n,x_rgb,y_rgb;
	int x_sc,y_sc,z_sc,x_c,y_c,z_c;
	int x,y;

	float sdf,td_map;
	float tsdf,oldW,newW,inewW,lambda_inv,norm;

	if(tx < 0 || tx >= ww_sc ||
	   ty < 0 || ty >= hh_sc ||
	   tz < 0 || tz >= dd_sc)	return;

	//////////////////////////////////////////////////////////////////////////
	// check sub-cube validity.
	// �̰� �� �̻��� �� ����.
	// ����� sub_cube validity�� ���������� update �ϸ鼭 ���� �ϴµ�,
	// ��� volume update ������ ���� frame ������ sub_cube validity ���� �ʿ��ϴ�.
	// �̺κ� �����ؾ��Ѵ�!
	if(!sub_cube[tz*ww_sc*hh_sc +ty*ww_sc +tx])	return ;
	//////////////////////////////////////////////////////////////////////////

	// inside of each sub-cube.
	for(z_sc = 0; z_sc<sz_sc; z_sc++){
		z_c = tz*sz_sc + z_sc;
		for(y_sc = 0; y_sc<sz_sc; y_sc++){
			y_c = ty*sz_sc + y_sc;

			for(x_sc = 0; x_sc<sz_sc; x_sc++){

				x_c = tx*sz_sc + x_sc;

				tvox.x = x_c; tvox.y = y_c; tvox.z = z_c;
				//d_gpw_Get_Position_in_World2(tvox, origin_dev, dim_cube_dev, sz_vox_dev[0], tp3d);
				d_gpw_Get_Position_in_World(tvox,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d_g);

				// transform voxel from global coord. to depth camera coord.
				d_t_Transform(p3d_g,T_gc_dev_const,p3d_c);
				// project transformed voxel to depth camera.
				d_p_Project(p3d_c,K_dev,tp2d);

				// uses nearest neighbor lookup to prevent smearing of measurements at depth discontinuities.
				// + for depth
				//x = ROUNDF(tpix.x);	y = ROUNDF(tpix.y);
				x = __float2int_rn(tp2d.x);	y = __float2int_rn(tp2d.y);

				if(x<0 || x>ww-1 || y<0 || y>hh-1)		continue;

				// ================================================================
				td_map = map_depth[y*ww + x];		// get depth.
				// 				//if(!z_gid_Get_Interpolated_Depth(pp.x, pp.y, ww, hh, p_depth, td_map))	continue;
				// 				// ================================================================							
				//td_vox = p3d_c.z;

				if(td_map>0.0f){

					// compute lambda.
					tpix.x = x; tpix.y = y;
					d_bp_Back_Project(tpix,K_dev,1,tp3d);
					lambda_inv = 1.0f/sqrtf(tp3d.x*tp3d.x + tp3d.y*tp3d.y + 1.0f);

					// calculate SDF value.
					// if voxel is behind depth map, td value is (-), else (+).
					//////////////////////////////////////////////////////////////////////////
					tp3d.x = p3d_g.x - T_cg_dev_const[3];
					tp3d.y = p3d_g.y - T_cg_dev_const[7];
					tp3d.z = p3d_g.z - T_cg_dev_const[11];
					norm = sqrtf(tp3d.x*tp3d.x + tp3d.y*tp3d.y + tp3d.z*tp3d.z);

					sdf = td_map - lambda_inv*norm;	// difference(m) between depth of voxel and depth of pixel where the voxel was projected.
					//////////////////////////////////////////////////////////////////////////


					/// /////////////////////////////////////
					// truncate SDF value.
					if(sdf<-mu)	continue;		// no surface information is obtained from this range.
					//if(td<-mu || td>mu)	continue;		// no surface information is obtained from this range.
					tsdf = fminf(1.0f,sdf/mu);		// TSDF value.
					//tsdf = (sdf > 0.0f) ? fminf(1.0f, sdf/mu) : -fminf(1.0f, sdf/mu);		// TSDF value.
					/// /////////////////////////////////////				

					// calculate new weight.
					// + get global voxel index.
					tidx = z_c*ww_c*hh_c + y_c*ww_c + x_c;

					oldW = (float)cube_weight[tidx];

					// weight for SDF integration.
					newW = oldW + 1.0f;
					inewW = 1.0f/newW;

					/////////////////////////////////////////////////////////////////////////
					if(oldW >= float(max_w)) continue;
					//if(abs(tsdf) > 0.5f) continue; // encode color value close to surface only.
					//////////////////////////////////////////////////////////////////////////

					// update voxel information.
					// + for depth.
					cube_tsdf[tidx] = (oldW*cube_tsdf[tidx] + tsdf)*inewW;///newW;					
					cube_weight[tidx] = (uchar)newW;

					// + for color.
					if(!flag_on_rgb){
						// transform voxel from depth camera coord. to RGB camera coord.
						d_t_Transform(p3d_c,T_drgb_dev,tp3d);
						// project transformed voxel on RGB camera.
						d_p_Project(tp3d,K_rgb_dev,tp2d);

						x_rgb = __float2int_rn(tp2d.x);	y_rgb = __float2int_rn(tp2d.y);
						if(x_rgb<0 || x_rgb>ww_rgb-1 || y_rgb<0 || y_rgb>hh_rgb-1)		continue;
					} else{
						//// project voxel on RGB camera.
						//d_p_Project(p3d_c,K_rgb_dev,tpix);
						//tx_rgb = __float2int_rn(tpix.x);	ty_rgb = __float2int_rn(tpix.y);
						//if(tx_rgb<0 || tx_rgb>ww_rgb-1 || ty_rgb<0 || ty_rgb>hh_rgb-1)		continue;
						//x_rgb = x; y_rgb = y;
					}

					// RGB.
					//////////////////////////////////////////////////////////////////////////
					// no problem.
					for(int k=0; k<3; k++) d_gii_Get_Interpolated_Intensity(tp2d,ww,hh,&img_color[k*ww*hh],trgbf[k]);

					//for(int k=0; k<3; k++)	trgb[k] = img_color[y_rgb*ww_rgb+x_rgb + k*step_2d];	// get color.
					//////////////////////////////////////////////////////////////////////////
					//////////////////////////////////////////////////////////////////////////
					// cube_color �� R channel cube_color[0:step_3d] �� ��������� �ʱ�ȭ�ȴ�!!! 
					//////////////////////////////////////////////////////////////////////////
					for(int k=0; k<3; k++)	prgb[k] = cube_color[tidx + k*step_3d];

					// weight for SDF integration.
					newW = oldW + exp(-abs(tsdf));
					inewW = 1.0f/newW;

					for(int k=0; k<3; k++){
						//cube_color[tidx + k*step_3d] = (uchar)(sqrtf((oldW*SQUARE(float(prgb[k])) + SQUARE(float(trgbf[k])))*inewW));
						cube_color[tidx + k*step_3d] = (uchar)(sqrtf((oldW*SQUARE(float(prgb[k])) + SQUARE(float(trgbf[k])))*inewW));
					}

					// 					for(int k=0; k<3; k++){
					// 						cube_color[tidx + k*step_3d] = trgb[k];
					// 					}

					//////////////////////////////////////////////////////////////////////////
					// �ӽ� �������� G channel ���� R channel��!!
					// for R channel.
					//cube_color[tidx] = (uchar)(sqrtf((oldW*SQUARE(float(prgb[1])) + SQUARE(float(trgb[1])))*inewW));
					//////////////////////////////////////////////////////////////////////////
					// 					cube_color[tidx + step_3d] = trgb[1];
					// 					cube_color[tidx + 2*step_3d] = trgb[2];
					//for(int k=0; k<3; k++)	cube_color[tidx + 0*ww_c*hh_c*dd_c] = (uchar)255;
					//cube_color[tidx] = 255;

				}
			}
		}
	}
}

__global__ void g_uvv_Update_Valid_Volume(
	float *cube_tsdf,
	uchar *cube_weight,
	uchar *cube_color,
	const bool *vol_valid)
{
	const int sz_sc = dim_sc_dev[0];

	const int ww_c = dim_cube_dev[0];
	const int hh_c = dim_cube_dev[1];
	const int dd_c = dim_cube_dev[2];

	const int ww_sc = (ww_c + sz_sc - 1)/sz_sc;
	const int hh_sc = (hh_c + sz_sc - 1)/sz_sc;
	const int dd_sc = (dd_c + sz_sc - 1)/sz_sc;

	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;
	const int tz = threadIdx.z + blockIdx.z*blockDim.z;

	int tidx;
	int x_sc,y_sc,z_sc,x_c,y_c,z_c;
	int x,y;

	if(tx < 0 || tx >= ww_sc ||
	   ty < 0 || ty >= hh_sc ||
	   tz < 0 || tz >= dd_sc)	return;

	// inside of each sub-cube.
	for(z_sc = 0; z_sc<sz_sc; z_sc++){
		z_c = tz*sz_sc + z_sc;
		for(y_sc = 0; y_sc<sz_sc; y_sc++){
			y_c = ty*sz_sc + y_sc;

			// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
			// set range of X.
			// 						tX_min = X_sc*sz_sc;
			// 						tX_max = X_sc*sz_sc +sz_sc-1;
			// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
			for(x_sc = 0; x_sc<sz_sc; x_sc++){

				x_c = tx*sz_sc + x_sc;

				// calculate new weight.
				// + get global voxel index.
				tidx = z_c*ww_c*hh_c + y_c*ww_c + x_c;
				
				// re-initialize object model.
				if(!vol_valid[tidx]){
					cube_tsdf[tidx] = 1.0f;
					cube_weight[tidx] = 0.0f;
					for(int k=0; k<3; k++) cube_color[tidx + k*ww_c*hh_c*dd_c] = uchar(0);
				}
			}
		}
	}
}

__global__ void g_uvv_Update_Valid_Volume(
	float *cube_tsdf,
	uchar *cube_weight,
	uchar *cube_color,
	bool *cube_valid,
	const bool *vol_valid)
{
	const int sz_sc = dim_sc_dev[0];

	const int ww_c = dim_cube_dev[0];
	const int hh_c = dim_cube_dev[1];
	const int dd_c = dim_cube_dev[2];

	const int ww_sc = (ww_c + sz_sc - 1)/sz_sc;
	const int hh_sc = (hh_c + sz_sc - 1)/sz_sc;
	const int dd_sc = (dd_c + sz_sc - 1)/sz_sc;

	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;
	const int tz = threadIdx.z + blockIdx.z*blockDim.z;

	int tidx;
	int x_sc,y_sc,z_sc,x_c,y_c,z_c;
	int x,y;

	if(tx < 0 || tx >= ww_sc ||
	   ty < 0 || ty >= hh_sc ||
	   tz < 0 || tz >= dd_sc)	return;

	if(!cube_valid[tz*ww_sc*hh_sc +ty*ww_sc +tx])	return ;


	// inside of each sub-cube.
	int cnt = 0;
	// count valid voxels in current sub-cube.
	for(z_sc = 0; z_sc<sz_sc; z_sc++){
		z_c = tz*sz_sc + z_sc;
		for(y_sc = 0; y_sc<sz_sc; y_sc++){
			y_c = ty*sz_sc + y_sc;

			for(x_sc = 0; x_sc<sz_sc; x_sc++){

				x_c = tx*sz_sc + x_sc;

				// get global voxel index.
				tidx = z_c*ww_c*hh_c + y_c*ww_c + x_c;

				// count valid voxels in current sub-cube.
				if(vol_valid[tidx]) cnt++;

				
			}
		}
	}
	if(cnt > 0) return ;

	// re-initialize object model.
	cube_valid[tz*ww_sc*hh_sc +ty*ww_sc +tx] = false;

	for(z_sc = 0; z_sc<sz_sc; z_sc++){
		z_c = tz*sz_sc + z_sc;
		for(y_sc = 0; y_sc<sz_sc; y_sc++){
			y_c = ty*sz_sc + y_sc;

			for(x_sc = 0; x_sc<sz_sc; x_sc++){

				x_c = tx*sz_sc + x_sc;

				// calculate new weight.
				// + get global voxel index.
				tidx = z_c*ww_c*hh_c + y_c*ww_c + x_c;

				// re-initialize object model.
				if(!vol_valid[tidx]){
					cube_tsdf[tidx] = 1.0f;
					cube_weight[tidx] = 0.0f;
					for(int k=0; k<3; k++) cube_color[tidx + k*ww_c*hh_c*dd_c] = uchar(0);
				}
			}
		}
	}
}


/////////////////////////////////////////////////////////////////////////////////////////////
// LGKvVolumeIntegrator 
/////////////////////////////////////////////////////////////////////////////////////////////

// *******************************************************
__host__ LGKvVolumeIntegrator::LGKvVolumeIntegrator()
// *******************************************************
{

}

// *******************************************************
__host__ LGKvVolumeIntegrator::~LGKvVolumeIntegrator()
// *******************************************************
{

}


// *******************************************************
__host__ void LGKvVolumeIntegrator::ip_Initialize_Parameters(
	GKvObjectCubeFloat *in_cube,
	int ww, int hh,
	float fx, float fy,
	float px, float py,
	float mu,
	float max_w)
// *******************************************************
{
	float intrins_host[4], origin_host[3], light_host[3];
	float sz_vox_host[1], sz_vox_inv_host[1], mu_host[1], r_host[1], max_w_host[1];
	int dim_cube_host[3], dim_sc_host[1], dim_map_host[2];

	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();

	intrins_host[0] = fx;	intrins_host[1] = fy;	intrins_host[2] = px;	intrins_host[3] = py;
	origin_host[0] = cube_org.x;	origin_host[1] = cube_org.y;	origin_host[2] = cube_org.z;

	// 	printf("cube_org: %f %f %f\n", cube_org.x, cube_org.y, cube_org.z);
	// 	printf("cube_cen: %f %f %f\n", cube_cen.x, cube_cen.y, cube_cen.z);
	// 	printf("%f %f\n", mu, max_w);

	mu_host[0] = mu;
	sz_vox_host[0] = in_cube->sz_vox();
	sz_vox_inv_host[0] = 1.0f/in_cube->sz_vox();
	r_host[0] = length(cube_org - cube_cen);
	max_w_host[0] = max_w;

	in_cube->ts(dim_cube_host[0], dim_cube_host[1], dim_cube_host[2]);
	dim_map_host[0] = ww;	dim_map_host[1] = hh;
	dim_sc_host[0] = in_cube->dim_sc();

	hipMemcpyToSymbol(HIP_SYMBOL(K_dev), intrins_host, 4 * sizeof(float));
	//  	
	hipMemcpyToSymbol(HIP_SYMBOL(origin_dev), origin_host, 3 * sizeof(float));
	//  	
	hipMemcpyToSymbol(HIP_SYMBOL(mu_dev), mu_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(max_w_dev), max_w_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sz_vox_dev), sz_vox_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sz_vox_inv_dev), sz_vox_inv_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(r_cube_dev), r_host, sizeof(float));

	// 
	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_dev), dim_map_host, 2 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_cube_dev), dim_cube_host, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_sc_dev), dim_sc_host, sizeof(int));
}


// *******************************************************
__host__ void LGKvVolumeIntegrator::ip_Initialize_Parameters(
	GKvObjectCubeFloat *in_cube,
	int ww,int hh,
	float fx,float fy,
	float px,float py,
	int ww_rgb,int hh_rgb,
	float fx_rgb,float fy_rgb,
	float px_rgb,float py_rgb,
	const float *in_T_drgb,
	float mu,
	float max_w)
// *******************************************************
{
	float intrins_host[4],intrins_rgb_host[4],origin_host[3],light_host[3];
	float sz_vox_host[1],sz_vox_inv_host[1],mu_host[1],r_host[1],max_w_host[1];
	int dim_cube_host[3],dim_sc_host[1],dim_map_host[2],dim_map_rgb_host[2];

	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();

	intrins_host[0] = fx;	intrins_host[1] = fy;	
	intrins_host[2] = px;	intrins_host[3] = py;
	intrins_rgb_host[0] = fx_rgb;	intrins_rgb_host[1] = fy_rgb;	
	intrins_rgb_host[2] = px_rgb;	intrins_rgb_host[3] = py_rgb;

	origin_host[0] = cube_org.x;	origin_host[1] = cube_org.y;	origin_host[2] = cube_org.z;

	// 	printf("cube_org: %f %f %f\n", cube_org.x, cube_org.y, cube_org.z);
	// 	printf("cube_cen: %f %f %f\n", cube_cen.x, cube_cen.y, cube_cen.z);
	// 	printf("%f %f\n", mu, max_w);

	mu_host[0] = mu;
	sz_vox_host[0] = in_cube->sz_vox();
	sz_vox_inv_host[0] = 1.0f/in_cube->sz_vox();
	r_host[0] = length(cube_org - cube_cen);
	max_w_host[0] = max_w;

	in_cube->ts(dim_cube_host[0],dim_cube_host[1],dim_cube_host[2]);
	dim_map_host[0] = ww;	dim_map_host[1] = hh;
	dim_map_rgb_host[0] = ww_rgb;	dim_map_rgb_host[1] = hh_rgb;
	dim_sc_host[0] = in_cube->dim_sc();

	hipMemcpyToSymbol(HIP_SYMBOL(K_dev),intrins_host,4 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(K_rgb_dev),intrins_rgb_host,4 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(T_drgb_dev),in_T_drgb,sizeof(float) * 16);

	//  	
	hipMemcpyToSymbol(HIP_SYMBOL(origin_dev),origin_host,3 * sizeof(float));
	//  	
	hipMemcpyToSymbol(HIP_SYMBOL(mu_dev),mu_host,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(max_w_dev),max_w_host,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sz_vox_dev),sz_vox_host,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sz_vox_inv_dev),sz_vox_inv_host,sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(r_cube_dev),r_host,sizeof(float));

	// 
	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_dev),dim_map_host,2 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_rgb_dev),dim_map_rgb_host,2 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_cube_dev),dim_cube_host,3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_sc_dev),dim_sc_host,sizeof(int));
}

// *******************************************************
__host__ void LGKvVolumeIntegrator::cdtoc_Convert_Depth_to_TSDF_on_Cube(
	GKvObjectCubeFloat *io_cube,
	const float *in_map_depth_dev,
	int in_ww, int in_hh,
	const float *in_T_gc_dev,
	const float *in_T_cg_dev)
// *******************************************************
{
	int block_sz, grid_sz;
	int ww, hh, dd;

	io_cube->ts(ww, hh, dd);

	// Camera pose.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const), in_T_gc_dev, sizeof(float) * 16, 0, hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const), in_T_cg_dev, sizeof(float) * 16, 0, hipMemcpyDeviceToDevice);

	// For model cube update.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X, CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(in_ww, threads.x), iDivUp(in_hh, threads.y));
	
	g_fvsc_Find_Visible_Sub_Cubes<<<blocks, threads>>>(
		io_cube->vp_valid(),
		in_map_depth_dev);

	// For model cube update.
	dim3 threads2(CV_CUDA_BLOCK_SIZE_X, CV_CUDA_BLOCK_SIZE_Y, CV_CUDA_BLOCK_SIZE_Z);
	dim3 blocks2(iDivUp(ww, threads2.x), iDivUp(hh, threads2.y), iDivUp(dd, threads2.z));


	g_ctv_Compute_TSDF_of_Voxels<<<blocks2, threads2>>>(
		io_cube->vp_tsdf(),
		io_cube->vp_w(),
		io_cube->vp_valid(),		
		in_map_depth_dev);

}


// *******************************************************
__host__ void LGKvVolumeIntegrator::cdtocc_Convert_Depth_to_TSDF_on_Cube_with_Color(
	GKvObjectCubeFloat *io_cube,
	const float *in_map_depth_dev,
	const uchar *in_img_color_dev,
	int in_ww,int in_hh,
	const float *in_T_gc_dev,
	const float *in_T_cg_dev,
	bool in_flag_on_rgb)
// *******************************************************
{
	int block_sz,grid_sz;
	int ww,hh,dd;

	io_cube->ts(ww,hh,dd);

	// Camera pose.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const),in_T_gc_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const),in_T_cg_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);

	// For model cube update.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(in_ww,threads.x),iDivUp(in_hh,threads.y));

 	//////////////////////////////////////////////////////////////////////////
 	// initialize sub-cube!!!
 	// set voxel validity to true for current depth map only.
	//////////////////////////////////////////////////////////////////////////
	// �����!!! ����� sub cube dimension �ε� full cube dimension ���� ������
	// ������ ���� color cube ������ ħ���Ͽ� �ʱ�ȭ �Ѵ�!!!
	//////////////////////////////////////////////////////////////////////////
 	int3 dim_cube; 
	dim_cube.x = iDivUp(ww, io_cube->dim_sc()); 
	dim_cube.y = iDivUp(hh, io_cube->dim_sc()); 
	dim_cube.z = iDivUp(dd, io_cube->dim_sc());
	setDeviceMem3D(io_cube->vp_valid(),dim_cube,false);
 	//////////////////////////////////////////////////////////////////////////
 
 	g_fvsc_Find_Visible_Sub_Cubes<<<blocks,threads>>>(
 		io_cube->vp_valid(),
 		in_map_depth_dev);

	// For model cube update.
	dim3 threads2(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	dim3 blocks2(iDivUp(ww,threads2.x),iDivUp(hh,threads2.y),iDivUp(dd,threads2.z));

	g_ctvc_Compute_TSDF_of_Voxels_with_Color<<<blocks2,threads2>>>(
		io_cube->vp_tsdf(),
		io_cube->vp_w(),
		io_cube->vp_rgb(),
		io_cube->vp_valid(),
		in_map_depth_dev,
		in_img_color_dev,
		in_flag_on_rgb);


}

// *******************************************************

__host__ void LGKvVolumeIntegrator::uvv_Update_Valid_Volume(
	GKvObjectCubeFloat *io_cube,
	GKvVolumeBool *in_vol_valid)
// *******************************************************
{
	int ww,hh,dd;

	io_cube->ts(ww,hh,dd);

	// For model cube update.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	dim3 blocks(iDivUp(ww,threads.x),iDivUp(hh,threads.y),iDivUp(dd,threads.z));

// 	g_uvv_Update_Valid_Volume<<<blocks,threads>>>(
// 		io_cube->vp_tsdf(),
// 		io_cube->vp_w(),
// 		io_cube->vp_rgb(),
// 		in_vol_valid->vp());

	g_uvv_Update_Valid_Volume<<<blocks,threads>>>(
		io_cube->vp_tsdf(),
		io_cube->vp_w(),
		io_cube->vp_rgb(),
		io_cube->vp_valid(),
		in_vol_valid->vp());
}