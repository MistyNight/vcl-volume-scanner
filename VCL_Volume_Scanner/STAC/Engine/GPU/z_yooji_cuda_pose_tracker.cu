#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////
// z_yooji_cuda_volume_integrator.cpp
/////////////////////////////////////////////////////////////////////////////////////////////
//#include "_yooji_2017_cuda_object_scanner.cuh"
//#define __CUDASCAN__
#include "../../_yooji_2017_cuda_object_scanner.cuh"

__constant__ float K_dev[GK_LEVEL_OF_IMAGE_PYRAMID*4];
__constant__ float T_gc_dev_const[16];
__constant__ float T_cg_dev_const[16];
__constant__ int dim_map_dev[GK_LEVEL_OF_IMAGE_PYRAMID*2];

__constant__ float th_icp_dev[1];

// ////////////////////////////////////////////////////////////////////////////////
// Local functions .///////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////
// ===============================================================================
// Host functions.
// ===============================================================================
__host__ bool z_uit_Update_Incremental_Tracking(const float *sol_x, float *T)
//********************************************************************************************
{

 	static float tm[16];
 
 	const float *p_x = sol_x;
 	float *p_mat = T;
 	
 	/// CAUTION
 	// in_x is 6-d vector.	[ r1, r2, r3, t1, t2, t3 ]	
 	// + this transformation Tg,k transforms the k-th camera coordinates to global model coordinates.
 	//        |	1	r3	-r2	t1	|
 	// Tg,k = |	-r3	1	r1	t2	|
 	//        |	r2	-r1	1	t3	|
 	for(int i=0; i<16; i++)	tm[i] = p_mat[i];
 	
 	for(int i = 0; i<4; i++)	p_mat[i] = +1.0f	 *tm[i]	+p_x[2]	*tm[i+4]	-p_x[1]	*tm[i+8]	+p_x[3]*tm[i+12];
 	for(int i = 0; i<4; i++)	p_mat[i+4] = -p_x[2] *tm[i]	+1.0f	*tm[i+4]	+p_x[0]	*tm[i+8]	+p_x[4]*tm[i+12];
 	for(int i = 0; i<4; i++)	p_mat[i+8] = +p_x[1] *tm[i]	-p_x[0]	*tm[i+4]	+1.0f	*tm[i+8]	+p_x[5]*tm[i+12];

	//p_mat[12] = p_mat[13] =  p_mat[14] = 0.0f; p_mat[15] = 1.0f;
	
	return true;
}

//********************************************************************************************
// For KinectFusion.
// REFERENCE: Ma et al. An Invitation to 3D Vision.
__host__ bool z_uitme_Update_Incremental_Tracking_with_Matrix_Exponential(
	float *in_x,
	float *io_hmat_t0_to_t1)
//********************************************************************************************
{
	/// CAUTION
	// in_x is 6-d vector.	[ r1, r2, r3, t1, t2, t3 ]	
	// + this transformation Tg,k transforms the k-th camera coordinates to global model coordinates.
	//        |	1	-r3	r2	t1	|
	// Tg,k = |	r3	1	-r1	t2	|
	//        |	-r2	r1	1	t3	|

 	float *p_mat, *p_x;
 	float ts, ts2;
 	float tv3[3], tv3_2[3];
 	float tm3[9], tm3_2[9], tm4[16], tm4_2[16];
 	float exp_xi[16], exp_xi_inv[16], w[3], wx[9], wx_sq[9], exp_wx[9], v[3];
 	float R[9], t[3];
 	float norm;
 
 	p_x = in_x;
 	p_mat = io_hmat_t0_to_t1;		// 4x4 matrix.
 
 	//////////////////////////////////////////////////////////////////////////
 	/// BASIC EQUATION FOR ESTIMATING RIGID BODY MOTION.
 	// REFERENCE: Ma et al. An Invitation to 3D Vision.
 	// p. 29. Equation 2.30.
 	//////////////////////////////////////////////////////////////////////////
 	// in_x is 6-d twist vector.	[ w1, w2, w3, v1, v2, v3 ]	
 	//		  |	0		-w3		 w2		v1	|
 	// Xi =   |	w3		 0		-w1		v2	|
 	//        |	w2		 w1		 0		v3	|
 	//		  | 0		 0		 0		0	|
 
 	// T(t1) = exp( (t1-t0)*Xi )*T(t0).
 
 	//		  |   0		-w3		w2  |
 	// [w]x = |  w3		 0		-w1 |
 	//		  | -w2		w1		0   |
 
 	// exp( [w]x*s ) 
 	// = I + sin(s)*[w]x + (1 - cos(s))*[w]x^2.
 
 	// exp( Xi ) 
 	// = | exp([w]x*s)  (I-exp([w]x*s))*[w]x*v + w*w^T*v*s |
 	//   |    0                     1                |
 
 	// we set t = 1.
 
 	// normalized w.
 	for(int i = 0; i<3; i++)	w[i] = p_x[i];
 	d_n2v_Norm_2_Vector(w, 3, norm);
 	for(int i=0; i<3; i++)	w[i] /= norm;
 	// v normalized by norm of w.
 	for(int i=0; i<3; i++)	v[i] = p_x[3 + i]/norm;
 	// [w]x.
 	d_cvmcp_Convert_Vector3_to_Matrix_for_Cross_Product(w, wx);
 	// [w]x^2
 	d_cv_Copy_Vector(wx, 9, tm3);
 	d_mms_Multiply_Matrix_Square(wx, tm3, 3, wx_sq);
 	
 	// R = exp([w]x*s)
 	// + tm3 = sin(s)*[w]x + (1 - cos(s))*[w]x^2
 	d_avw_Add_Vector_Weighted(wx, wx_sq, sin(norm), (1.0f - cos(norm)), 9, tm3);
 	// + R = I + tm3
 	d_cv_Copy_Vector(tm3, 9, R);	R[0] += 1.0f;	R[4] += 1.0f; R[8] += 1.0f;
 
 	//d_pm_Printf_Matrix(p_x, 1, 6, "p_x");
 	//d_pm_Printf_Matrix(w, 1, 3, "w");
 	//d_pm_Printf_Matrix(wx, 3, 3, "wx");
 	//d_pm_Printf_Matrix(wx_sq, 3, 3, "wx_sq");
 	//d_pm_Printf_Matrix(tm3, 3, 3, "R");
 	//if (!Kv_Printf("Cost"))	exit(0);
 
 	// t = (I-exp([w]x*s))*[w]x*v + w*w^T*v*s
 	// + tv3 = (I-exp([w]x*s))*[w]x*v
 	// + I-exp([w]x*s) = -tm3
 	for(int i=0; i<9; i++)	tm3[i] *= -1.0f;
 	d_mms_Multiply_Matrix_Square(tm3, wx, 3, tm3_2);
 	d_mmsv_Multiply_Matrix_Square_Vector(tm3_2, v, 3, tv3);
 	// + tv3_2 = w*w^T*v*s
 	d_ipv_Inner_Product_Vector(w, v, 3, ts);	ts *= norm;
 	for(int i=0; i<3; i++)	tv3_2[i] = w[i]*ts;
 	// + t = tv3 + tv3_2
 	for(int i=0; i<3; i++)	t[i] = tv3[i] + tv3_2[i];
 
 	// T(t) = exp(Xi).
 	for(int j=0; j<3; j++)	for(int i=0; i<3; i++)	exp_xi[j*4 + i] = R[j*3 + i];
 	for(int j=0; j<3; j++)	exp_xi[j*4 + 3] = t[j];
 	exp_xi[12] = exp_xi[13] = exp_xi[14] = 0.0f;	exp_xi[15] = 1.0f;
 
 	// T(t1) = exp( (t1-t0)*Xi )*T(t0).
 	for(int i=0; i<16; i++)	tm4[i]=p_mat[i];
 
 	d_mms_Multiply_Matrix_Square(exp_xi, tm4, 4, p_mat);
	
	
	
	return true;
}

// ===============================================================================
// Device functions.
// ===============================================================================
__device__ bool d_cssbp_Compute_Single_Summand_with_Backward_Params(
	int x,int y,
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_vertex_t0,
	const float *in_map_normal_t0,
	const float *in_T_cg_est,
	int in_lev_of_pyram,
	int ww,int hh,
	float in_th_icp,
	float *ATA,
	float *ATb)
	//float *out_b,
	//bool *flag_inlier)
{
	// for rigid transformation.
	Vector2f tpix,p2d_pred;	Vector3f tp3d,p3d_proj,p3d_pred,norm_pred_g;

	// for linear system.
	float A[6] ={0.0f}; float b = 0.0f;  // A = [A1 A2]
	float *p_K_dev = &K_dev[in_lev_of_pyram*4];

	float td0,td1;
	int tidx,i,j;
	bool flag;

	tpix.x = x; tpix.y = y;
	tidx = y*ww + x;

	// check depth validity of map t1.
	//if((td1 = in_map_depth_t1[tidx]) < 1.0e-6f) return false;
	td1 = in_map_depth_t1[tidx];
	if(td1 < 1.0e-6f) return false;

	// compute 3d point back-projected from current estimated pose in the global coordinates.
	// + [p3d_proj]
	d_bp_Back_Project(tpix,p_K_dev,td1,tp3d);
	d_t_Transform(tp3d,in_T_cg_est,p3d_proj);
	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
	// + [p3d_pred]
	d_t_Transform(p3d_proj,T_gc_dev_const,tp3d);
	d_p_Project(tp3d,p_K_dev,p2d_pred);
	// + check depth validity.
	//	if(!d_gid_Get_Interpolated_Depth(p2d_pred,ww,hh,in_map_depth_t0,td0)) return false;
	// 	d_bp_Back_Project(p2d_pred,K_dev,td0,tp3d);

	//if(!d_giv_Get_Interpolated_Vertex(p2d_pred,ww,hh,in_map_vertex_t0,tp3d)) return false;
	flag = d_giv_Get_Interpolated_Vertex(p2d_pred,ww,hh,in_map_vertex_t0,tp3d);
	if(!flag) return false;
	d_t_Transform(tp3d,T_cg_dev_const,p3d_pred);

	// /////////////////////////////////////////////////////////////////////////////
	// check distance between [p3d_proj] and [p3d_pred].
	td0 = SQUARE(p3d_pred.x - p3d_proj.x) + SQUARE(p3d_pred.y - p3d_proj.y) + SQUARE(p3d_pred.z - p3d_proj.z);

	//////////////////////////////////////////////////////////////////////////
	if(td0 > in_th_icp)	return false;
	//////////////////////////////////////////////////////////////////////////

	// check correlation between normals of [p3d_proj] and [p3d_pred].
	// + compute normal of [p3d_proj].



	// + compute normal of [p3d_pred].
	//if(!d_gin_Get_Interpolated_Normal(p2d_pred,ww,hh,in_map_normal_t0,norm_pred_g)) return false;
	flag = d_gin_Get_Interpolated_Normal(p2d_pred,ww,hh,in_map_normal_t0,norm_pred_g);
	if(!flag) return false;

	//////////////////////////////////////////////////////////////////////////
	//norm_pred_g.x = norm_pred_g.y = 0.0f; norm_pred_g.z = -1.0f;
	//////////////////////////////////////////////////////////////////////////

	// generate 6x6 linear system by computing the derivative of the objective function (22).
	// refer to equation (24) of "KinectFusion: Real-Time Dense Surface Mapping and Tracking", ISMAR 2011.
	// + compute A.	

	A[0]= __fadd_rn(__fmul_rn(p3d_proj.z,norm_pred_g.y),__fmul_rn(-p3d_proj.y,norm_pred_g.z));
	A[1]= __fadd_rn(__fmul_rn(-p3d_proj.z,norm_pred_g.x),__fmul_rn(p3d_proj.x,norm_pred_g.z));
	A[2]= __fadd_rn(__fmul_rn(p3d_proj.y,norm_pred_g.x),__fmul_rn(-p3d_proj.x,norm_pred_g.y));
	A[3]= norm_pred_g.x;	A[4]= norm_pred_g.y;	A[5]= norm_pred_g.z;

	// + compute b.
	// b = Ng,t-1^T*(Vg,t-1 - Vg,t)
	//float tsum,b1,b2,b3;

	b = dot(norm_pred_g,(p3d_pred - p3d_proj));

	// 	tp3d = p3d_pred - p3d_proj; b = 0.0f;
	// 	b = __fmaf_rn(norm_pred_g.x,tp3d.x,b);
	// 	b = __fmaf_rn(norm_pred_g.y,tp3d.y,b);
	// 	b = __fmaf_rn(norm_pred_g.z,tp3d.z,b);

	// update linear system.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// This part is important.
	// Shared memory �Ἥ block ������ ��� �������� ��������
	// reduction �Ἥ block ���� sum ���ľ� ��.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// copy ATA and ATb values of current thread to shared memory.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// ������ ��� total thread number (=grid_dim * block dim) �� vector ��ü
	// size ���� �۱� ������ �� �κ��� �ʿ��ϴ�.
	// ����ó�� �̷��� while ������ control �ϴ� ���� �ʿ��Ѱ�?
	// �ϳ��� thread ���� �ϳ��� cache �� ������ �Ǵ°� �ƴ� �׳�?	
	//while(tx<ww && ty<hh){

	// for ATA
	for(i = 0; i<6; i++) for(j = 0; j<6; j++){
		{
			ATA[i*6 + j] = __fmul_rn(A[i],A[j]);
		}
	}
	// for ATb
	for(i = 0; i<6; i++) ATb[i] = __fmul_rn(A[i],b);
	//////////////////////////////////////
	// for b
	//out_b[0] = td0; // squared point-to-point distance.
	// for inlier.
	//if(td0 > /*0.09**/in_th_icp) return false;
	//////////////////////////////////////

	return true;
}

__device__ bool d_cssfp_Compute_Single_Summand_with_Forward_Params(
	int x,int y,
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_vertex_t0,
	const float *in_map_normal_t0,
	const float *in_T_21_est,
	int in_lev_of_pyram,
	int ww,int hh,
	float in_th_icp,
	float *ATA,
	float *ATb)
	//float *out_b,
	//bool *flag_inlier)
{
	// for rigid transformation.
	Vector2f tpix,p2d_t0;	Vector3f tp3d,p3d_t1,p3d_t0,norm_t0;

	// for linear system.
	float A[6] ={0.0f}; float b = 0.0f;  // A = [A1 A2]
	float *p_K_dev = &K_dev[in_lev_of_pyram*4];

	float td0,td1;
	int tidx,i,j;
	bool flag;

	tpix.x = x; tpix.y = y;
	tidx = y*ww + x;

	// check depth validity of map t1.
	//if((td1 = in_map_depth_t1[tidx]) < 1.0e-6f) return false;
	td1 = in_map_depth_t1[tidx];
	if(td1 < 1.0e-6f) return false;

	// compute 3d point back-projected from current estimated pose in the global coordinates.
	// + [p3d_proj]
	d_bp_Back_Project(tpix,p_K_dev,td1,tp3d);	// 2D cam t1->3D cam t1
	d_t_Transform(tp3d,in_T_21_est,p3d_t1);	// 3D cam t1->3D cam t0	
	// compute 3d point predicted from the global model and previous camera pose in the global coordinates.
	// + [p3d_pred]
	d_p_Project(p3d_t1,p_K_dev,p2d_t0);		// 3D cam t0->2D cam t0
	flag = d_giv_Get_Interpolated_Vertex(p2d_t0,ww,hh,in_map_vertex_t0,p3d_t0);		// 2D cam t0->3D glob t0
	if(!flag) return false;

	// /////////////////////////////////////////////////////////////////////////////
	// check distance between [p3d_proj] and [p3d_pred].
	td0 = SQUARE(p3d_t0.x - p3d_t1.x) + SQUARE(p3d_t0.y - p3d_t1.y) + SQUARE(p3d_t0.z - p3d_t1.z);

	//////////////////////////////////////////////////////////////////////////
	if(td0 > in_th_icp)	return false;
	//////////////////////////////////////////////////////////////////////////

	// check correlation between normals of [p3d_proj] and [p3d_pred].
	// + compute normal of [p3d_proj].



	// + compute normal of [p3d_pred].
	//if(!d_gin_Get_Interpolated_Normal(p2d_pred,ww,hh,in_map_normal_t0,norm_pred_g)) return false;
	flag = d_gin_Get_Interpolated_Normal(p2d_t0,ww,hh,in_map_normal_t0,norm_t0);
	if(!flag) return false;

	//////////////////////////////////////////////////////////////////////////
	//norm_pred_g.x = norm_pred_g.y = 0.0f; norm_pred_g.z = -1.0f;
	//////////////////////////////////////////////////////////////////////////

	// generate 6x6 linear system by computing the derivative of the objective function (22).
	// refer to equation (24) of "KinectFusion: Real-Time Dense Surface Mapping and Tracking", ISMAR 2011.
	// + compute A.	

	A[0]= __fadd_rn(__fmul_rn(+p3d_t1.z,norm_t0.y),__fmul_rn(-p3d_t1.y,norm_t0.z));
	A[1]= __fadd_rn(__fmul_rn(-p3d_t1.z,norm_t0.x),__fmul_rn(+p3d_t1.x,norm_t0.z));
	A[2]= __fadd_rn(__fmul_rn(+p3d_t1.y,norm_t0.x),__fmul_rn(-p3d_t1.x,norm_t0.y));
	A[3]= -norm_t0.x;	A[4]= -norm_t0.y;	A[5]= -norm_t0.z;

	// + compute b.
	// b = Ng,t-1^T*(Vg,t-1 - Vg,t)
	//float tsum,b1,b2,b3;

	b = dot(norm_t0,(p3d_t0 - p3d_t1));

	// 	tp3d = p3d_pred - p3d_proj; b = 0.0f;
	// 	b = __fmaf_rn(norm_pred_g.x,tp3d.x,b);
	// 	b = __fmaf_rn(norm_pred_g.y,tp3d.y,b);
	// 	b = __fmaf_rn(norm_pred_g.z,tp3d.z,b);

	// update linear system.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// This part is important.
	// Shared memory �Ἥ block ������ ��� �������� ��������
	// reduction �Ἥ block ���� sum ���ľ� ��.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// copy ATA and ATb values of current thread to shared memory.
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// ������ ��� total thread number (=grid_dim * block dim) �� vector ��ü
	// size ���� �۱� ������ �� �κ��� �ʿ��ϴ�.
	// ����ó�� �̷��� while ������ control �ϴ� ���� �ʿ��Ѱ�?
	// �ϳ��� thread ���� �ϳ��� cache �� ������ �Ǵ°� �ƴ� �׳�?	
	//while(tx<ww && ty<hh){

	// for ATA
	for(i = 0; i<6; i++) for(j = 0; j<6; j++){
		{
			ATA[i*6 + j] = __fmul_rn(A[i],A[j]);
		}
	}
	// for ATb
	for(i = 0; i<6; i++) ATb[i] = __fmul_rn(A[i],b);
	//////////////////////////////////////
	// for b
	//out_b[0] = td0; // squared point-to-point distance.
	// for inlier.
	//if(td0 > /*0.09**/in_th_icp) return false;
	//////////////////////////////////////

	return true;
}

__global__ void g_gls_Generate_Linear_System(
	float *out_ATA_partial,
	float *out_ATb_partial,
//	float *out_b_partial,
// 	int *out_num_val_partial,
// 	int *out_num_inlier_partial,
	int in_lev_of_pyram,
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_vertex_t0,
	const float *in_map_normal_t0,
	const float *in_T_cg_est)
{
	// for computing overall summation.
// 	__shared__ float cache_ATA[6*6*CV_CUDA_MAX_BLOCK_SIZE], cache_ATb[6*CV_CUDA_MAX_BLOCK_SIZE];
// 	__shared__ int cache_cnt[CV_CUDA_MAX_BLOCK_SIZE];
	
	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;

	const int cidx = threadIdx.x + threadIdx.y*blockDim.x;
	const int bidx = blockIdx.x + blockIdx.y*gridDim.x;

	__shared__ float cache[CV_CUDA_MAX_BLOCK_SIZE];

	// for etc. parameters.	
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// initialize cache values.
	cache[cidx] = 0.0f;
	__syncthreads();

	// for constant variables.
	//////////////////////////////////////////////////////////////////////////
	const int ww = dim_map_dev[in_lev_of_pyram*2 + 0];
	const int hh = dim_map_dev[in_lev_of_pyram*2 + 1];
	//////////////////////////////////////////////////////////////////////////
	const float th_icp = th_icp_dev[0];

	const int dim_ATA = 6*6,dim_ATb = 6;
	const int dim_block = blockDim.x*blockDim.y;

	// for linear system.
	float tATA[6*6],tATb[6],tb[1]={0.0f};
	bool flag_valid = false,tinlier[1]={false};

	// initialize linear system.
	for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
	for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;

	if(tx >= 0 && tx < ww && ty >= 0 && ty < hh){
		flag_valid = d_cssfp_Compute_Single_Summand_with_Forward_Params(
			tx,ty,
			in_map_depth_t1,
			in_map_depth_t0,
			in_map_vertex_t0,
			in_map_normal_t0,
			in_T_cg_est,
			in_lev_of_pyram,
			ww,hh,
			th_icp,
			tATA,
			tATb);
			//tb,
			//tinlier);

		if(!flag_valid){
			for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
			for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;
			//tb[0] = 0.0f;
			//tinlier[0] = false;
		}
	}

	// set cache values.
	// for ATA
	for(int pidx = 0; pidx<dim_ATA; pidx++){
		cache[cidx] = tATA[pidx];
		__syncthreads();

		int i = dim_block/2;	// half of total thread number per block.
		while(i!=0){
			if(cidx < i){
				// i is idx in cache memory.
				//cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
				cache[cidx] += cache[cidx + i];
			}
			//////////////////////////////////////////////////////////////////////////
			if(i>32) __syncthreads();
			//////////////////////////////////////////////////////////////////////////
			i /= 2;
		}

		// Update local block sum from shared memory to global memory.
		// Size of global memory should be block number.
		if(cidx == 0)	out_ATA_partial[bidx*dim_ATA + pidx] = cache[0];
		__syncthreads();
	}	

	// for ATb
	for(int pidx = 0; pidx<6; pidx++){

		cache[cidx] = tATb[pidx];
		__syncthreads();

		int i = dim_block/2;	// half of total thread number per block.
		while(i!=0){
			if(cidx < i){
				// i is idx in cache memory.
				//cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
				cache[cidx] += cache[cidx + i];
			}
			//////////////////////////////////////////////////////////////////////////
			if(i>32) __syncthreads();
			//////////////////////////////////////////////////////////////////////////
			i /= 2;
		}

		// Update local block sum from shared memory to global memory.
		// Size of global memory should be block number.
		if(cidx == 0)	out_ATb_partial[bidx*6 + pidx] = cache[0];
		__syncthreads();
	}
	//__syncthreads();

	// for b
// 	{
// 		cache[cidx] = tb[0];
// 		__syncthreads();
// 
// 		int i = dim_block/2;	// half of total thread number per block.
// 		while(i!=0){
// 			if(cidx < i){
// 				// i is idx in cache memory.
// 				//cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
// 				cache[cidx] += cache[cidx + i];
// 			}
// 			//////////////////////////////////////////////////////////////////////////
// 			if(i>32) __syncthreads();
// 			//////////////////////////////////////////////////////////////////////////
// 			i /= 2;
// 		}
// 
// 		// Update local block sum from shared memory to global memory.
// 		// Size of global memory should be block number.
// 		//if(cidx == 0)	out_b_partial[bidx] = cache[0];
// 	}
// 	__syncthreads();

		
}

__global__ void g_glsbw_Generate_Linear_System_using_Backward_Warping(
	float *out_ATA_partial,
	float *out_ATb_partial,
//	float *out_b_partial,
// 	int *out_num_val_partial,
// 	int *out_num_inlier_partial,
	int in_lev_of_pyram,
	const float *in_map_depth_t1,
	const float *in_map_depth_t0,
	const float *in_map_vertex_t0,
	const float *in_map_normal_t0,
	const float *in_T_21_est)
{
	// for computing overall summation.
	// 	__shared__ float cache_ATA[6*6*CV_CUDA_MAX_BLOCK_SIZE], cache_ATb[6*CV_CUDA_MAX_BLOCK_SIZE];
	// 	__shared__ int cache_cnt[CV_CUDA_MAX_BLOCK_SIZE];

	const int tx = threadIdx.x + blockIdx.x*blockDim.x;
	const int ty = threadIdx.y + blockIdx.y*blockDim.y;

	const int cidx = threadIdx.x + threadIdx.y*blockDim.x;
	const int bidx = blockIdx.x + blockIdx.y*gridDim.x;

	__shared__ float cache[CV_CUDA_MAX_BLOCK_SIZE];

	// for etc. parameters.	
	// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
	// initialize cache values.
	cache[cidx] = 0.0f;
	__syncthreads();

	// for constant variables.
	//////////////////////////////////////////////////////////////////////////
	const int ww = dim_map_dev[in_lev_of_pyram*2 + 0];
	const int hh = dim_map_dev[in_lev_of_pyram*2 + 1];
	//////////////////////////////////////////////////////////////////////////
	const float th_icp = th_icp_dev[0];

	const int dim_ATA = 6*6,dim_ATb = 6;
	const int dim_block = blockDim.x*blockDim.y;

	// for linear system.
	float tATA[6*6],tATb[6],tb[1]={0.0f};
	bool flag_valid = false,tinlier[1]={false};

	// initialize linear system.
	for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
	for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;

	if(tx < ww && ty < hh){
		flag_valid = d_cssfp_Compute_Single_Summand_with_Forward_Params(
			tx,ty,
			in_map_depth_t1,
			in_map_depth_t0,
			in_map_vertex_t0,
			in_map_normal_t0,
			in_T_21_est,
			in_lev_of_pyram,
			ww,hh,
			th_icp,
			tATA,
			tATb);
		//tb,
		//tinlier);

		if(!flag_valid){
			for(int pidx = 0; pidx<dim_ATA; pidx++) tATA[pidx] = 0.0f;
			for(int pidx = 0; pidx<dim_ATb; pidx++) tATb[pidx] = 0.0f;
			//tb[0] = 0.0f;
			//tinlier[0] = false;
		}
	}

	// set cache values.
	// for ATA
	for(int pidx = 0; pidx<dim_ATA; pidx++){
		cache[cidx] = tATA[pidx];
		__syncthreads();

		int i = dim_block/2;	// half of total thread number per block.
		while(i!=0){
			if(cidx < i){
				// i is idx in cache memory.
				//cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
				cache[cidx] += cache[cidx + i];
			}
			//////////////////////////////////////////////////////////////////////////
			if(i>32) __syncthreads();
			//////////////////////////////////////////////////////////////////////////
			i /= 2;
		}

		// Update local block sum from shared memory to global memory.
		// Size of global memory should be block number.
		if(cidx == 0)	out_ATA_partial[bidx*dim_ATA + pidx] = cache[0];
		__syncthreads();
	}

	// for ATb
	for(int pidx = 0; pidx<6; pidx++){

		cache[cidx] = tATb[pidx];
		__syncthreads();

		int i = dim_block/2;	// half of total thread number per block.
		while(i!=0){
			if(cidx < i){
				// i is idx in cache memory.
				//cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
				cache[cidx] += cache[cidx + i];
			}
			//////////////////////////////////////////////////////////////////////////
			if(i>32) __syncthreads();
			//////////////////////////////////////////////////////////////////////////
			i /= 2;
		}

		// Update local block sum from shared memory to global memory.
		// Size of global memory should be block number.
		if(cidx == 0)	out_ATb_partial[bidx*6 + pidx] = cache[0];
		__syncthreads();
	}
	//__syncthreads();

	// for b
	// 	{
	// 		cache[cidx] = tb[0];
	// 		__syncthreads();
	// 
	// 		int i = dim_block/2;	// half of total thread number per block.
	// 		while(i!=0){
	// 			if(cidx < i){
	// 				// i is idx in cache memory.
	// 				//cache[cidx] = __fadd_rn(cache[cidx],cache[cidx + i]);
	// 				cache[cidx] += cache[cidx + i];
	// 			}
	// 			//////////////////////////////////////////////////////////////////////////
	// 			if(i>32) __syncthreads();
	// 			//////////////////////////////////////////////////////////////////////////
	// 			i /= 2;
	// 		}
	// 
	// 		// Update local block sum from shared memory to global memory.
	// 		// Size of global memory should be block number.
	// 		//if(cidx == 0)	out_b_partial[bidx] = cache[0];
	// 	}
	// 	__syncthreads();


}

/////////////////////////////////////////////////////////////////////////////////////////////
// LGKvVolumeIntegrator
/////////////////////////////////////////////////////////////////////////////////////////////

// *******************************************************
__host__ LGKvPoseTracker::LGKvPoseTracker()
// *******************************************************
{
	for(int i=0; i<GK_LEVEL_OF_IMAGE_PYRAMID; i++){
		z_ATA_partial_dev[i].create(6, 6, 1); 
		z_ATb_partial_dev[i].create(6, 1, 1);
		z_b_partial_dev[i].create(6, 1, 1);
	}
	z_T_cg_est.create(4, 4, 1);
	z_T_cg_prev.create(4, 4, 1);

	z_mat_4x4.create(4, 4, 1);
	//z_T_cg_est1.create(4, 4, 1);

	z_ATA_partial_host = z_ATb_partial_host = z_b_partial_host = NULL;
	z_num_partial_host = z_num_inlier_partial_host = NULL;
}

// *******************************************************
__host__ LGKvPoseTracker::~LGKvPoseTracker()
// *******************************************************
{
	if(z_ATA_partial_host) delete[] z_ATA_partial_host;
	if(z_ATb_partial_host) delete[] z_ATb_partial_host;
	if(z_b_partial_host) delete[] z_b_partial_host;
	if(z_num_partial_host) delete[] z_num_partial_host;
	if(z_num_inlier_partial_host) delete[] z_num_inlier_partial_host;
}


// *******************************************************
__host__ void LGKvPoseTracker::ip_Initialize_Parameters(
	int ww, int hh,
	float fx, float fy,
	float px, float py,
	float th_icp)
// *******************************************************
{
// 	float intrins_host[4];
// 	float th_icp_host[1];
// 	int dim_map_host[2];
// 
// 	dim_map_host[0] = ww;	dim_map_host[1] = hh;
// 	intrins_host[0] = fx;	intrins_host[1] = fy;	intrins_host[2] = px;	intrins_host[3] = py;
// 	th_icp_host[0] = th_icp;		
// 
// 	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_dev), dim_map_host, 2 * sizeof(int));
// 	hipMemcpyToSymbol(HIP_SYMBOL(K_dev), intrins_host, 4 * sizeof(float));	
// 	hipMemcpyToSymbol(HIP_SYMBOL(th_icp_dev), th_icp_host, sizeof(float)); 

	//////////////////////////////////////////////////////////////////////////
	float intrins_host[GK_LEVEL_OF_IMAGE_PYRAMID*4];
	int dim_map_host[GK_LEVEL_OF_IMAGE_PYRAMID*2];
	float th_icp_host[1];

	//printf("dim_cube_host: %d %d %d\n", dim_cube_host[0], dim_cube_host[1], dim_cube_host[2]);
	int tww,thh;	tww = ww; thh = hh;
	float *p_intrins = &intrins_host[0];
	int *p_dim_maps = &dim_map_host[0];

	for(int k=0; k<GK_LEVEL_OF_IMAGE_PYRAMID; k++){

		p_intrins[0] = fx;	p_intrins[1] = fy;	p_intrins[2] = px;	p_intrins[3] = py;
		p_dim_maps[0] = tww;	p_dim_maps[1] = thh;

		// downsizing.
		fx = 0.5f*fx; fy = 0.5f*fy; px = 0.5f*(px - 0.5f); py = 0.5f*(py - 0.5f);
		tww /= 2; thh /= 2;

		p_intrins += 4;
		p_dim_maps += 2;

	}

	th_icp_host[0] = th_icp;

	hipMemcpyToSymbol(HIP_SYMBOL(K_dev),intrins_host,GK_LEVEL_OF_IMAGE_PYRAMID*4 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_dev),dim_map_host,GK_LEVEL_OF_IMAGE_PYRAMID*2 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(th_icp_dev),th_icp_host,sizeof(float));

	
}

// *******************************************************
// Frame-to-model depth tracking.
// Backward warping with forward parameterization.
__host__ bool LGKvPoseTracker::tp_Track_Pose(
	GKvTrackingState *io_track_state,
	GKvRgbdFrame *in_rgbd_frame,
	GKvMatrixFloat *in_pose_init_t0_t1)
// *******************************************************
{
	//static int iter_num[4] ={5,3,3,3};
	static int iter_num[4] ={10,7,7,7};
	static int th_num[4] ={500,300,100,100};
	static float T_12_est_host[16], T_21_est_host[16], T_gc_est_host[16];
	static float T_12_init_host[16], T_tmp[16];

	// current pose to estimate.
	float *p_T_21_est_dev = z_T_cg_est.vp();
	float *T_cg_est_dev;

	//float *ATA_partial_dev,*ATb_partial_dev,*b_partial_dev;
	//int *num_partial_dev,*num_inlier_partial_dev;

	Vector3f cent_prev,cent_est;
	Vector2i sz_lv0 = io_track_state->sz_map;
	float *p_T_gc_dev = io_track_state->vp_T_gc();
	float *p_T_cg_dev = io_track_state->vp_T_cg();

	bool flag_valid;

	//////////////////////////////////////////////////////////////////////////
	int lev_of_pyram = GK_LEVEL_OF_IMAGE_PYRAMID;
	//////////////////////////////////////////////////////////////////////////

	//hipMemcpy(T_12_est_host,p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToHost);
	//hipMemcpy(T_21_est_host,p_T_gc_dev,16*sizeof(float),hipMemcpyDeviceToHost);

	// set identity matrix.
	for(int i=0;i<16;i++)	T_12_est_host[i] = T_21_est_host[i] = (i%4 == i/4) ? 1.0f : 0.0f;		

	//d_pm_Printf_Matrix(T_12_est_host, 4, 4, "T_12");
	
	//////////////////////////////////////////////////////////////////////////
	// get initial pose.
	if(in_pose_init_t0_t1){
		hipMemcpy(T_12_init_host,in_pose_init_t0_t1->vp(),16*sizeof(float),hipMemcpyDeviceToHost);
		d_cv_Copy_Vector(T_12_init_host, 16, T_12_est_host);
		d_im_Inverse_Matrix_4x4(T_12_init_host, T_21_est_host);
		//hipMemcpy(z_mat_4x4.vp(),T_tmp,16*sizeof(float),hipMemcpyHostToDevice);

		//io_track_state->set_transform(&z_mat_4x4);

		lev_of_pyram = 1;
	}
	 hipMemcpy(p_T_21_est_dev,T_21_est_host,16*sizeof(float),hipMemcpyHostToDevice);
	//////////////////////////////////////////////////////////////////////////

	// Camera pose at t0.
//	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const),p_T_gc_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
//	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const),p_T_cg_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
	// Camera pose going to be estimated in current step.
	//hipMalloc((void**)&T_cg_est_dev, 16*sizeof(float));
// 	hipMemcpy(p_T_21_est_dev,p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToDevice);
// 	hipMemcpy(T_12_est_host,p_T_21_est_dev,16*sizeof(float),hipMemcpyDeviceToHost);
	// Previous camera center.
// 	hipMemcpy(z_T_cg_prev.vp(),p_T_cg_dev,16*sizeof(float),hipMemcpyDeviceToDevice);
// 	d_gcc_Get_Camera_Center(T_12_est_host,cent_prev);

	// Cuda kernel.
	dim3 threads_lv0(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks_lv0(iDivUp(sz_lv0.x,threads_lv0.x),iDivUp(sz_lv0.y,threads_lv0.y));
	int dim_block_lv0 = blocks_lv0.x*blocks_lv0.y;
	
//	printf("dim_block_lv0: %d\n", dim_block_lv0);

// 	if(z_ATA_partial_dev.nch() != dim_block_lv0)	z_ATA_partial_dev.create(6,6,dim_block_lv0);
// 	if(z_ATb_partial_dev.nch() != dim_block_lv0)	z_ATb_partial_dev.create(6,1,dim_block_lv0);
// 	if(z_b_partial_dev.nch() != dim_block_lv0)	z_b_partial_dev.create(1,1,dim_block_lv0);

	//hipMalloc((void**)&ATA_partial_dev,6*6*dim_block*sizeof(float));
	//hipMalloc((void**)&ATb_partial_dev,6*dim_block*sizeof(float));
	//hipMalloc((void**)&b_partial_dev,dim_block*sizeof(float));
// 	hipMalloc((void**)&num_partial_dev,dim_block_lv0*sizeof(int));
// 	hipMalloc((void**)&num_inlier_partial_dev,dim_block_lv0*sizeof(int));

	if(!z_ATA_partial_host) z_ATA_partial_host = new float[6*6*dim_block_lv0];
	if(!z_ATb_partial_host) z_ATb_partial_host = new float[6*dim_block_lv0];
// 	if(!z_b_partial_host) z_b_partial_host = new float[dim_block_lv0];
// 	if(!z_num_partial_host) z_num_partial_host = new int[dim_block_lv0];
// 	if(!z_num_inlier_partial_host) z_num_inlier_partial_host = new int[dim_block_lv0];

	flag_valid = true;

	for(int n=lev_of_pyram-1; n>=0; n--){

		const float *map_d_t1 = in_rgbd_frame->vp_map_depth(n);
		const float *map_d_t0 = io_track_state->vp_map_depth(n);
		const float *map_ver_t0 = io_track_state->vp_map_vertex(n);
		const float *map_norm_t0 = io_track_state->vp_map_normal(n);

		Vector2i sz = sz_lv0/pow(2,n);
		dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
		dim3 blocks(iDivUp(sz.x,threads.x),iDivUp(sz.y,threads.y));
		int dim_block = blocks.x*blocks.y;

		if(z_ATA_partial_dev[n].nch() != dim_block)	z_ATA_partial_dev[n].create(6,6,dim_block);
		if(z_ATb_partial_dev[n].nch() != dim_block)	z_ATb_partial_dev[n].create(6,1,dim_block);

		for(int i=0; i<iter_num[n]; i++){

			if(!flag_valid) break;

// 			g_gls_Generate_Linear_System<<<blocks,threads>>>(
// 				z_ATA_partial_dev[n].vp(),z_ATb_partial_dev[n].vp(),
// 				//z_b_partial_dev[n].vp(),
// 				//ATA_partial_dev,ATb_partial_dev,b_partial_dev,
// 				//num_partial_dev,num_inlier_partial_dev,
// 				n,
// 				map_d_t1,
// 				map_d_t0,map_ver_t0,map_norm_t0,
// 				p_T_cg_est_dev);

			g_glsbw_Generate_Linear_System_using_Backward_Warping<<<blocks,threads>>>(
			 	z_ATA_partial_dev[n].vp(),z_ATb_partial_dev[n].vp(),
			 	//z_b_partial_dev[n].vp(),
			 	//ATA_partial_dev,ATb_partial_dev,b_partial_dev,
			 	//num_partial_dev,num_inlier_partial_dev,
			 	n,
			 	map_d_t1,
			 	map_d_t0,map_ver_t0,map_norm_t0,
			 	p_T_21_est_dev);


			hipMemcpy(z_ATA_partial_host,z_ATA_partial_dev[n].vp(),6*6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATA_partial_dev.vp(), 
			hipMemcpy(z_ATb_partial_host,z_ATb_partial_dev[n].vp(),6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
			//hipMemcpy(z_b_partial_host,z_b_partial_dev[n].vp(),dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
			
			//hipMemcpy(z_ATA_partial_host,ATA_partial_dev,6*6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATA_partial_dev.vp(), 
			//hipMemcpy(z_ATb_partial_host,ATb_partial_dev,6*dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
			//hipMemcpy(z_b_partial_host,b_partial_dev,dim_block*sizeof(float),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
			
			//hipMemcpy(z_num_partial_host,num_partial_dev,dim_block*sizeof(int),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),
			//hipMemcpy(z_num_inlier_partial_host,num_inlier_partial_dev,dim_block*sizeof(int),hipMemcpyDeviceToHost); //z_ATb_partial_dev.vp(),

			// Add partial sums.
			for(int k=0; k<6*6; k++) z_ATA[k] = 0.0f;
			for(int k=0; k<6; k++) z_ATb[k] = 0.0f;
// 			z_b = 0.0f;
// 			z_num_val = 0;
// 			z_num_inlier = 0;

			for(int k = 0; k<dim_block; k++){
				for(int j = 0; j<6*6; j++) z_ATA[j] += z_ATA_partial_host[k*6*6 + j];
				for(int j = 0; j<6; j++) z_ATb[j] += z_ATb_partial_host[k*6 + j];
// 				z_b += z_b_partial_host[k];     // squared point-to-point distance
// 				z_num_val += z_num_partial_host[k];
// 				z_num_inlier += z_num_inlier_partial_host[k];
			}

			d_pm_Printf_Matrix(z_ATA, 6, 6);

// 			printf("(#%d) z_ATb: %f %f %f | %f %f %f\n",
// 			z_num_val,z_ATb[0],z_ATb[1],z_ATb[2]
// 			,z_ATb[3],z_ATb[4],z_ATb[5]);

			// Check number of valid pixels.		
// 			if(z_num_val < th_num[n]){
// 				flag_valid = false;
// 				break;
// 			}

			// solve linear system.
			float norm2_x;
			d_lld_LL_Decomposition(z_ATA,6,z_L);

// 			d_pm_Printf_Matrix(z_ATA,6,6,"ATA");
// 			d_pm_Printf_Matrix(z_L,6,6,"L");

			//if(!d_sls_Solve_Linear_System_using_LLD(z_L,z_ATb,z_y,6,z_sol_x)) { flag_valid = false; break; }
			flag_valid = d_sls_Solve_Linear_System_using_LLD(z_L,z_ATb,z_y,6,z_sol_x);
			if(!flag_valid) break;

			// Check 2-norm of solution vector x.
			// 		d_n2v_Norm_2_Vector(z_sol_x,6,norm2_x);
			// 		if(norm2_x < 6.0e-3) break;
			float norm_inf;	d_niv_Norm_Infinity_Vector(z_sol_x,6,norm_inf);
			if(norm_inf < 5.0e-7f) break;

			// Update incremental transformation. (T_cg_est_dev)
			//z_uit_Update_Incremental_Tracking(z_sol_x,T_cg_est_host);
			z_uitme_Update_Incremental_Tracking_with_Matrix_Exponential(z_sol_x,T_12_est_host);
			d_im_Inverse_Matrix_4x4(T_12_est_host, T_21_est_host);
			hipMemcpy(p_T_21_est_dev,T_21_est_host,16*sizeof(float),hipMemcpyHostToDevice);

			// check motion validity.
			//	double translation = norm(Rt(Rect(3,0,1,3)));
			//	double rotation = norm(rvec) * 180. / CV_PI;

			// 		for(int j = 0; j<6; j++) printf("%f ", z_sol_x[j]);
			// 		printf("\n");
			//for(int k=0; k<6*6; k++)
		}
	}

	

	if(flag_valid){


		hipMemcpy(T_tmp,p_T_gc_dev,16*sizeof(float),hipMemcpyDeviceToHost);
		d_mmm_Multiply_Matrix_Matrix(T_12_est_host, T_tmp, 4, 4, 4, T_gc_est_host);
		hipMemcpy(z_mat_4x4.vp(),T_gc_est_host,16*sizeof(float),hipMemcpyHostToDevice);
		io_track_state->set_transform(&z_mat_4x4,false);

		///////////////////////////////////////////////////////////////////////////////////
		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
		// ���� rmse ���� Euclidean distance �� ���µ� normal vector ���̵� ������ �غ���....
		// detect motion drift.
		// Estimated current camera center.
		float dist,rmse_surf,inlier_perc;
		//cent_est = io_track_state->center;
		//dist = length(cent_est - cent_prev);

		rmse_surf = sqrt(float(z_b)/(float)z_num_val);
		inlier_perc = 100.0f*float(z_num_inlier)/float(z_num_val);
		//printf(" >>> rmse_surf: %f mm (%% %5.2f)\n",rmse_surf,inlier_perc);

		//if(dist > 0.175f){  // 0.5m*20(deg)*PI/180.0f.
		// 		if(rmse_surf > 0.05f || inlier_perc < 95.0f){  // 0.5m*20(deg)*PI/180.0f.
		// 			io_track_state->set_transform(&z_T_cg_prev,true);
		// 			printf("=================================== Drift! ==========================================");
		// 			flag_valid = false;
		// 		}
		///////////////////////////////////////////////////////////////////////////////////

		// Transform from camera to global coordinates.
		//  		hipMemcpy(p_T_cg_dev, p_T_cg_est_dev, 16*sizeof(float), hipMemcpyDeviceToDevice);
		//  		// Transform from global to camera coordinates.
		//  		d_im_Inverse_Matrix_4x4(T_cg_est_host, T_gc_est_host);
		//  		hipMemcpy(p_T_gc_dev, T_gc_est_host, 16*sizeof(float), hipMemcpyHostToDevice);
		//  		// Update camera center.
		//  		d_gcc_Get_Camera_Center(T_gc_est_host, io_track_state->center);
	}


	//hipFree(ATA_partial_dev);
	//hipFree(ATb_partial_dev);
	//hipFree(b_partial_dev);

// 	hipFree(num_partial_dev);
// 	hipFree(num_inlier_partial_dev);

	//hipFree(T_cg_est_dev);

	return flag_valid;
}

////********************************************************************************************
//__host__ bool cmv_Check_Motion_Validity(
//	CKvMatrixFloat *io_hmat_4x4,
//	float in_max_translation,	// m
//	float in_max_rotation)		// rad.
////********************************************************************************************
//{
//	Mat Rt, rvec;
//	aa_ilib.cfko_Convert_Format_from_KAISION_to_Opencv(*io_hmat_4x4, Rt);
//
//	Rodrigues(Rt(Rect(0, 0, 3, 3)), rvec);
//
//	double translation = norm(Rt(Rect(3,0,1,3)));
//	double rotation = norm(rvec) * 180. / CV_PI;
//
//	//printf("trans: %f rot: %f\n", translation, rotation);
//
//	return translation <= in_max_translation && rotation <= in_max_rotation;
//}