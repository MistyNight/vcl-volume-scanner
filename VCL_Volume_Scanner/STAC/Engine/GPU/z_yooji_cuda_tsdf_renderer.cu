#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////////////////////////////
// z_yooji_dense_stereo.cpp
/////////////////////////////////////////////////////////////////////////////////////////////
// #include "_yooji_2017_cuda_object_scanner.cuh"
// #define __CUDASCAN__
#include "../../_yooji_2017_cuda_object_scanner.cuh"

__constant__ float K_dev[20];		// Maximum pyramid level is 5.
__constant__ int dim_map_dev[10];	// Maximum pyramid level is 5.

__constant__ float T_gc_dev_const[16];
__constant__ float T_cg_dev_const[16];

__constant__ float origin_dev[3];
__constant__ int dim_cube_dev[3];
__constant__ int dim_sc_dev[1];

__constant__ float mu_dev[1];
__constant__ float r_cube_dev[1];
__constant__ float sz_vox_inv_dev[1];
__constant__ float max_w_dev[1];

///////////////////////////////////////////////////////////////////////////////
/// Device functions.
///////////////////////////////////////////////////////////////////////////////
//
////********************************************************************************************
//__device__ inline bool d_gpv_Get_Position_in_Voxel3(Vector3f in_p3d, 
//	const float *origin, const int *dim_cube, const float sz_vox_inv,
//	Vector3f &out_vox)
////********************************************************************************************
//{
//	bool valid = true;
//
//	// assume that position of origin point in the world is (-0.5, -0.5, -0.5) in voxel coordinates..
//	out_vox.x = __fmaf_rn((in_p3d.x - origin[0]),sz_vox_inv, -0.5f);
//	out_vox.y = __fmaf_rn((in_p3d.y - origin[1]),sz_vox_inv, -0.5f);
//	out_vox.z = __fmaf_rn((in_p3d.z - origin[2]),sz_vox_inv, -0.5f);
//
//	if(out_vox.x < -0.5f || out_vox.x >= (float)dim_cube[0] - 0.5f ||
//		out_vox.y < -0.5f || out_vox.y >= (float)dim_cube[1] - 0.5f ||
//		out_vox.z < -0.5f || out_vox.z >= (float)dim_cube[2] - 0.5f)
//		valid = false;
//
//	return valid;
//}
////********************************************************************************************
//__device__ inline bool d_gpw_Get_Position_in_World3(Vector3f in_vox, 
//	const float *origin, const int *dim_cube, const float sz_vox,
//	Vector3f &out_p3d)
////********************************************************************************************
//{
//	bool valid = true;
//
//	if(in_vox.x < -0.5f || in_vox.x >= (float)dim_cube[0] - 0.5f ||
//		in_vox.y < -0.5f || in_vox.y >= (float)dim_cube[1] - 0.5f ||
//		in_vox.z < -0.5f || in_vox.z >= (float)dim_cube[2] - 0.5f)
//		valid = false;
//
//	out_p3d.x = __fmaf_rn((in_vox.x + 0.5f),sz_vox, origin[0]);
//	out_p3d.y = __fmaf_rn((in_vox.y + 0.5f),sz_vox, origin[1]);
//	out_p3d.z = __fmaf_rn((in_vox.z + 0.5f),sz_vox, origin[2]);
//
//	return valid;
//}

//********************************************************************************************
__device__ bool d_gtvu_Get_TSDF_Value_Uninterpolated(
	Vector3f in_vox, 
	const float *vol_tsdf,
	const uchar *vol_w,
	float &out_tsdf)
//********************************************************************************************
{
	int ww, hh, dd;
	int x, y, z, tidx;

	out_tsdf = 1.0f;	// set default value.

	// calculate local indices.		
	x = ROUNDF(in_vox.x);	y = ROUNDF(in_vox.y);	z = ROUNDF(in_vox.z);
	//x = int(in_vox.x);	y = int(in_vox.y);	z = int(in_vox.z);

	if(x<0 || x>=dim_cube_dev[0] || 
	   y<0 || y>=dim_cube_dev[1] || 
	   z<0 || z>=dim_cube_dev[2]) 
	   return false; 
	
 	tidx = z*dim_cube_dev[0]*dim_cube_dev[1] + y*dim_cube_dev[0] + x;
	 
 	if(vol_w[tidx] <= uchar(0)) return false;
 
 	out_tsdf = vol_tsdf[tidx];

	return true;
	
}

//********************************************************************************************
__device__ bool d_gtvi_Get_TSDF_Value_Interpolated(	
	Vector3f in_vox,
	const float *vol_tsdf,
	const uchar *vol_w,
	float &out_tsdf)
//********************************************************************************************
{
	Vector3i offset;
	Vector3f residu;
	Vector3f tpos;

	float inter_f, inter_b;
	float front[4], back[4];
	float dx1[4], dy1[4];
	int i;

	//     5-----6
	//  1=====2
	//     7-----8
	//  3=====4
	/// Get TSDF values of 8 neighbor voxels.
	out_tsdf = 1.0f;
	// get offset voxel index and float residual vector of input 3d point.
	offset = in_vox.toInt(residu);

	// set delta x, y for slice access.
	dx1[0] = 0.0f; dx1[1] = 1.0f; dx1[2] = 0.0f; dx1[3] = 1.0f;
	dy1[0] = 0.0f; dy1[1] = 0.0f; dy1[2] = 1.0f; dy1[3] = 1.0f;

	// get TSDF values of neighbors on the front plane.
	tpos.z = offset.z;	// set Z value.
	for(i = 0; i<4; i++){
		tpos.x = offset.x + dx1[i]; tpos.y = offset.y + dy1[i];
		if(!d_gtvu_Get_TSDF_Value_Uninterpolated(tpos, vol_tsdf, vol_w, front[i])) return false;
	}

	//// get TSDF values of neighbors on the back plane.
	tpos.z = offset.z + 1.0f;	// set Z value.
	for(i = 0; i<4; i++){
		tpos.x = offset.x + dx1[i]; tpos.y = offset.y + dy1[i];
		if(!d_gtvu_Get_TSDF_Value_Uninterpolated(tpos, vol_tsdf, vol_w, back[i])) return false;
	}

	
	// interpolates TSDF value on XY plane.
	inter_f = (1.0f - residu.y)*((1.0f - residu.x)*front[0] + residu.x*front[1])
		+ residu.y*((1.0f - residu.x)*front[2] + residu.x*front[3]);
	inter_b = (1.0f - residu.y)*((1.0f - residu.x)*back[0] + residu.x*back[1])
		+ residu.y*((1.0f - residu.x)*back[2] + residu.x*back[3]);
 
 	out_tsdf = (1.0f - residu.z)*inter_f + residu.z*inter_b;

	return true;
}


//********************************************************************************************
__device__ bool d_grvu_Get_RGB_Value_Uninterpolated(
	Vector3f in_vox,
	const uchar *vol_color,
	const uchar *vol_w,
	Vector3u &out_color)
//********************************************************************************************
{
	int ww,hh,dd;
	int x,y,z,tidx;

	out_color = uchar(0);	// set default value.

	ww = dim_cube_dev[0]; hh = dim_cube_dev[1]; dd = dim_cube_dev[2];

	// calculate local indices.		
	x = ROUNDF(in_vox.x);	y = ROUNDF(in_vox.y);	z = ROUNDF(in_vox.z);
	//x = int(in_vox.x);	y = int(in_vox.y);	z = int(in_vox.z);

	if(x<0 || x>=ww ||
	   y<0 || y>=hh ||
	   z<0 || z>=dd)
	   return false;

	tidx = z*ww*hh + y*ww + x;

	if(vol_w[tidx] <= uchar(0)) return false;

	//////////////////////////////////////////////////////////////////////////
	out_color.x = vol_color[tidx+ww*hh*dd];
	//////////////////////////////////////////////////////////////////////////
	out_color.y = vol_color[tidx+ww*hh*dd];
	out_color.z = vol_color[tidx+2*ww*hh*dd];

	return true;
}

//********************************************************************************************
__device__ bool d_grvu_Get_RGB_Value_Uninterpolated(
	Vector3f in_vox,
	const uchar *vol_color,
	const uchar *vol_w,
	Vector3f &out_color)
//********************************************************************************************
{
	int ww,hh,dd;
	int x,y,z,tidx;

	out_color = 0.0f;	// set default value.

	ww = dim_cube_dev[0]; hh = dim_cube_dev[1]; dd = dim_cube_dev[2];

	// calculate local indices.		
	x = ROUNDF(in_vox.x);	y = ROUNDF(in_vox.y);	z = ROUNDF(in_vox.z);
	//x = int(in_vox.x);	y = int(in_vox.y);	z = int(in_vox.z);

	if(x<0 || x>=ww ||
	   y<0 || y>=hh ||
	   z<0 || z>=dd)
	   return false;

	tidx = z*ww*hh + y*ww + x;

	//////////////////////////////////////////////////////////////////////////
	// �̺κ�?
	if(vol_w[tidx] <= uchar(0)) return false;
	//////////////////////////////////////////////////////////////////////////

	out_color.x = float(vol_color[tidx]);		// __int2float_rn
	out_color.y = float(vol_color[tidx+ww*hh*dd]);
	out_color.z = float(vol_color[tidx+2*ww*hh*dd]);

	return true;
}

//********************************************************************************************
__device__ bool d_grvi_Get_RGB_Value_Interpolated(
	Vector3f in_vox,
	const uchar *vol_color,
	const uchar *vol_w,
	Vector3u &out_color)
//********************************************************************************************
{
	Vector3i offset;
	Vector3f residu;
	Vector3f tpos;

	Vector3f inter_f,inter_b;
	Vector3f front[4],back[4];
	float dx1[4],dy1[4];
	int i;

	//     5-----6
	//  1=====2
	//     7-----8
	//  3=====4
	/// Get RGB values of 8 neighbor voxels.
	out_color = uchar(0);
	// get offset voxel index and float residual vector of input 3d point.
	offset = in_vox.toInt(residu);

	// set delta x, y for slice access.
	dx1[0] = 0.0f; dx1[1] = 1.0f; dx1[2] = 0.0f; dx1[3] = 1.0f;
	dy1[0] = 0.0f; dy1[1] = 0.0f; dy1[2] = 1.0f; dy1[3] = 1.0f;

	// get RGB values of neighbors on the front plane.
	tpos.z = offset.z;	// set Z value.
	for(i = 0; i<4; i++){
		tpos.x = offset.x + dx1[i]; tpos.y = offset.y + dy1[i];
		if(!d_grvu_Get_RGB_Value_Uninterpolated(tpos,vol_color,vol_w,front[i])) return false;
	}

	//// get RGB values of neighbors on the back plane.
	tpos.z = offset.z + 1.0f;	// set Z value.
	for(i = 0; i<4; i++){
		tpos.x = offset.x + dx1[i]; tpos.y = offset.y + dy1[i];
		if(!d_grvu_Get_RGB_Value_Uninterpolated(tpos,vol_color,vol_w,back[i])) return false;
	}

	// interpolates RGB value on XY plane.
	inter_f = (1.0f - residu.y)*((1.0f - residu.x)*front[0] + residu.x*front[1])
		+ residu.y*((1.0f - residu.x)*front[2] + residu.x*front[3]);
	inter_b = (1.0f - residu.y)*((1.0f - residu.x)*back[0] + residu.x*back[1])
		+ residu.y*((1.0f - residu.x)*back[2] + residu.x*back[3]);
	
	out_color = ((1.0f - residu.z)*inter_f + residu.z*inter_b).toUChar();
	
	return true;
}


//********************************************************************************************
__device__ bool d_csnt_Compute_Surface_Normal_from_TSDF(
	Vector3f in_p3d, 
	const float *vol_tsdf,
	const uchar *vol_w,
	Vector3f &out_surf_norm)
//********************************************************************************************
{
	// we need total 32 neighbors for computing a single surface normal of input 3D point. (refer following pictures.)
	// the offset voxel index is 4 in front XY plane.
	// + the foremost XY slice (Z=-1)
	//    -- X      
	// Y |
	//       1     2      
	//          X
	//       3     4     
	//
	//            
	//float XY_foremost[4];
	// + front XY slice (Z=0)
	//       1     2
	//
	// 3     4     5     6
	//          X
	// 7     8     9     10
	//
	//       11    12
	//float XY_front[12];
	// + back XY slice (Z=1)
	//       1     2
	//
	// 3     4     5     6
	//          X
	// 7     8     9     10
	//
	//       11    12
	//float XY_back[12];
	// + the backmost XY slice (Z=2)
	//       
	//
	//       1     2      
	//          X
	//       3     4     
	//
	//            
	//float XY_backmost[4];

	/// Get TSDF values of 32 neighbor voxels.
 	Vector3i offset;
 	Vector3f residu;
 	Vector3f tpos, vox, norm;
 	
 	float sz_voxel, gx, gy, gz;
 	float foremost[4], front[12], back[12], backmost[4];
 	float dx1[4], dy1[4], dx2[12], dy2[12];

	// coverts input 3D point to position of voxel coordinates.
 	if(!d_gpv_Get_Position_in_Voxel(in_p3d, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], vox)) return false;
 
 	// get offset voxel index and float residual vector of input 3d point.
 	offset = vox.toInt(residu);
 
 	// set delta x, y for slice access.
 	dx1[0] = 0.0f; dx1[1] = 1.0f; dx1[2] = 0.0f; dx1[3] = 1.0f;
 	dy1[0] = 0.0f; dy1[1] = 0.0f; dy1[2] = 1.0f; dy1[3] = 1.0f;
 
 	dx2[0] = 0.0f;   dx2[1] = 1.0f;
 	dx2[2] = -1.0f;  dx2[3] = 0.0f; dx2[4] = 1.0f; dx2[5] = 2.0f;
 	dx2[6] = -1.0f;  dx2[7] = 0.0f; dx2[8] = 1.0f; dx2[9] = 2.0f;
 	dx2[10] = 0.0f;  dx2[11] = 1.0f;
 
 	dy2[0] = -1.0f;  dy2[1] = -1.0f;
 	dy2[2] = 0.0f;  dy2[3] = 0.0f;  dy2[4] = 0.0f; dy2[5] = 0.0f;
 	dy2[6] = 1.0f;  dy2[7] = 1.0f;  dy2[8] = 1.0f; dy2[9] = 1.0f;
 	dy2[10] = 2.0f; dy2[11] = 2.0f;

	// get TSDF values of neighbors on the foremost XY plane. (Z = -1)
	tpos.z = offset.z - 1.0f;
	for(int i=0; i<4; i++){
		tpos.x = offset.x + dx1[i]; tpos.y = offset.y + dy1[i];
		if(!d_gtvu_Get_TSDF_Value_Uninterpolated(tpos, vol_tsdf, vol_w, foremost[i])) return false;
	}
	// get TSDF values of neighbors on the front XY plane. (Z = 0)
	tpos.z = offset.z;
	for(int i = 0; i<12; i++){
		tpos.x = offset.x + dx2[i]; tpos.y = offset.y + dy2[i];
		if(!d_gtvu_Get_TSDF_Value_Uninterpolated(tpos, vol_tsdf, vol_w, front[i])) return false;
	}
	// get TSDF values of neighbors on the back XY plane. (Z = 1)
	tpos.z = offset.z + 1.0f;
	for(int i = 0; i<12; i++){
		tpos.x = offset.x + dx2[i]; tpos.y = offset.y + dy2[i];
		if(!d_gtvu_Get_TSDF_Value_Uninterpolated(tpos, vol_tsdf, vol_w, back[i])) return false;
	}
	// get TSDF values of neighbors on the foremost XY plane. (Z = 2)
	tpos.z = offset.z + 2.0f;
	for(int i = 0; i<4; i++){
		tpos.x = offset.x + dx1[i]; tpos.y = offset.y + dy1[i];
		if(!d_gtvu_Get_TSDF_Value_Uninterpolated(tpos, vol_tsdf, vol_w, backmost[i])) return false;
	}

	
	/// Calculate surface normal.
	float inter2a[2], inter4a[4], inter2b[2], inter4b[4];
	// compute x-direction SDF gradient at point.
	// + get z-direction interpolated SDF values of (3 4 5 6) and (7 8 9 10).
	inter4a[0]=(1.0f-residu.z)*front[2] +residu.z*back[2];	
	inter4a[1]=(1.0f-residu.z)*front[3] +residu.z*back[3];
	inter4a[2]=(1.0f-residu.z)*front[4] +residu.z*back[4];	
	inter4a[3]=(1.0f-residu.z)*front[5] +residu.z*back[5];

	inter4b[0]=(1.0f-residu.z)*front[6] +residu.z*back[6];	
	inter4b[1]=(1.0f-residu.z)*front[7] +residu.z*back[7];
	inter4b[2]=(1.0f-residu.z)*front[8] +residu.z*back[8];	
	inter4b[3]=(1.0f-residu.z)*front[9] +residu.z*back[9];	
	
	// + dF(x, y, z)/dx = F(x+1, y, z)-F(x-1, y, z).
	inter2a[0]=(1.0f-residu.x)*inter4a[0] +residu.x*inter4a[1];	// x-direction interpolation of 3 and 4.
	inter2a[1]=(1.0f-residu.x)*inter4b[0] +residu.x*inter4b[1];	// x-direction interpolation of 7 and 8.
	inter2b[0]=(1.0f-residu.x)*inter4a[2] +residu.x*inter4a[3];	// x-direction interpolation of 5 and 6.
	inter2b[1]=(1.0f-residu.x)*inter4b[2] +residu.x*inter4b[3];	// x-direction interpolation of 9 and 10.

	gx=(1.0f-residu.y)*inter2b[0]+residu.y*inter2b[1];		// y-direction interpolation of (5 6) and (9 10).
	gx-=(1.0f-residu.y)*inter2a[0]+residu.y*inter2a[1];		// y-direction interpolation of (3 4) and (5 6).


	// compute y-direction SDF gradient at point.
	// + get z-direction interpolated SDF values of (1 4 8 11) and (2 5 9 12).
	inter4a[0]=(1.0f-residu.z)*front[0]	 +residu.z*back[0];	
	inter4a[1]=(1.0f-residu.z)*front[3]	 +residu.z*back[3];
	inter4a[2]=(1.0f-residu.z)*front[7]	 +residu.z*back[7];	
	inter4a[3]=(1.0f-residu.z)*front[10] +residu.z*back[10];

	inter4b[0]=(1.0f-residu.z)*front[1]	 +residu.z*back[1];	
	inter4b[1]=(1.0f-residu.z)*front[4]	 +residu.z*back[4];
	inter4b[2]=(1.0f-residu.z)*front[8]	 +residu.z*back[8];	
	inter4b[3]=(1.0f-residu.z)*front[11] +residu.z*back[11];

	// + dF(x, y, z)/dx = F(x, y+1, z)-F(x, y-1, z).
	inter2a[0]=(1.0f-residu.y)*inter4a[0] +residu.y*inter4a[1];	// y-direction interpolation of 1 and 4.
	inter2a[1]=(1.0f-residu.y)*inter4b[0] +residu.y*inter4b[1];	// y-direction interpolation of 2 and 5.
	inter2b[0]=(1.0f-residu.y)*inter4a[2] +residu.y*inter4a[3];	// y-direction interpolation of 8 and 11.
	inter2b[1]=(1.0f-residu.y)*inter4b[2] +residu.y*inter4b[3];	// y-direction interpolation of 9 and 12.


	gy=(1.0f-residu.x)*inter2b[0]	+residu.x*inter2b[1];			// x-direction interpolation of (8 11) and (9 12).
	gy-=(1.0f-residu.x)*inter2a[0]	+residu.x*inter2a[1];			// x-direction interpolation of (1 4) and (2 5).

	// compute z-direction SDF gradient at point.
	// + get y-direction interpolated SDF values of (X=0|Z=-1 0 1 2) and (X=1|Z=-1 0 1 2).
	inter4a[0]=(1.0f-residu.y)*foremost[0]	+residu.y*foremost[2];		
	inter4a[1]=(1.0f-residu.y)*front[3]		+residu.y*front[7];
	inter4a[2]=(1.0f-residu.y)*back[3]		+residu.y*back[7];				
	inter4a[3]=(1.0f-residu.y)*backmost[0]	+residu.y*backmost[2];

	inter4b[0]=(1.0f-residu.y)*foremost[1]	+residu.y*foremost[3];		
	inter4b[1]=(1.0f-residu.y)*front[4]		+residu.y*front[8];
	inter4b[2]=(1.0f-residu.y)*back[4]		+residu.y*back[8];				
	inter4b[3]=(1.0f-residu.y)*backmost[1]	+residu.y*backmost[3];
	
	// + dF(x, y, z)/dx = F(x, y, z+1)-F(x, y, z-1).
	inter2a[0]=(1.0f-residu.z)*inter4a[0] +residu.z*inter4a[1];	// z-direction interpolation of (X=0|Z=-1) and (X=0|Z=0).
	inter2a[1]=(1.0f-residu.z)*inter4b[0] +residu.z*inter4b[1];	// z-direction interpolation of (X=1|Z=-1) and (X=1|Z=0).
	inter2b[0]=(1.0f-residu.z)*inter4a[2] +residu.z*inter4a[3];	// z-direction interpolation of (X=0|Z=1) and (X=0|Z=2).
	inter2b[1]=(1.0f-residu.z)*inter4b[2] +residu.z*inter4b[3];	// z-direction interpolation of (X=1|Z=1) and (X=1|Z=2).

	gz=(1.0f-residu.x)*inter2b[0]  +residu.x*inter2b[1];			// x-direction interpolation of (X=0|Z= 1 2) and (X=1|Z= 1 2).
	gz-=(1.0f-residu.x)*inter2a[0] +residu.x*inter2a[1];			// x-direction interpolation of (X=0|Z=-1 0) and (X=1|Z=-1 0).

	if(gx==0.0f && gy==0.0f && gz==0.0f) return false;

	// save calculation result of surface normal.
	norm.x = gx; norm.y = gy; norm.z = gz;
	out_surf_norm = norm.normalised();

	return true;
	
}

__device__ bool d_crtc_Cast_Ray_on_TSDF_Cube_NEW(
	// variable parameters.
	Vector2f p2d,
	const float *T_gc, const float *T_cg,
	Vector3f cam_cen, Vector3f light, Vector3f norm_cc, float dist_cg_cen, float theta_max,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_w,
	int ww, int hh,
	int lev_of_pyram,
	float r_cube, float mu, float sz_vox_inv, int dim_sc,
	// for output cross point.
	Vector3f &p3d)
{

	// RENDERING NOW!!!!!
	Vector3f p3d_c,p3d_s,p3d_e,vox_s,vox_e,rd_g,rd_vox;
	float cos_val,theta,mag,dist_min,dist_max;

	bool flag_valid = false;
	bool flag_result = false;
	int cnt = 0;
	float step_sz,step_sz_coarse,step_scale,sz_sub_cube,total_step,total_step_max,tsdf;

	enum { SEARCH_COARSE,SEARCH_FINE,BEHIND_SURFACE,SEARCH_FINE_BACK } state;

	dist_min = dist_cg_cen - r_cube;
	dist_max = dist_cg_cen + r_cube;

	// compute the end point of the pixel ray for the input pixel.
	d_bp_Back_Project(p2d,&K_dev[lev_of_pyram*4],dist_max,p3d_c);
	d_t_Transform(p3d_c,T_cg,p3d_e);

	// ==================================================================
	// Ray validity test
	// ==================================================================
	// compute the direction of the pixel ray for the input pixel using camera center and far plane of view frustum.
	rd_g = (p3d_e - cam_cen).normalised();

	// compute angle between the pixel ray and direction vector between the two centers.
	// check whether the ray crossed the object cube.
	//cos_val = rd.x*norm_cc.x + rd.y*norm_cc.y + rd.z*norm_cc.z;
	theta = acosf(dot(rd_g,norm_cc));

	if(theta > theta_max){ return false; }

	// ==================================================================
	// Ray casting in voxel coordinates.
	// ==================================================================
	// calculates start point and ray direction for ray-casting in voxel coordinates.	
	p3d_s = cam_cen + dist_min*rd_g;
	// converts points and parameter coordinates from the real global coordinates to the cube voxel coordinates. (mm->voxel)
	// CAUTION: the offset position of global voxel cube is (-0.5f, -0.5f, -0.5f) in the cube voxel coordinates.
	// compute scale factor for converting TSDF value to 'mm' unit.	
	step_scale = 0.9f*mu*sz_vox_inv;
	// ======================================================
	// ���� coarse search �� step_scale �� �����ؾ� ��.
	// ���� ray �� invalid �� ���� ������ ���� ��� (tsdf ���� ����) 
	// ���� coarse search ���� �� �ܰ� ���� coarse search (step size = dim_sc) �� �����ص� �� ��.
	step_sz = step_sz_coarse = step_scale; //fmaxf(step_scale+0.1f, float(dim_sc));
	// ======================================================
	// set maximum search length of the pixel ray to 2*in_radius_cube.
	total_step_max = 2.0f*r_cube*sz_vox_inv;		//	 total_step_max=pe_g.d_Distance(ps_g)*in_voxel_sz_inv;

	// starts casting ray.
	total_step = 0.0f;
	// Validity check ���ص� �ǳ�??
	d_gpv_Get_Position_in_Voxel(p3d_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],vox_s);
	// Validity check ���ص� �ǳ�??
	flag_valid = d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s,cube_tsdf,cube_w,tsdf);
	if(!flag_valid)		state = SEARCH_COARSE;
	else if(tsdf>=0.0f)	state = SEARCH_FINE;
	else				return false; //state = SEARCH_FINE_BACK;
	

	// ===================================================
	// Casts ray.
	// ===================================================
	while(state!=BEHIND_SURFACE){

		// ===================================================
		// Sets step size.
		// ===================================================
		if(state == SEARCH_COARSE)  step_sz = step_sz_coarse;
		else step_sz = (tsdf>0.0f) ? fmaxf(step_scale*tsdf,1.0f) : fminf(step_scale*tsdf,-1.0f);
//		else if(state == SEARCH_FINE) step_sz = fmaxf(step_scale*tsdf,0.1f);	// in front of surface.  
//		else if(state == SEARCH_FINE_BACK)  step_sz = fminf(step_scale*tsdf,-0.1f);  // behind surface.

		// ===================================================
		// Proceeds ray casting.
		// ===================================================
		vox_s += step_sz*rd_g;	//vox_s.x += step_sz*rd.x; vox_s.y += step_sz*rd.y; vox_s.z += step_sz*rd.z;
		total_step += step_sz;
		// if total step length exceeds maximum step size, break while loop.
		if(total_step > total_step_max)	break;

		// compute uninterpolated TSDF value.
		flag_valid = d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s,cube_tsdf,cube_w,tsdf);

		// check which current points is in mu band or not. (near the surface)
		if(tsdf>-1.0f && tsdf<1.0f){
			// ===========================================================
			// Mu-band processing.
			// ===========================================================
			// compute trilinear interpolated TSDF value.
			flag_valid = d_gtvi_Get_TSDF_Value_Interpolated(vox_s,cube_tsdf,cube_w,tsdf);

			// ===================================================
			// Checks casting mode.
			// ===================================================
			// check current TSDF value is (-) at fine search step.
			if(tsdf<0.0f){
				// terminate ray casting if current TSDF value is (-) at coarse searching mode.
				if(state == SEARCH_COARSE) break;
				else state = BEHIND_SURFACE;
			}
			else if(tsdf == 0.0f){
				// on surface.
				d_gpw_Get_Position_in_World(vox_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d);
				flag_result=true;
				break;
			} 
			else if(state == SEARCH_COARSE)	state = SEARCH_FINE;
		} 
		else{
			if(state == SEARCH_FINE) state == SEARCH_COARSE;
		}

		//// ===================================================
		//// Checks casting mode.
		//// ===================================================
		//if(state == SEARCH_COARSE){
		//	if(tsdf<=0.0f) state = SEARCH_FINE_BACK; // go back.
		//	else           state = SEARCH_FINE;		 // start fine search.
		//} else if(state == SEARCH_FINE){
		//	// ================================================================
		//	// Infinite Loop Cause 1
		//	// : In case of noisy TSDF data processing.
		//	// Solution: Remove noisy data processing part.
		//	// if(tsdf >= 1.0f || tsdf <= -1.0f) state = SEARCH_COARSE;  // convert to coarse search. (ray met noise voxel data)
		//	// ================================================================
		//	if(tsdf<=0.0f)  state = BEHIND_SURFACE; // break while loop, and find intersection.
		//} else if(state == SEARCH_FINE_BACK){
		//	if(tsdf>0.0f) state = SEARCH_FINE;     // start fine search.
		//}

		// for debugging.
		//if(cnt++ > 100)	break;
		// for debugging.
	}

	// ===================================================
	// Find intersection between ray and surface.
	// ===================================================
	if(state==BEHIND_SURFACE){

		// compute trilinear interpolated TSDF value for the last step.
		step_sz = step_scale*tsdf;	vox_s += step_sz*rd_g;
		d_gtvi_Get_TSDF_Value_Interpolated(vox_s,cube_tsdf,cube_w,tsdf);
		// compute the final step of current ray for extracting the surface point.
		step_sz = step_scale*tsdf;	vox_s += step_sz*rd_g;
		// converts points coordinates from the cube voxel coordinates to the real global coordinates . (voxel->mm)
		d_gpw_Get_Position_in_World(vox_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d);

		flag_result = true;
	}

	return flag_result;
	
}

__device__ bool d_crtc_Cast_Ray_on_TSDF_Cube(
	// variable parameters.
	Vector2f p2d,
	const float *T_gc,const float *T_cg,
	Vector3f cam_cen,Vector3f light,Vector3f norm_cc,float dist_cg_cen,float theta_max,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_w,
	int ww,int hh,
	float r_cube,float mu,float sz_vox_inv,int dim_sc,
	// for output cross point.
	Vector3f &p3d)
{

	// RENDERING NOW!!!!!
	Vector3f p3d_c, p3d_s, p3d_e, vox_s, vox_e, rd_g, rd_vox;
	float cos_val, theta, mag, dist_min, dist_max;

	bool flag_valid = false;
	bool flag_result = false;
	int cnt = 0;
	float step_sz, step_sz_coarse, step_scale, sz_sub_cube, total_step, total_step_max, tsdf;

	// cast ray.
	enum { SEARCH_COARSE,SEARCH_FINE,BEHIND_SURFACE,WRONG_STEP } state;

	dist_min = dist_cg_cen - r_cube;
	dist_max = dist_cg_cen + r_cube;

	// compute the end point of the pixel ray for the input pixel.
	d_bp_Back_Project(p2d, K_dev, dist_max, p3d_c);
	d_t_Transform(p3d_c, T_cg, p3d_e);

	// ==================================================================
	// Ray validity test
	// ==================================================================
	// compute the direction of the pixel ray for the input pixel using camera center and far plane of view frustum.
	rd_g = (p3d_e - cam_cen).normalised();

	// compute angle between the pixel ray and direction vector between the two centers.
	// check whether the ray crossed the object cube.
	//cos_val = rd.x*norm_cc.x + rd.y*norm_cc.y + rd.z*norm_cc.z;
	theta = acosf(dot(rd_g, norm_cc));

	if(theta > theta_max){ return false; }

	//// ==================================================================
	//// Ray casting in voxel coordinates.
	//// ==================================================================
	// initialization.
	// calculates start point and ray direction for ray-casting.	
	p3d_s=cam_cen+dist_min*rd_g;
	// converts points and parameter coordinates from the real global coordinates to the cube voxel coordinates. (mm->voxel)
	// CAUTION: the offset position of global voxel cube is (-0.5f, -0.5f, -0.5f) in the cube voxel coordinates.
	d_gpv_Get_Position_in_Voxel(p3d_s, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], vox_s);
	// compute scale factor for converting TSDF value to 'mm' unit.	
	step_scale=mu*sz_vox_inv;
	// set initial and coarse-search step size for ray-casting as the side length of the sub cube.	
	// + coarse-search step size should be larger than 'step_scale'.
	sz_sub_cube=dim_sc;
	step_sz=step_sz_coarse=fmaxf(step_scale+0.1f,sz_sub_cube);
	// set maximum search length of the pixel ray to 2*in_radius_cube.
	total_step_max=2.0f*r_cube*sz_vox_inv;		//	 total_step_max=pe_g.d_Distance(ps_g)*in_voxel_sz_inv;

	// starts casting ray.
	total_step=0.0f;
	flag_valid=d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s,cube_tsdf,cube_w,tsdf);
	if(!flag_valid)		state=SEARCH_COARSE;
	else if(tsdf<=0.0f)	state=WRONG_STEP;
	else				state=SEARCH_FINE;

	while(state!=BEHIND_SURFACE){
		if(!flag_valid){
			switch(state){

			case SEARCH_COARSE:
				step_sz=step_sz_coarse;
				break;

			default:
			case WRONG_STEP:
			case SEARCH_FINE:
				state=SEARCH_COARSE;
				step_sz=step_sz_coarse;
				break;
			}
		} else{
			switch(state){

			case SEARCH_COARSE:
				// at the first SEARCH_COARSE state, 
				// return previous step (the last SEARCH_BLOCK_COARSE state) and do fine search
				// stepLength: SDF_BLOCK_SIZE-> sdfValue * stepScale (at the first fine search, assume that sdfValue is 1.0f).
				//state=BEHIND_SURFACE;
				state=SEARCH_FINE;
				step_sz=step_scale*tsdf;
				break;
			case WRONG_STEP:
				step_sz=fminf(step_scale*tsdf,-1.0f);
				break;

			default:
			case SEARCH_FINE:
				step_sz=fmaxf(step_scale*tsdf,1.0f);
			}
		}

		// proceeds ray casting.
		vox_s+=step_sz*rd_g;	total_step+=step_sz;
		if(total_step>total_step_max)	break;

		flag_valid=d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s,cube_tsdf,cube_w,tsdf);

		if(tsdf>-1.0f && tsdf<1.0f){
			// add interpolation version of TSDF value calculator.
			d_gtvi_Get_TSDF_Value_Interpolated(vox_s,cube_tsdf,cube_w,tsdf);
		}

		if(tsdf<0.0f){			// behind surface.
			if(state==SEARCH_COARSE)	state=WRONG_STEP;
			else						state=BEHIND_SURFACE;	//else if(state==SEARCH_FINE)		state=BEHIND_SURFACE;	
		} 
		else if(tsdf>0.0f){		// in front of surface.
			if(state==WRONG_STEP)	state=SEARCH_FINE;
		} 
		else{					// on surface.
			// converts points coordinates from the cube voxel coordinates to the real global coordinates . (voxel->mm)
			d_gpw_Get_Position_in_World(vox_s, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], p3d);

			return true;
		}

	}

	if(state==BEHIND_SURFACE){

		step_sz=step_scale*tsdf;	vox_s+=step_sz*rd_g;
		//in_cube->gtvu_Get_TSDF_Value_Uninterpolated(ps_vox, tsdf);
		d_gtvi_Get_TSDF_Value_Interpolated(vox_s,cube_tsdf,cube_w,tsdf);

		step_sz=step_scale*tsdf;	vox_s+=step_sz*rd_g;

		// converts points coordinates from the cube voxel coordinates to the real global coordinates . (voxel->mm)
		d_gpw_Get_Position_in_World(vox_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d);

		flag_result=true;
	}

	return flag_result;

	//dist_min = dist_cg_cen - r_cube;
	//dist_max = dist_cg_cen + r_cube;

	//// compute the end point of the pixel ray for the input pixel.
	//d_bp_Back_Project(p2d, K_dev, dist_max, p3d_c);
	//d_t_Transform(p3d_c, T_cg, p3d_e);

	//// ==================================================================
	//// Ray validity test
	//// ==================================================================
	//// compute the direction of the pixel ray for the input pixel using camera center and far plane of view frustum.
	//rd_g = (p3d_e - cam_cen).normalised();

	//// compute angle between the pixel ray and direction vector between the two centers.
	//// check whether the ray crossed the object cube.
	////cos_val = rd.x*norm_cc.x + rd.y*norm_cc.y + rd.z*norm_cc.z;
	//theta = acosf(dot(rd_g, norm_cc));

	//if(theta > theta_max){ return false; }

	//// ==================================================================
	//// Ray casting in voxel coordinates.
	//// ==================================================================
	//// calculates start point and ray direction for ray-casting in voxel coordinates.	
	//p3d_s = cam_cen + dist_min*rd_g;
	//// converts points and parameter coordinates from the real global coordinates to the cube voxel coordinates. (mm->voxel)
	//// CAUTION: the offset position of global voxel cube is (-0.5f, -0.5f, -0.5f) in the cube voxel coordinates.
	//// compute scale factor for converting TSDF value to 'mm' unit.	
	//step_scale = mu*sz_vox_inv;
	//// ======================================================
	//// ���� coarse search �� step_scale �� �����ؾ� ��.
	//// ���� ray �� invalid �� ���� ������ ���� ��� (tsdf ���� ����) 
	//// ���� coarse search ���� �� �ܰ� ���� coarse search (step size = dim_sc) �� �����ص� �� ��.
	//step_sz = step_sz_coarse = step_scale; //fmaxf(step_scale+0.1f, float(dim_sc));
	//// ======================================================
	//// set maximum search length of the pixel ray to 2*in_radius_cube.
	//total_step_max = 2.0f*r_cube*sz_vox_inv;		//	 total_step_max=pe_g.d_Distance(ps_g)*in_voxel_sz_inv;

	//// starts casting ray.
	//total_step = 0.0f;
	//// Validity check ���ص� �ǳ�??
	//d_gpv_Get_Position_in_Voxel(p3d_s, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], vox_s);
	//// Validity check ���ص� �ǳ�??
	//flag_valid = d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s, cube_tsdf, cube_w, tsdf);
	//if(!flag_valid)		state = SEARCH_COARSE;
	//else if(tsdf<=0.0f)	state = SEARCH_FINE_BACK;
	//else				state = SEARCH_FINE;

	//// ===================================================
	//// Casts ray.
	//// ===================================================
	//while(state!=BEHIND_SURFACE){

	//	// ===================================================
	//	// Sets step size.
	//	// ===================================================
	//	if(state == SEARCH_COARSE)    step_sz = step_sz_coarse;
	//	else if(state == SEARCH_FINE) step_sz = fmaxf(step_scale*tsdf, 1.0f);	// in front of surface.  
	//	else if(state == SEARCH_FINE_BACK)  step_sz = fminf(step_scale*tsdf, -1.0f);  // behind surface.

	//	// ===================================================
	//	// Proceeds ray casting.
	//	// ===================================================
	//	vox_s += step_sz*rd_g;	//vox_s.x += step_sz*rd.x; vox_s.y += step_sz*rd.y; vox_s.z += step_sz*rd.z;
	//	total_step += step_sz;
	//	// if total step length exceeds maximum step size, break while loop.
	//	if(total_step > total_step_max)	break;

	//	flag_valid = d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s, cube_tsdf, cube_w, tsdf);

	//	// approached near the surface.
	//	if(tsdf>-1.0f && tsdf<1.0f){
	//		// add interpolation version of TSDF value calculator.
	//		flag_valid = d_gtvi_Get_TSDF_Value_Interpolated(vox_s, cube_tsdf, cube_w, tsdf);
	//	}

	//	// ===================================================
	//	// Checks casting mode.
	//	// ===================================================
	//	if(state == SEARCH_COARSE){
	//		if(tsdf<=0.0f) state = SEARCH_FINE_BACK; // go back.
	//		else if(tsdf<1.0f)    state = SEARCH_FINE;		 // start fine search.
	//	}
	//	else if(state == SEARCH_FINE){
	//		// ================================================================
	//		// Infinite Loop Cause 1
	//		// : In case of noisy TSDF data processing.
	//		// Solution: Remove noisy data processing part.
	//		// if(tsdf >= 1.0f || tsdf <= -1.0f) state = SEARCH_COARSE;  // convert to coarse search. (ray met noise voxel data)
	//		// ================================================================
	//		if(tsdf<=0.0f)  state = BEHIND_SURFACE; // break while loop, and find intersection.
	//	}
	//	else if(state == SEARCH_FINE_BACK){
	//		if(tsdf>0.0f) state = SEARCH_FINE;     // start fine search.
	//	}

	//	// for debugging.
	//	//if(cnt++ > 100)	break;
	//	// for debugging.
	//}

	//// ===================================================
	//// Find intersection between ray and surface.
	//// ===================================================
	//if(state==BEHIND_SURFACE){

	//	// compute trilinear interpolated TSDF value for the last step.
	//	step_sz = step_scale*tsdf;	vox_s += step_sz*rd_g;
	//	d_gtvi_Get_TSDF_Value_Interpolated(vox_s, cube_tsdf, cube_w, tsdf);
	//	// compute the final step of current ray for extracting the surface point.
	//	step_sz = step_scale*tsdf;	vox_s += step_sz*rd_g;
	//	// converts points coordinates from the cube voxel coordinates to the real global coordinates . (voxel->mm)
	//	d_gpw_Get_Position_in_World(vox_s, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], p3d);

	//	flag_result = true;
	//}

	//return flag_result;

}

__device__ bool d_crtcs_Cast_Ray_on_TSDF_Cube_for_Scene(
	// variable parameters.
	Vector2f p2d,
	const float *T_gc,const float *T_cg,
	Vector3f cam_cen,Vector3f light,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_w,
	int ww,int hh,
	float r_cube,float mu,float sz_vox_inv,int dim_sc,
	// for output cross point.
	Vector3f &p3d)
{
	// RENDERING NOW!!!!!
	Vector3f p3d_c,p3d_s,p3d_e,vox_s,vox_e,rd_g,rd_vox;
	float mag,dist_min,dist_max;

	bool flag_valid = false;
	bool flag_result = false;
	int cnt = 0;
	float step_sz,step_sz_coarse,step_scale,sz_sub_cube,total_step,total_step_max,tsdf;

	enum { SEARCH_COARSE,SEARCH_FINE,BEHIND_SURFACE,SEARCH_FINE_BACK } state;

	dist_min = 0.0f;
	dist_max = 2*r_cube;

	// compute the end point of the pixel ray for the input pixel.
	d_bp_Back_Project(p2d,K_dev,dist_max,p3d_c);
	d_t_Transform(p3d_c,T_cg,p3d_e);

	// ==================================================================
	// Ray validity test
	// ==================================================================
	// compute the direction of the pixel ray for the input pixel using camera center and far plane of view frustum.
	rd_g = (p3d_e - cam_cen).normalised();

	// ==================================================================
	// Ray casting in voxel coordinates.
	// ==================================================================
	// calculates start point and ray direction for ray-casting in voxel coordinates.	
	p3d_s = cam_cen + dist_min*rd_g;
	// converts points and parameter coordinates from the real global coordinates to the cube voxel coordinates. (mm->voxel)
	// CAUTION: the offset position of global voxel cube is (-0.5f, -0.5f, -0.5f) in the cube voxel coordinates.
	// compute scale factor for converting TSDF value to 'mm' unit.	
	step_scale = 0.9f*mu*sz_vox_inv;
	// ======================================================
	// ���� coarse search �� step_scale �� �����ؾ� ��.
	// ���� ray �� invalid �� ���� ������ ���� ��� (tsdf ���� ����) 
	// ���� coarse search ���� �� �ܰ� ���� coarse search (step size = dim_sc) �� �����ص� �� ��.
	step_sz = step_sz_coarse = step_scale; //fmaxf(step_scale+0.1f, float(dim_sc));
	// ======================================================
	// set maximum search length of the pixel ray to 2*in_radius_cube.
	total_step_max = 2.0f*r_cube*sz_vox_inv;		//	 total_step_max=pe_g.d_Distance(ps_g)*in_voxel_sz_inv;

	// starts casting ray.
	total_step = 0.0f;
	// Validity check ���ص� �ǳ�??
	d_gpv_Get_Position_in_Voxel(p3d_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],vox_s);
	// Validity check ���ص� �ǳ�??
	flag_valid = d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s,cube_tsdf,cube_w,tsdf);
	if(!flag_valid)		state = SEARCH_COARSE;
	else if(tsdf>=0.0f)	state = SEARCH_FINE;
	else				return false; //state = SEARCH_FINE_BACK;


	// ===================================================
	// Casts ray.
	// ===================================================
	while(state!=BEHIND_SURFACE){

		// ===================================================
		// Sets step size.
		// ===================================================
		if(state == SEARCH_COARSE)  step_sz = step_sz_coarse;
		else step_sz = (tsdf>0.0f) ? fmaxf(step_scale*tsdf,1.0f) : fminf(step_scale*tsdf,-1.0f);
		//		else if(state == SEARCH_FINE) step_sz = fmaxf(step_scale*tsdf,0.1f);	// in front of surface.  
		//		else if(state == SEARCH_FINE_BACK)  step_sz = fminf(step_scale*tsdf,-0.1f);  // behind surface.

		// ===================================================
		// Proceeds ray casting.
		// ===================================================
		vox_s += step_sz*rd_g;	//vox_s.x += step_sz*rd.x; vox_s.y += step_sz*rd.y; vox_s.z += step_sz*rd.z;
		total_step += step_sz;
		// if total step length exceeds maximum step size, break while loop.
		if(total_step > total_step_max)	break;

		// compute uninterpolated TSDF value.
		flag_valid = d_gtvu_Get_TSDF_Value_Uninterpolated(vox_s,cube_tsdf,cube_w,tsdf);

		// check which current points is in mu band or not. (near the surface)
		if(tsdf>-1.0f && tsdf<1.0f){
			// ===========================================================
			// Mu-band processing.
			// ===========================================================
			// compute trilinear interpolated TSDF value.
			flag_valid = d_gtvi_Get_TSDF_Value_Interpolated(vox_s,cube_tsdf,cube_w,tsdf);

			// ===================================================
			// Checks casting mode.
			// ===================================================
			// check current TSDF value is (-) at fine search step.
			if(tsdf<0.0f){
				// terminate ray casting if current TSDF value is (-) at coarse searching mode.
				if(state == SEARCH_COARSE) break;
				else state = BEHIND_SURFACE;
			} else if(tsdf == 0.0f){
				// on surface.
				d_gpw_Get_Position_in_World(vox_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d);
				flag_result=true;
				break;
			} else if(state == SEARCH_COARSE)	state = SEARCH_FINE;
		} else{
			if(state == SEARCH_FINE) state == SEARCH_COARSE;
		}

		//// ===================================================
		//// Checks casting mode.
		//// ===================================================
		//if(state == SEARCH_COARSE){
		//	if(tsdf<=0.0f) state = SEARCH_FINE_BACK; // go back.
		//	else           state = SEARCH_FINE;		 // start fine search.
		//} else if(state == SEARCH_FINE){
		//	// ================================================================
		//	// Infinite Loop Cause 1
		//	// : In case of noisy TSDF data processing.
		//	// Solution: Remove noisy data processing part.
		//	// if(tsdf >= 1.0f || tsdf <= -1.0f) state = SEARCH_COARSE;  // convert to coarse search. (ray met noise voxel data)
		//	// ================================================================
		//	if(tsdf<=0.0f)  state = BEHIND_SURFACE; // break while loop, and find intersection.
		//} else if(state == SEARCH_FINE_BACK){
		//	if(tsdf>0.0f) state = SEARCH_FINE;     // start fine search.
		//}

		// for debugging.
		//if(cnt++ > 100)	break;
		// for debugging.
	}

	// ===================================================
	// Find intersection between ray and surface.
	// ===================================================
	if(state==BEHIND_SURFACE){

		// compute trilinear interpolated TSDF value for the last step.
		step_sz = step_scale*tsdf;	vox_s += step_sz*rd_g;
		d_gtvi_Get_TSDF_Value_Interpolated(vox_s,cube_tsdf,cube_w,tsdf);
		// compute the final step of current ray for extracting the surface point.
		step_sz = step_scale*tsdf;	vox_s += step_sz*rd_g;
		// converts points coordinates from the cube voxel coordinates to the real global coordinates . (voxel->mm)
		d_gpw_Get_Position_in_World(vox_s,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d);

		flag_result = true;
	}

	return flag_result;

}



__global__ void g_rmi_Render_Maps_for_ICP(
	//const float *T_gc_dev, const float *T_cg_dev,
	Vector3f cam_cen, Vector3f light, Vector3f norm_cc, float dist_cg_cen, float theta_max,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_color,
	const uchar *cube_w,	

	int lev_of_pyramid,
	// output.
	float *io_map_depth, float *io_map_vertex,
	float *io_map_normals, uchar *io_img_normals,
	uchar *io_img_color)
{
	Vector2f p2d; Vector3f p3d_g, p3d_c, p3d_v, sn_g, sn_c; Vector3u rgb;	

	bool valid = false;
	int tidx;
	float surf_angle;

	int tx = threadIdx.x + blockIdx.x*blockDim.x;
	int ty = threadIdx.y + blockIdx.y*blockDim.y;

	int ww = dim_map_dev[lev_of_pyramid*2 + 0];
	int hh = dim_map_dev[lev_of_pyramid*2 + 1];

	if(tx < 0 || tx >= ww || ty < 0 || ty >= hh)	return;

	//	if(tx % 3 || ty % 3)	return ;

	tidx = ty*ww + tx;

	// do ray casting.
	p2d.x = tx;	p2d.y = ty;
	valid = d_crtc_Cast_Ray_on_TSDF_Cube_NEW(
	//valid = d_crtc_Cast_Ray_on_TSDF_Cube(
		p2d,
		T_gc_dev_const, T_cg_dev_const,
		//T_gc_dev, T_cg_dev,
		cam_cen, light, norm_cc, dist_cg_cen, theta_max,

		cube_tsdf,
		cube_w,
// 		K_dev,
// 		origin_dev,
// 		dim_cube_dev,
//		sz_vox_inv_dev[0],
		ww, hh,
		lev_of_pyramid,

		r_cube_dev[0], mu_dev[0], sz_vox_inv_dev[0], dim_sc_dev[0],

		p3d_g);

	// update maps.
	if(valid){
		if(d_csnt_Compute_Surface_Normal_from_TSDF(
			p3d_g, 
			cube_tsdf, cube_w, 
// 			origin_dev,
// 			dim_cube_dev,
//			sz_vox_inv_dev[0],
			sn_g)){

			// set depth.
			d_t_Transform(p3d_g, T_gc_dev_const, p3d_c);
			io_map_depth[tidx] = p3d_c.z;

			//////////////////////////////////////////////////////////////////////////
			// + in global coordinates.
			//////////////////////////////////////////////////////////////////////////
// 			// set vertex.
// 			io_map_vertex[3*tidx] = p3d_g.x;
// 			io_map_vertex[3*tidx + 1] = p3d_g.y;
// 			io_map_vertex[3*tidx + 2] = p3d_g.z;
// 			// set normal.
//  		io_map_normals[3*tidx] = sn.x; 
//  		io_map_normals[3*tidx + 1] = sn.y; 
//  		io_map_normals[3*tidx + 2] = sn.z;

			//////////////////////////////////////////////////////////////////////////
			// + in local camera coordinates.
			//////////////////////////////////////////////////////////////////////////
			// set vertex.
			io_map_vertex[3*tidx] = p3d_c.x;
			io_map_vertex[3*tidx + 1] = p3d_c.y;
			io_map_vertex[3*tidx + 2] = p3d_c.z;
			// set normal.
			d_r_Rotate(sn_g,T_gc_dev_const,sn_c);
			io_map_normals[3*tidx] = sn_c.x;
			io_map_normals[3*tidx + 1] = sn_c.y;
			io_map_normals[3*tidx + 2] = sn_c.z;

			// set normal image.
			surf_angle = sn_c.x*light.x + sn_c.y*light.y + sn_c.z*light.z;
			io_img_normals[tidx] = (uchar)(fmaxf(0.0f, fminf(255.0f, (0.8f * surf_angle + 0.2f) * 255.0f)));
			// set RGB color image.
			if(io_img_color){
				d_gpv_Get_Position_in_Voxel(p3d_g, origin_dev, dim_cube_dev, sz_vox_inv_dev[0], p3d_v);
				
				d_grvi_Get_RGB_Value_Interpolated(p3d_v, cube_color, cube_w, rgb);

				io_img_color[tidx] = rgb.x;
				io_img_color[tidx+ww*hh] = rgb.y;
				io_img_color[tidx+2*ww*hh] = rgb.z;

				//////////////////////////////////////////////////////////////////////////
				// �ӽ� �������� G channel ���� R channel��!!
// 				d_grvu_Get_RGB_Value_Uninterpolated(p3d_v,cube_color,cube_w,rgb);
// 				io_img_color[tidx] = rgb.y;
// 				io_img_color[tidx+ww*hh] = rgb.y;
// 				io_img_color[tidx+2*ww*hh] = rgb.z;
				//////////////////////////////////////////////////////////////////////////


			}
		}
		else valid = false;
	}
	
	if(!valid){
		io_map_depth[tidx] = 0.0f;
		// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		// CPU version ���� default value �� -100.0f �� �ѵ�...	
// 		const char type = '0';
// 		io_map_vertex[3*tidx] = io_map_vertex[3*tidx + 1] = io_map_vertex[3*tidx + 2] = nanf(&type);
// 		io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = nanf(&type);
		io_map_vertex[3*tidx] = io_map_vertex[3*tidx + 1] = io_map_vertex[3*tidx + 2] = -100.0f;
		io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = -100.0f;
		// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		io_img_normals[tidx] = uchar(0);
		if(io_img_color){
			io_img_color[tidx] = uchar(0);
			io_img_color[tidx+ww*hh] = uchar(0);
			io_img_color[tidx+2*ww*hh] = uchar(0);

		}
	}


}

__global__ void g_rmi_Render_Maps_for_ICP(
	//const float *T_gc_dev, const float *T_cg_dev,
	Vector3f cam_cen,Vector3f light,Vector3f norm_cc,float dist_cg_cen,float theta_max,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_color,
	const uchar *cube_w,
	int lev_of_pyramid,
	// output.
	float *io_map_depth,float *io_map_normals,uchar *io_img_normals,
	uchar *io_img_color)
{
	Vector2f p2d; Vector3f p3d_g,p3d_c,p3d_v,sn; Vector3u rgb;

	bool valid = false;
	int tidx;
	float surf_angle;

	int tx = threadIdx.x + blockIdx.x*blockDim.x;
	int ty = threadIdx.y + blockIdx.y*blockDim.y;

	int ww = dim_map_dev[lev_of_pyramid*2 + 0];
	int hh = dim_map_dev[lev_of_pyramid*2 + 1];

	if(tx < 0 || tx >= ww || ty < 0 || ty >= hh)	return;

	//	if(tx % 3 || ty % 3)	return ;

	tidx = ty*ww + tx;

	// do ray casting.
	p2d.x = tx;	p2d.y = ty;
	valid = d_crtc_Cast_Ray_on_TSDF_Cube_NEW(
	//valid = d_crtc_Cast_Ray_on_TSDF_Cube(
		p2d,
		T_gc_dev_const,T_cg_dev_const,
		//T_gc_dev, T_cg_dev,
		cam_cen,light,norm_cc,dist_cg_cen,theta_max,

		cube_tsdf,
		cube_w,
// 		K_dev,
// 		origin_dev,
// 		dim_cube_dev,
//		sz_vox_inv_dev[0],
		ww,hh,
		lev_of_pyramid,

		r_cube_dev[0],mu_dev[0],sz_vox_inv_dev[0],dim_sc_dev[0],
		p3d_g);

	// update maps.
	if(valid){
		if(d_csnt_Compute_Surface_Normal_from_TSDF(
			p3d_g,
			cube_tsdf,cube_w,
// 			origin_dev,
// 			dim_cube_dev,
//			sz_vox_inv_dev[0],
			sn)){

			surf_angle = sn.x*light.x + sn.y*light.y + sn.z*light.z;

			// set depth.
			d_t_Transform(p3d_g,T_gc_dev_const,p3d_c);
			io_map_depth[tidx] = p3d_c.z;
			// set normal.
			io_map_normals[3*tidx] = sn.x;
			io_map_normals[3*tidx + 1] = sn.y;
			io_map_normals[3*tidx + 2] = sn.z;
			// set normal image.
			io_img_normals[tidx] = (uchar)(fmaxf(0.0f,fminf(255.0f,(0.8f * surf_angle + 0.2f) * 255.0f)));
			// set RGB color image.
			if(io_img_color){
				d_gpv_Get_Position_in_Voxel(p3d_g,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d_v);
					
				d_grvi_Get_RGB_Value_Interpolated(p3d_v,cube_color,cube_w,rgb);

				io_img_color[tidx] = rgb.x;
				io_img_color[tidx+ww*hh] = rgb.y;
				io_img_color[tidx+2*ww*hh] = rgb.z;

				//////////////////////////////////////////////////////////////////////////
				// �ӽ� �������� G channel ���� R channel��!!
// 					d_grvu_Get_RGB_Value_Uninterpolated(p3d_v,cube_color,cube_w,rgb);
// 					io_img_color[tidx] = rgb.y;
// 					io_img_color[tidx+ww*hh] = rgb.y;
// 					io_img_color[tidx+2*ww*hh] = rgb.z;
				//////////////////////////////////////////////////////////////////////////

				//////////////////////////////////////////////////////////////////////////
				// �ӽ� �������� G channel ���� R channel��!!
				//d_grvu_Get_RGB_Value_Uninterpolated(p3d_v,cube_color,cube_w,rgb);
				//io_img_color[tidx] = rgb.y;
				//////////////////////////////////////////////////////////////////////////

			}
		} else valid = false;
	}

	if(!valid){
		io_map_depth[tidx] = 0.0f;
		// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		// CPU version ���� default value �� -100.0f �� �ѵ�...	
		//const char type = '0';
		//io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = nanf(&type);
		io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = -100.0f;
		// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		io_img_normals[tidx] = uchar(0);
		if(io_img_color){
			// black background
			io_img_color[tidx] = uchar(0);
			io_img_color[tidx+ww*hh] = uchar(0);
			io_img_color[tidx+2*ww*hh] = uchar(0);

			// white background
			io_img_color[tidx] = uchar(255);
			io_img_color[tidx+ww*hh] = uchar(255);
			io_img_color[tidx+2*ww*hh] = uchar(255);


		}
	}


}

__global__ void g_rmis_Render_Maps_for_Scene(
//const float *T_gc_dev, const float *T_cg_dev,
	Vector3f cam_cen,Vector3f light,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_color,
	const uchar *cube_w,
	// output.
	float *io_map_depth,float *io_map_vertex,float *io_map_normals,uchar *io_img_normals,
	uchar *io_img_color)
{
	Vector2f p2d; Vector3f p3d_g,p3d_c,p3d_v,sn; Vector3u rgb;

	bool valid = false;
	int tidx;
	float surf_angle;

	int tx = threadIdx.x + blockIdx.x*blockDim.x;
	int ty = threadIdx.y + blockIdx.y*blockDim.y;

	int ww = dim_map_dev[0];
	int hh = dim_map_dev[1];

	if(tx < 0 || tx >= ww || ty < 0 || ty >= hh)	return;

	//	if(tx % 3 || ty % 3)	return ;

	tidx = ty*ww + tx;

	// do ray casting.
	p2d.x = tx;	p2d.y = ty;
	valid = d_crtcs_Cast_Ray_on_TSDF_Cube_for_Scene(
		p2d,
		T_gc_dev_const,T_cg_dev_const,
		//T_gc_dev, T_cg_dev,
		cam_cen,light,

		cube_tsdf,
		cube_w,
// 		K_dev,
// 		origin_dev,
// 		dim_cube_dev,
//		sz_vox_inv_dev[0],
		ww,hh,
		r_cube_dev[0],mu_dev[0],sz_vox_inv_dev[0],dim_sc_dev[0],

		p3d_g);

	// update maps.
	if(valid){
		if(d_csnt_Compute_Surface_Normal_from_TSDF(
			p3d_g,
			cube_tsdf,cube_w,
// 			origin_dev,
// 			dim_cube_dev,
//			sz_vox_inv_dev[0],
			sn)){

			surf_angle = sn.x*light.x + sn.y*light.y + sn.z*light.z;

			// set depth.
			d_t_Transform(p3d_g,T_gc_dev_const,p3d_c);
			io_map_depth[tidx] = p3d_c.z;
			// set vertex.
			io_map_vertex[3*tidx] = p3d_c.x;
			io_map_vertex[3*tidx + 1] = p3d_c.y;
			io_map_vertex[3*tidx + 2] = p3d_c.z;
			// set normal.
			io_map_normals[3*tidx] = sn.x;
			io_map_normals[3*tidx + 1] = sn.y;
			io_map_normals[3*tidx + 2] = sn.z;
			// set normal image.
			io_img_normals[tidx] = (uchar)(fmaxf(0.0f,fminf(255.0f,(0.8f * surf_angle + 0.2f) * 255.0f)));
			// set RGB color image.
			if(io_img_color){
				d_gpv_Get_Position_in_Voxel(p3d_g,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d_v);
				//d_grvi_Get_RGB_Value_Interpolated(p3d_v,cube_color,cube_w,rgb);
				d_grvu_Get_RGB_Value_Uninterpolated(p3d_v,cube_color,cube_w,rgb);

				// �ӽ� �������� G channel ���� R channel��!!
				io_img_color[tidx] = rgb.y;
				//io_img_color[tidx] = rgb.x;
				io_img_color[tidx+ww*hh] = rgb.y;
				io_img_color[tidx+2*ww*hh] = rgb.z;

			}
		} else valid = false;
	}

	if(!valid){
		io_map_depth[tidx] = 0.0f;
		// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		// CPU version ���� default value �� -100.0f �� �ѵ�...	
// 		const char type = '0';
// 		io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = nanf(&type);
		io_map_vertex[3*tidx] = io_map_vertex[3*tidx + 1] = io_map_vertex[3*tidx + 2] = -100.0f;
		io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = -100.0f;
		// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
		io_img_normals[tidx] = uchar(0);
		if(io_img_color){
			io_img_color[tidx] = uchar(0);
			io_img_color[tidx+ww*hh] = uchar(0);
			io_img_color[tidx+2*ww*hh] = uchar(0);

		}
	}


}

__global__ void g_rmis_Render_Maps_for_Scene(
//const float *T_gc_dev, const float *T_cg_dev,
	Vector3f cam_cen,Vector3f light,
	// fixed parameters.
	const float *cube_tsdf,
	const uchar *cube_color,
	const uchar *cube_w,
	// output.
	float *io_map_depth,float *io_map_normals,uchar *io_img_normals,
	uchar *io_img_color)
{
		Vector2f p2d; Vector3f p3d_g,p3d_c,p3d_v,sn; Vector3u rgb;

		bool valid = false;
		int tidx;
		float surf_angle;

		int tx = threadIdx.x + blockIdx.x*blockDim.x;
		int ty = threadIdx.y + blockIdx.y*blockDim.y;

		int ww = dim_map_dev[0];
		int hh = dim_map_dev[1];

		if(tx < 0 || tx >= ww || ty < 0 || ty >= hh)	return;

		//	if(tx % 3 || ty % 3)	return ;

		tidx = ty*ww + tx;

		// do ray casting.
		p2d.x = tx;	p2d.y = ty;
		valid = d_crtcs_Cast_Ray_on_TSDF_Cube_for_Scene(
			p2d,
			T_gc_dev_const,T_cg_dev_const,
			//T_gc_dev, T_cg_dev,
			cam_cen,light,

			cube_tsdf,
			cube_w,
	// 		K_dev,
	// 		origin_dev,
	// 		dim_cube_dev,
	//		sz_vox_inv_dev[0],
			ww,hh,
			r_cube_dev[0],mu_dev[0],sz_vox_inv_dev[0],dim_sc_dev[0],

			p3d_g);

		// update maps.
		if(valid){
			if(d_csnt_Compute_Surface_Normal_from_TSDF(
				p3d_g,
				cube_tsdf,cube_w,
	// 			origin_dev,
	// 			dim_cube_dev,
	//			sz_vox_inv_dev[0],
				sn)){

				surf_angle = sn.x*light.x + sn.y*light.y + sn.z*light.z;

				// set depth.
				d_t_Transform(p3d_g,T_gc_dev_const,p3d_c);
				io_map_depth[tidx] = p3d_c.z;
				// set normal.
				io_map_normals[3*tidx] = sn.x;
				io_map_normals[3*tidx + 1] = sn.y;
				io_map_normals[3*tidx + 2] = sn.z;
				// set normal image.
				io_img_normals[tidx] = (uchar)(fmaxf(0.0f,fminf(255.0f,(0.8f * surf_angle + 0.2f) * 255.0f)));
				// set RGB color image.
				if(io_img_color){
					d_gpv_Get_Position_in_Voxel(p3d_g,origin_dev,dim_cube_dev,sz_vox_inv_dev[0],p3d_v);
					d_grvi_Get_RGB_Value_Interpolated(p3d_v,cube_color,cube_w,rgb);

					io_img_color[tidx] = rgb.x;
					io_img_color[tidx+ww*hh] = rgb.y;
					io_img_color[tidx+2*ww*hh] = rgb.z;

				}
			} else valid = false;
		}

		if(!valid){
			io_map_depth[tidx] = 0.0f;
			// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
			// CPU version ���� default value �� -100.0f �� �ѵ�...	
			// 		const char type = '0';
			// 		io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = nanf(&type);
			io_map_normals[3*tidx] = io_map_normals[3*tidx + 1] = io_map_normals[3*tidx + 2] = -100.0f;
			// <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<
			io_img_normals[tidx] = uchar(0);
			if(io_img_color){
				io_img_color[tidx] = uchar(0);
				io_img_color[tidx+ww*hh] = uchar(0);
				io_img_color[tidx+2*ww*hh] = uchar(0);

			}
		}


	}


// ==================================================================================================
// HOST / DEVICE BARRIER | HOST / DEVICE BARRIER | HOST / DEVICE BARRIER | HOST / DEVICE BARRIER | 
// ==================================================================================================


/////////////////////////////////////////////////////////////////////////////////////////////
// LGKvRendererTSDF 
/////////////////////////////////////////////////////////////////////////////////////////////

// *******************************************************
__host__ LGKvRendererTSDF::LGKvRendererTSDF()
// *******************************************************
{

}

// *******************************************************
__host__ LGKvRendererTSDF::~LGKvRendererTSDF()
// *******************************************************
{

}

// *******************************************************
// CAUTION: �� Ŭ������ ���ÿ� ������ ������ ���ο��� ���� constant memory ���� �������Ѿ� �ҵ� �ϴ�.
// ���� �̰͵��� ������ �Ǿ ���� �ٸ� rendering class �� ���ÿ� ����ϸ� ������ �߻��ϴµ�.
__host__ void LGKvRendererTSDF::ip_Initialize_Parameters(
	GKvObjectCubeFloat *in_cube,
	int ww, int hh,
	float fx, float fy,
	float px, float py,
	float mu,
	float max_w)
// *******************************************************
{
	float intrins_host[3*4], origin_host[3], light_host[3];
	float sz_vox_inv_host[1], mu_host[1], r_host[1], max_w_host[1];
	int dim_cube_host[3], dim_sc_host[1], dim_map_host[3*2];

	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();
	
	origin_host[0] = cube_org.x;	origin_host[1] = cube_org.y;	origin_host[2] = cube_org.z;

//	printf("cube_org: %f %f %f\n", cube_org.x, cube_org.y, cube_org.z);
// 	printf("cube_cen: %f %f %f\n", cube_cen.x, cube_cen.y, cube_cen.z);
// 	printf("%f %f\n", mu, max_w);

	mu_host[0] = mu;
	sz_vox_inv_host[0] = 1.0f/in_cube->sz_vox();
	r_host[0] = length(cube_org - cube_cen);
	max_w_host[0] = max_w;
	
	in_cube->ts(dim_cube_host[0], dim_cube_host[1], dim_cube_host[2]);
	dim_sc_host[0] = in_cube->dim_sc();

	//printf("dim_cube_host: %d %d %d\n", dim_cube_host[0], dim_cube_host[1], dim_cube_host[2]);
	int tww, thh;	tww = ww; thh = hh; 
	float *p_intrins = &intrins_host[0];
	int *p_dim_maps = &dim_map_host[0];
	for(int k=0; k<3; k++){
		
		p_intrins[0] = fx;	p_intrins[1] = fy;	p_intrins[2] = px;	p_intrins[3] = py;
		p_dim_maps[0] = tww;	p_dim_maps[1] = thh;

		// downsizing.
		fx = 0.5f*fx; fy = 0.5f*fy; px = 0.5f*(px - 0.5f); py = 0.5f*(py - 0.5f);
		tww /= 2; thh /= 2;

		p_intrins += 4;
		p_dim_maps += 2;
		
	}
// 	for(int i=0; i<12; i++) printf("%f\n", intrins_host[i]);
// 	for(int i=0; i<6; i++) printf("%d\n", dim_map_host[i]);
	hipMemcpyToSymbol(HIP_SYMBOL(K_dev),intrins_host,3*4 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_map_dev),dim_map_host,3*2 * sizeof(int));
	//  	
	hipMemcpyToSymbol(HIP_SYMBOL(origin_dev), origin_host, 3 * sizeof(float));
	//  	
	hipMemcpyToSymbol(HIP_SYMBOL(mu_dev), mu_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(max_w_dev), max_w_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sz_vox_inv_dev), sz_vox_inv_host, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(r_cube_dev), r_host, sizeof(float));

	// 
	hipMemcpyToSymbol(HIP_SYMBOL(dim_cube_dev), dim_cube_host, 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dim_sc_dev), dim_sc_host, sizeof(int));
}

// *******************************************************
__host__ void LGKvRendererTSDF::rmi_Render_Maps_for_ICP(
	GKvObjectCubeFloat *in_cube,
	int in_ww, int in_hh,
	const float *in_T_gc_dev, const float *in_T_cg_dev,	

	Vector3f in_cam_cen, Vector3f in_light,

	int in_lev_of_pyram,

	float *io_map_depth_dev,
	float *io_map_vertex_dev,
	float *io_map_normals_dev,
	uchar *io_img_normals_dev,
	
	uchar *io_img_color_dev)
// *******************************************************
{
	float *map_depth_dev = io_map_depth_dev;
	float *map_vertex_dev = io_map_vertex_dev;
	float *map_normal_dev = io_map_normals_dev;
	uchar *img_normal_dev = io_img_normals_dev;
	
	int block_sz, grid_sz;

	// Rendering parameters.
	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();
	Vector3f cam_cen = in_cam_cen;
	Vector3f vec_cc = cube_cen - cam_cen;
	Vector3f norm_cc = vec_cc.normalised();

	float r_cube = length(cube_org - cube_cen); //sqrtf(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	float dist_cc = length(vec_cc);
	float theta_max = asin(r_cube/dist_cc);
	
	// For depth map rendering.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X, CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(in_ww, threads.x), iDivUp(in_hh, threads.y));

	//printf("block: %d %d / thread: %d %d\n", blocks.x, blocks.y, threads.x, threads.y);

	// Camera pose.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const), in_T_gc_dev, sizeof(float) * 16, 0, hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const), in_T_cg_dev, sizeof(float) * 16, 0, hipMemcpyDeviceToDevice);

  	g_rmi_Render_Maps_for_ICP<<<blocks, threads>>>(
		//in_T_gc_dev, in_T_cg_dev,		
		in_cam_cen, in_light, norm_cc, dist_cc, theta_max, 		
		//test_tsdf_dev, 
		in_cube->vp_tsdf(),
		in_cube->vp_rgb(),
		in_cube->vp_w(),

		in_lev_of_pyram,

 		map_depth_dev, map_vertex_dev, map_normal_dev, img_normal_dev,
		io_img_color_dev);

}

// *******************************************************
__host__ void LGKvRendererTSDF::rmi_Render_Maps_for_ICP(
	GKvObjectCubeFloat *in_cube,
	int in_ww,int in_hh,
	const float *in_T_gc_dev,const float *in_T_cg_dev,

	Vector3f in_cam_cen,Vector3f in_light,

	int in_lev_of_pyram,

	float *io_map_depth_dev,
	float *io_map_normals_dev,
	uchar *io_img_normals_dev,

	uchar *io_img_color_dev)
// *******************************************************
{
	float *map_depth_dev = io_map_depth_dev;
	float *map_normal_dev = io_map_normals_dev;
	uchar *img_normal_dev = io_img_normals_dev;

	int block_sz,grid_sz;

	// Rendering parameters.
	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();
	Vector3f cam_cen = in_cam_cen;
	Vector3f vec_cc = cube_cen - cam_cen;
	Vector3f norm_cc = vec_cc.normalised();

	float r_cube = length(cube_org - cube_cen); //sqrtf(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);
	float dist_cc = length(vec_cc);
	float theta_max = asin(r_cube/dist_cc);

	// For depth map rendering.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(in_ww,threads.x),iDivUp(in_hh,threads.y));

	// Camera pose.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const),in_T_gc_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const),in_T_cg_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);

	g_rmi_Render_Maps_for_ICP<<<blocks,threads>>>(
		//in_T_gc_dev, in_T_cg_dev,		
		in_cam_cen,in_light,norm_cc,dist_cc,theta_max,
		//test_tsdf_dev, 

		in_cube->vp_tsdf(),
		in_cube->vp_rgb(),
		in_cube->vp_w(),

		in_lev_of_pyram,

		map_depth_dev,map_normal_dev,img_normal_dev,
		io_img_color_dev);

}

// *******************************************************
__host__ void LGKvRendererTSDF::rmis_Render_Maps_for_Scene(
	GKvObjectCubeFloat *in_cube,
	int in_ww, int in_hh,
	const float *in_T_gc_dev, const float *in_T_cg_dev,	

	Vector3f in_cam_cen, Vector3f in_light,

	float *io_map_depth_dev,
	float *io_map_vertex_dev,
	float *io_map_normals_dev,
	uchar *io_img_normals_dev,
	
	uchar *io_img_color_dev)
// *******************************************************
{
	float *map_depth_dev = io_map_depth_dev;
	float *map_vertex_dev = io_map_vertex_dev;
	float *map_normal_dev = io_map_normals_dev;
	uchar *img_normal_dev = io_img_normals_dev;
	
	int block_sz, grid_sz;

	// Rendering parameters.
	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();
	Vector3f cam_cen = in_cam_cen;

	float r_cube = length(cube_org - cube_cen); //sqrtf(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);

// 	printf("r_cube: %f\n", r_cube);
// 	printf("cube_cen: %f %f %f\n", cube_cen.x ,cube_cen.y, cube_cen.z);
// 	printf("in_light: %f %f %f\n", in_light.x, in_light.y, in_light.z);
// 	printf("in_cam_cen: %f %f %f\n", in_cam_cen.x, in_cam_cen.y, in_cam_cen.z);
	
	// For depth map rendering.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X, CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(in_ww, threads.x), iDivUp(in_hh, threads.y));

	// Camera pose.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const), in_T_gc_dev, sizeof(float) * 16, 0, hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const), in_T_cg_dev, sizeof(float) * 16, 0, hipMemcpyDeviceToDevice);

  	g_rmis_Render_Maps_for_Scene<<<blocks, threads>>>(
		//in_T_gc_dev, in_T_cg_dev,		
		in_cam_cen, in_light,	
		//test_tsdf_dev, 
		in_cube->vp_tsdf(),
		in_cube->vp_rgb(),
		in_cube->vp_w(),

 		map_depth_dev, map_vertex_dev, map_normal_dev, img_normal_dev,
		io_img_color_dev);

}

// *******************************************************
__host__ void LGKvRendererTSDF::rmis_Render_Maps_for_Scene(
	GKvObjectCubeFloat *in_cube,
	int in_ww,int in_hh,
	const float *in_T_gc_dev,const float *in_T_cg_dev,

	Vector3f in_cam_cen,Vector3f in_light,

	float *io_map_depth_dev,
	float *io_map_normals_dev,
	uchar *io_img_normals_dev,

	uchar *io_img_color_dev)
// *******************************************************
{
	float *map_depth_dev = io_map_depth_dev;
	float *map_normal_dev = io_map_normals_dev;
	uchar *img_normal_dev = io_img_normals_dev;

	int block_sz,grid_sz;

	// Rendering parameters.
	Vector3f cube_org = in_cube->origin();
	Vector3f cube_cen = in_cube->center();
	Vector3f cam_cen = in_cam_cen;

	float r_cube = length(cube_org - cube_cen); //sqrtf(rvec.x*rvec.x + rvec.y*rvec.y + rvec.z*rvec.z);

	// 	printf("r_cube: %f\n", r_cube);
	// 	printf("cube_cen: %f %f %f\n", cube_cen.x ,cube_cen.y, cube_cen.z);
	// 	printf("in_light: %f %f %f\n", in_light.x, in_light.y, in_light.z);
	// 	printf("in_cam_cen: %f %f %f\n", in_cam_cen.x, in_cam_cen.y, in_cam_cen.z);

	// For depth map rendering.
	dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	dim3 blocks(iDivUp(in_ww,threads.x),iDivUp(in_hh,threads.y));

	// Camera pose.
	hipMemcpyToSymbol(HIP_SYMBOL(T_gc_dev_const),in_T_gc_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(T_cg_dev_const),in_T_cg_dev,sizeof(float) * 16,0,hipMemcpyDeviceToDevice);

	g_rmis_Render_Maps_for_Scene<<<blocks,threads>>>(
		//in_T_gc_dev, in_T_cg_dev,		
		in_cam_cen,in_light,
		//test_tsdf_dev, 
		in_cube->vp_tsdf(),
		in_cube->vp_rgb(),
		in_cube->vp_w(),

		map_depth_dev,map_normal_dev,img_normal_dev,
		io_img_color_dev);

}
