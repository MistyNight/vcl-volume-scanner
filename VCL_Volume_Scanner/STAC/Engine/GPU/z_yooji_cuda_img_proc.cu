#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/////////////////////////////////////////////////////////////////////////////////////////////
// z_yooji_bilateral_kernel.cpp
/////////////////////////////////////////////////////////////////////////////////////////////
//#include "_yooji_2017_cuda_object_scanner.cuh"
//#define __CUDASCAN__
#include "../../_yooji_2017_cuda_object_scanner.cuh"

// for CUDA samples.
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions

__constant__ float cGaussian[64];   //gaussian array in device side
//texture<ushort,2,hipReadModeElementType> ushortTex;
texture<float,2,hipReadModeElementType> floatTex;
 
// uint *dImg  = NULL;   //original image
// uint *dTemp   = NULL;   //temp array for iterations

//ushort *hMap = NULL;		// original image in host array.
//ushort *dMap  = NULL;   //original image in device array.
//ushort *dTemp  = NULL;   //temporal image in device array.

float *hMapF = NULL;		// original image in host array.
float *dMapF  = NULL;   //original image in device array.
float *dTempF  = NULL;   //temporal image in device array.
  
int dwidth = 1, dheight = 1;
size_t dpitch = 1, dpitchF = 1;

int iter_num = 1;
float gaussian_delta = 4;
float euclidean_delta = 0.1f;//0.1f;
int filter_radius = 2;

/*
    Perform a simple bilateral filter.

    Bilateral filter is a nonlinear filter that is a mixture of range
    filter and domain filter, the previous one preserves crisp edges and
    the latter one filters noise. The intensity value at each pixel in
    an image is replaced by a weighted average of intensity values from
    nearby pixels.

    The weight factor is calculated by the product of domain filter
    component(using the gaussian distribution as a spatial distance) as
    well as range filter component(Euclidean distance between center pixel
    and the current neighbor pixel). Because this process is nonlinear,
    the sample just uses a simple pixel by pixel step.

    Texture fetches automatically clamp to edge of image. 1D gaussian array
    is mapped to a 1D texture instead of using shared memory, which may
    cause severe bank conflict.

    Threads are y-pass(column-pass), because the output is coalesced.

    Parameters
    od - pointer to output data in global memory
    d_f - pointer to the 1D gaussian array
    e_d - euclidean delta
    w  - image width
    h  - image height
    r  - filter radius
*/

// ////////////////////////////////////////////////////////////////////////////////
// Local functions .///////////////////////////////////////////////////////////////
// ////////////////////////////////////////////////////////////////////////////////

// ===============================================================================
// Device functions.
// ===============================================================================

//Euclidean Distance (x, y, d) = exp(-(|x - y| / d)^2 / 2)
 __device__ float euclideanDist(float a,float b,float d)
 {

	 float mod = (b - a) * (b - a);

	 return __expf(-mod / (2.f * d * d));
 }

 //Euclidean Similarity (x, y, d) = exp(-(|x - y| / d)^2 / 2)
 __device__ float euclideanSim(float a,float b,float d)
 {

	 float mod = (b - a) * (b - a);

	 return __expf(mod / (2.f * d * d));
 }
 
__global__ void
d_bilateral_filter(float *od,int w,int h,
				   float e_d,int r)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= w || y >= h)
	{
		return;
	}

	float sum = 0.0f;
	float factor;
	float t =0.f;

	float center = tex2D(floatTex,x,y);

 	for(int i = -r; i <= r; i++)
 	{
 		for(int j = -r; j <= r; j++)
 		{
 			//ushort curPix = tex2D(ushortTex,x + j,y + i);
 			float curPix = tex2D(floatTex,x + j,y + i);
			// remove boundary pixels.
			if(curPix <= 0.0f){ od[y * w + x] = 0.0f; return ; }
 			//Euclidean Distance (x, y, d) = exp(-(|x - y| / d)^2 / 2)
 			factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
 					 euclideanDist(curPix,center,e_d);             //range factor
 
 			t += factor * curPix;
 			sum += factor;
 		}
 	}

	// convert depth scale from mm to m. (x0.001) 
	//od[y * w + x] = 0.001f*t/sum;
	od[y * w + x] = t/sum;
}

__global__ void
d_inverse_bilateral_filter(float *od,int w,int h,
				   float e_d,int r)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x >= w || y >= h)
	{
		return;
	}

	float sum = 0.0f;
	float factor;
	float t =0.f;

	float center = tex2D(floatTex,x,y);

	for(int i = -r; i <= r; i++)
	{
		for(int j = -r; j <= r; j++)
		{
			//ushort curPix = tex2D(ushortTex,x + j,y + i);
			float curPix = tex2D(floatTex,x + j,y + i);
			// remove boundary pixels.
			if(curPix <= 0.0f){ od[y * w + x] = 0.0f; return ; }
			//Euclidean Distance (x, y, d) = exp(-(|x - y| / d)^2 / 2)
			factor = cGaussian[i + r] * cGaussian[j + r] *     //domain factor
					 euclideanSim(curPix,center,e_d);             //range factor

			t += factor * curPix;
			sum += factor;
		}
	}

	// convert depth scale from mm to m. (x0.001) 
	//od[y * w + x] = 0.001f*t/sum;
	od[y * w + x] = t/sum;
}

 /*
    Perform 2D bilateral filter on image using CUDA

    Parameters:
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    e_d    - euclidean delta
    radius - filter radius
    iterations - number of iterations
*/

// ===============================================================================
// Host functions.
// ===============================================================================
 __host__ void initTexture(int width,int height)
 {
// 	 if(dMap) checkCudaErrors(hipFree(dMap));
// 	 if(dTemp) checkCudaErrors(hipFree(dTemp));
// 	 if(hMap) delete[] hMap; 

	 if(dMapF) checkCudaErrors(hipFree(dMapF));
	 if(dTempF) checkCudaErrors(hipFree(dTempF));
	 if(hMapF) delete[] hMapF;

	 // texture setting.
// 	 ushortTex.addressMode[0] = hipAddressModeMirror;
// 	 ushortTex.addressMode[1] = hipAddressModeMirror;
// 	 ushortTex.filterMode = hipFilterModeLinear;
// 	 ushortTex.normalized = false; // don't access with normalized texture coords

	 floatTex.addressMode[0] = hipAddressModeMirror;
	 floatTex.addressMode[1] = hipAddressModeMirror;
	 floatTex.filterMode = hipFilterModeLinear;
	 floatTex.normalized = false; // don't access with normalized texture coords

	 int stride = iAlignUp(width, CV_CUDA_BLOCK_SIZE_X);

	 dwidth = width; dheight = height;
	 //pitchF = stride*sizeof(float);

	 // copy image data to array
// 	 hMap = new ushort[width*height];
// 	 checkCudaErrors(hipMallocPitch((void **)&dMap,&dpitch,sizeof(ushort)*width,height));
// 	 checkCudaErrors(hipMallocPitch((void **)&dTemp,&dpitch,sizeof(ushort)*width,height));

	 hMapF = new float[width*height];
	 checkCudaErrors(hipMallocPitch((void **)&dMapF,&dpitchF,sizeof(float)*width,height));
	 checkCudaErrors(hipMallocPitch((void **)&dTempF,&dpitchF,sizeof(float)*width,height));
 }

//  __host__ bool importTexture(int width,int height,const float *hMap)
//  {
// 	 if(width != dwidth || height != dheight) return false;
// 
// 	 checkCudaErrors(hipMemcpy2D(dMap,pitchF,hMap,sizeof(float)*width,
// 								  sizeof(float)*width,height,hipMemcpyHostToDevice));
// 	 return true;
//  }

 __host__ void freeTextures()
 {

// 	 if(dMap) checkCudaErrors(hipFree(dMap));		dMap = NULL;
// 	 if(dTemp) checkCudaErrors(hipFree(dTemp));	dTemp = NULL;
// 	 if(hMap) delete[] hMap;						hMap = NULL;

	 if(dMapF){ 
		 checkCudaErrors(hipFree(dMapF));	
		 hipUnbindTexture(floatTex);
	 }
	 dMapF = NULL;
	 if(dTempF) checkCudaErrors(hipFree(dTempF));  dTempF = NULL;
	 if(hMapF) delete[] hMapF;						hMapF = NULL;

	 dwidth = dheight = dpitch = 1;
 }

 /*
    Because a 2D gaussian mask is symmetry in row and column,
    here only generate a 1D mask, and use the product by row
    and column index later.

    1D gaussian distribution :
        g(x, d) -- C * exp(-x^2/d^2), C is a constant amplifier

    parameters:
    og - output gaussian array in global memory
    delta - the 2nd parameter 'd' in the above function
    radius - half of the filter size
             (total filter size = 2 * radius + 1)
*/
__host__ void updateGaussian(float delta, int radius)
{
    float  fGaussian[64];

    for (int i = 0; i < 2*radius + 1; ++i)
    {
        float x = i-radius;
        fGaussian[i] = expf(-(x*x) / (2*delta*delta));
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cGaussian), fGaussian, sizeof(float)*(2*radius+1)));
}

// for depth map.
__host__ bool bilateralFilterDepth(float *dDest,	
	const float *hMapDepth,
	int width,int height,
	float e_d,int radius,int iterations)
{
	// Bind the array to the texture
	if(width != dwidth || height != dheight) return false;

	int ww = width;
	int hh = height;

	//for(int i=0; i<ww*hh; i++) hMapF[i] = hMapDepth[i];
	// + copy depth map from host to device memory.
	hipMemcpy2D(dMapF,dpitchF,hMapDepth,sizeof(float)*ww,sizeof(float)*ww,hh,hipMemcpyHostToDevice);
	// + bind texture memory.
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	checkCudaErrors(hipBindTexture2D(0,floatTex,dMapF,desc,ww,hh,sizeof(float)*ww));
 
 	// Perform bilateral filtering.
 	 dim3 blockSize(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
 	 dim3 gridSize(iAlignUp(ww,blockSize.x),iAlignUp(hh,blockSize.y));
 
 	 d_bilateral_filter<<< gridSize,blockSize>>>(
 		 dDest,ww,hh,e_d,radius);

	//checkCudaErrors(hipUnbindTexture(floatTex));

	return true;
}

// for depth map.
__host__ bool inverseBilateralFilter(float *dDest,
	const float *hMapDepth,
	int width,int height,
	float e_d,int radius,int iterations)
{
	// Bind the array to the texture
	if(width != dwidth || height != dheight) return false;

	int ww = width;
	int hh = height;

	//for(int i=0; i<ww*hh; i++) hMapF[i] = hMapDepth[i];
	// + copy depth map from host to device memory.
	hipMemcpy2D(dMapF,dpitchF,hMapDepth,sizeof(float)*ww,sizeof(float)*ww,hh,hipMemcpyHostToDevice);
	// + bind texture memory.
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	checkCudaErrors(hipBindTexture2D(0,floatTex,dMapF,desc,ww,hh,sizeof(float)*ww));

	// Perform bilateral filtering.
	dim3 blockSize(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	dim3 gridSize(iAlignUp(ww,blockSize.x),iAlignUp(hh,blockSize.y));

	d_inverse_bilateral_filter<<< gridSize,blockSize>>>(
		dDest,ww,hh,e_d,radius);

	//checkCudaErrors(hipUnbindTexture(floatTex));

	return true;
}




/////////////////////////////////////////////////////////////////////////////////////////////
// LGKvImageProcessor
/////////////////////////////////////////////////////////////////////////////////////////////
// *******************************************************
LGKvImageProcessor::LGKvImageProcessor()
// *******************************************************
{
	z_flag_init = false;
}

// *******************************************************
LGKvImageProcessor::~LGKvImageProcessor()
// *******************************************************
{
	freeTextures();
}

// *******************************************************
bool LGKvImageProcessor::i_Initialize(int ww, int hh)
// *******************************************************
{
	if(ww<=0 || hh<=0) return false;

	if(z_flag_init) freeTextures();
	initTexture(ww, hh);
	updateGaussian(gaussian_delta, filter_radius);

	z_flag_init = true;

	return true;
}

// *******************************************************
void LGKvImageProcessor::r_Release()
// *******************************************************
{
	freeTextures();
	z_flag_init = false;
}

// *******************************************************
bool LGKvImageProcessor::bfd_Bilateral_Filter_Depth(
	float *map_depth_filtered_dev,
	const float *map_depth_host,
	int ww,int hh)
// *******************************************************
{
	if(ww<=0 || hh<=0) return false;

	if(!z_flag_init) i_Initialize(ww, hh);

//  	hipMemcpy2D(map_depth_filtered_dev, sizeof(float)*ww, map_depth_host, 
//  		sizeof(float)*ww, sizeof(float)*ww, hh, hipMemcpyHostToDevice);

	bilateralFilterDepth(
		map_depth_filtered_dev,
		map_depth_host,
		ww,hh,
		euclidean_delta,
		filter_radius,
		iter_num);


	return true;
}

// *******************************************************
bool LGKvImageProcessor::bfd_Bilateral_Filter_Depth_Host(
	float *map_depth_filtered_host,
	const float *map_depth_host,
	int ww,int hh)
// *******************************************************
{
	if(ww<=0 || hh<=0) return false;

	if(!z_flag_init) i_Initialize(ww,hh);

	bilateralFilterDepth(
		dTempF,
		map_depth_host,
		ww,hh,
		euclidean_delta,
		filter_radius,
		iter_num);

	hipMemcpy2D(map_depth_filtered_host,sizeof(float)*ww,dTempF,
		sizeof(float)*ww,sizeof(float)*ww,hh,hipMemcpyDeviceToHost);

	return true;
}

// *******************************************************
bool LGKvImageProcessor::ibfd_Inverse_Bilateral_Filter_Depth_Host(
	float *map_depth_filtered_host,
	const float *map_depth_host,
	int ww,int hh)
// *******************************************************
{
	if(ww<=0 || hh<=0) return false;

	if(!z_flag_init) i_Initialize(ww,hh);

	inverseBilateralFilter(
		dTempF,
		map_depth_host,
		ww,hh,
		euclidean_delta,
		filter_radius,
		iter_num);

	hipMemcpy2D(map_depth_filtered_host,sizeof(float)*ww,dTempF,
		sizeof(float)*ww,sizeof(float)*ww,hh,hipMemcpyDeviceToHost);

	return true;
}