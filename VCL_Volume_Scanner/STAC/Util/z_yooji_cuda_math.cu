#include "hip/hip_runtime.h"
#include "../_yooji_2017_cuda_object_scanner.cuh"

// /////////////////////////////////////////////////////////////////////////////////////////////
// Set 1D array.
// /////////////////////////////////////////////////////////////////////////////////////////////
__global__ void set1D(float* in_vec, int in_dim, float in_value)
 {
 	int tidx = blockIdx.x*blockDim.x+threadIdx.x;
 	if(tidx < in_dim)	in_vec[tidx] = in_value;
 }
 __global__ void set1D(int* in_vec, int in_dim, int in_value)
 {
	 int tidx = blockIdx.x*blockDim.x+threadIdx.x;
	 if(tidx < in_dim)	in_vec[tidx] = in_value;
 }
 __global__ void set1D(short* in_vec, int in_dim, short in_value)
 {
	 int tidx = blockIdx.x*blockDim.x+threadIdx.x;
	 if(tidx < in_dim)	in_vec[tidx] = in_value;
 }
 __global__ void set1D(uchar* in_vec, int in_dim, uchar in_value)
 {
	 int tidx = blockIdx.x*blockDim.x+threadIdx.x;
	 if(tidx < in_dim)	in_vec[tidx] = in_value;
 }
 __global__ void set1D(bool* in_vec, int in_dim, bool in_value)
 {
	 int tidx = blockIdx.x*blockDim.x+threadIdx.x;
	 if(tidx < in_dim)	in_vec[tidx] = in_value;
 }
 
 // /////////////////////////////////////////////////////////////////////////////////////////////
 __host__ void setDeviceMem1D(float* in_vec, int in_dim, float in_value)
 {
  	int block_sz, grid_sz;
  	block_sz = 128;		grid_sz = in_dim/block_sz;	
 	set1D<<<grid_sz, block_sz>>>(in_vec, in_dim, in_value);
 }
 __host__ void setDeviceMem1D(int* in_vec, int in_dim, int in_value)
 {
	 int block_sz, grid_sz;
	 block_sz = 128;		grid_sz = in_dim/block_sz;
	 set1D<<<grid_sz, block_sz>>>(in_vec, in_dim, in_value);
 }
 __host__ void setDeviceMem1D(short* in_vec, int in_dim, short in_value)
 {
	 int block_sz, grid_sz;
	 block_sz = 128;		grid_sz = in_dim/block_sz;
	 set1D<<<grid_sz, block_sz>>>(in_vec, in_dim, in_value);
 }
 __host__ void setDeviceMem1D(uchar* in_vec, int in_dim, uchar in_value)
 {
	 int block_sz, grid_sz;
	 block_sz = 128;		grid_sz = in_dim/block_sz;
	 set1D<<<grid_sz, block_sz>>>(in_vec, in_dim, in_value);
 }
 __host__ void setDeviceMem1D(bool* in_vec, int in_dim, bool in_value)
 {
	 int block_sz, grid_sz;
	 block_sz = 128;		grid_sz = in_dim/block_sz;
	 set1D<<<grid_sz, block_sz>>>(in_vec, in_dim, in_value);
 }
 //////////////////////////////////////////////////////////////////////////

 // /////////////////////////////////////////////////////////////////////////////////////////////
 // Set 2D array.
 // /////////////////////////////////////////////////////////////////////////////////////////////
 __global__ void set2D(float* in_vec, int2 in_dim,float in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;

	 if(tx < in_dim.x && ty < in_dim.y)	in_vec[ty*in_dim.x + tx] = in_value;
 }
 __global__ void set2D(int* in_vec,int2 in_dim,int in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;

	 if(tx < in_dim.x && ty < in_dim.y)	in_vec[ty*in_dim.x + tx] = in_value;
 }
 __global__ void set2D(short* in_vec,int2 in_dim,short in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;

	 if(tx < in_dim.x && ty < in_dim.y)	in_vec[ty*in_dim.x + tx] = in_value;
 }
 __global__ void set2D(uchar* in_vec,int2 in_dim, uchar in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;

	 if(tx < in_dim.x && ty < in_dim.y)	in_vec[ty*in_dim.x + tx] = in_value;
 }
 __global__ void set2D(bool* in_vec,int2 in_dim,bool in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;

	 if(tx < in_dim.x && ty < in_dim.y)	in_vec[ty*in_dim.x + tx] = in_value;
 }

 // /////////////////////////////////////////////////////////////////////////////////////////////
 __host__ void setDeviceMem2D(float* in_vec,int2 in_dim,float in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y));

	 set2D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem2D(int* in_vec,int2 in_dim,int in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y));

	 set2D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem2D(short* in_vec,int2 in_dim,short in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y));

	 set2D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem2D(uchar* in_vec,int2 in_dim,uchar in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y));

	 set2D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem2D(bool* in_vec,int2 in_dim,bool in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y));

	 set2D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }

 // /////////////////////////////////////////////////////////////////////////////////////////////
 // Set 3D array.
 // /////////////////////////////////////////////////////////////////////////////////////////////
 __global__ void set3D(float* in_vec,int3 in_dim,float in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;
	 int tz = blockIdx.z*blockDim.z+threadIdx.z;

	 if(tx < in_dim.x && ty < in_dim.y && tz < in_dim.z)	
		 in_vec[tz*in_dim.x*in_dim.y + ty*in_dim.x + tx] = in_value;
 }
 __global__ void set3D(int* in_vec,int3 in_dim,int in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;
	 int tz = blockIdx.z*blockDim.z+threadIdx.z;

	 if(tx < in_dim.x && ty < in_dim.y && tz < in_dim.z)
		 in_vec[tz*in_dim.x*in_dim.y + ty*in_dim.x + tx] = in_value;
 }
 __global__ void set3D(short* in_vec,int3 in_dim,short in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;
	 int tz = blockIdx.z*blockDim.z+threadIdx.z;

	 if(tx < in_dim.x && ty < in_dim.y && tz < in_dim.z)
		in_vec[tz*in_dim.x*in_dim.y + ty*in_dim.x + tx] = in_value;
 }
 __global__ void set3D(uchar* in_vec,int3 in_dim,uchar in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;
	 int tz = blockIdx.z*blockDim.z+threadIdx.z;

	 if(tx < in_dim.x && ty < in_dim.y && tz < in_dim.z)
		in_vec[tz*in_dim.x*in_dim.y + ty*in_dim.x + tx] = in_value;
 }
 __global__ void set3D(bool* in_vec,int3 in_dim,bool in_value)
 {
	 int tx = blockIdx.x*blockDim.x+threadIdx.x;
	 int ty = blockIdx.y*blockDim.y+threadIdx.y;
	 int tz = blockIdx.z*blockDim.z+threadIdx.z;

	 if(tx < in_dim.x && ty < in_dim.y && tz < in_dim.z)
		 in_vec[tz*in_dim.x*in_dim.y + ty*in_dim.x + tx] = in_value;
 }

 // /////////////////////////////////////////////////////////////////////////////////////////////
 __host__ void setDeviceMem3D(float* in_vec,int3 in_dim,float in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y),iDivUp(in_dim.z,threads.z));

	 set3D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem3D(int* in_vec,int3 in_dim,int in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y),iDivUp(in_dim.z,threads.z));

	 set3D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem3D(short* in_vec,int3 in_dim,short in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y),iDivUp(in_dim.z,threads.z));

	 set3D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem3D(uchar* in_vec,int3 in_dim,uchar in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y),iDivUp(in_dim.z,threads.z));

	 set3D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }
 __host__ void setDeviceMem3D(bool* in_vec,int3 in_dim,bool in_value)
 {
	 dim3 threads(CV_CUDA_BLOCK_SIZE_X,CV_CUDA_BLOCK_SIZE_Y,CV_CUDA_BLOCK_SIZE_Z);
	 dim3 blocks(iDivUp(in_dim.x,threads.x),iDivUp(in_dim.y,threads.y),iDivUp(in_dim.z,threads.z));

	 set3D<<<blocks,threads>>>(in_vec,in_dim,in_value);
 }